#include "hip/hip_runtime.h"
#include <cassert>
#include <memory>
#include <queue>

#include "../profiling/memoryManager.hpp"
#include "../profiling/peakMemoryUsageProfiler.hpp"
#include "../utilities/configurationManager.hpp"
#include "../utilities/cudaGraphUtilities.hpp"
#include "../utilities/cudaUtilities.hpp"
#include "../utilities/utilities.hpp"
#include "../utilities/logger.hpp"
#include "executor.hpp"

namespace memopt {

// Method implementations for OptimizedCudaGraphCreator

void OptimizedCudaGraphCreator::beginCaptureOperation(const std::vector<hipGraphNode_t> &dependencies) {
  checkCudaErrors(hipStreamBeginCaptureToGraph(this->stream, this->graph, dependencies.data(), nullptr, dependencies.size(), hipStreamCaptureModeGlobal));
}

std::vector<hipGraphNode_t> OptimizedCudaGraphCreator::endCaptureOperation() {
  checkCudaErrors(hipStreamEndCapture(this->stream, &this->graph));
  return this->getNewLeafNodesAddedByLastCapture();
}

hipGraphNode_t OptimizedCudaGraphCreator::addEmptyNode(const std::vector<hipGraphNode_t> &dependencies) {
  hipGraphNode_t newEmptyNode;
  checkCudaErrors(hipGraphAddEmptyNode(&newEmptyNode, this->graph, dependencies.data(), dependencies.size()));
  visited[newEmptyNode] = true;
  return newEmptyNode;
}

std::vector<hipGraphNode_t> OptimizedCudaGraphCreator::getNewLeafNodesAddedByLastCapture() {
  // Get all nodes in the graph
  size_t numNodes;
  checkCudaErrors(hipGraphGetNodes(this->graph, nullptr, &numNodes));
  auto nodes = std::make_unique<hipGraphNode_t[]>(numNodes);
  checkCudaErrors(hipGraphGetNodes(this->graph, nodes.get(), &numNodes));

  // Get all edges in the graph
  size_t numEdges;
  checkCudaErrors(hipGraphGetEdges(this->graph, nullptr, nullptr, &numEdges));
  auto from = std::make_unique<hipGraphNode_t[]>(numEdges);
  auto to = std::make_unique<hipGraphNode_t[]>(numEdges);
  checkCudaErrors(hipGraphGetEdges(this->graph, from.get(), to.get(), &numEdges));

  // Track which nodes have outgoing edges
  std::map<hipGraphNode_t, bool> hasOutGoingEdge;
  for (int i = 0; i < numEdges; i++) {
    hasOutGoingEdge[from[i]] = true;
  }

  // Find new leaf nodes (not visited before and have no outgoing edges)
  std::vector<hipGraphNode_t> newLeafNodes;
  for (int i = 0; i < numNodes; i++) {
    auto &node = nodes[i];
    if (!visited[node]) {
      visited[node] = true;
      if (!hasOutGoingEdge[node]) {
        newLeafNodes.push_back(node);
      }
    }
  }

  return newLeafNodes;
}

Executor *Executor::instance = nullptr;

Executor *Executor::getInstance() {
  if (instance == nullptr) {
    instance = new Executor();
  }
  return instance;
}

/*
 * executeOptimizedGraph - Executes a CUDA graph with memory optimization
 *
 * This function executes a computation graph that has been optimized to reduce memory usage
 * by dynamically managing data transfers between the main GPU and storage (host memory or secondary GPU).
 * It enables processing of workloads larger than would fit in GPU memory alone.
 *
 * Parameters:
 * - optimizedGraph: The optimized computation graph to execute
 * - executeRandomTask: Callback to execute specific computation tasks
 * - runningTime: Output parameter to store the execution time
 * - managedDeviceArrayToHostArrayMap: Mapping between device (key) and host (host) memory addresses
 */
void Executor::executeOptimizedGraph(
  OptimizationOutput &optimizedGraph,
  ExecuteRandomTask executeRandomTask,
  float &runningTime,
  std::map<void *, void *> &managedDeviceArrayToHostArrayMap
  // Register if 
) {
  LOG_TRACE_WITH_INFO("Initialize");

  // Reset the memory mapping
  managedDeviceArrayToHostArrayMap.clear();

  // Create CUDA resources 
  hipGraph_t graph;
  checkCudaErrors(hipGraphCreate(&graph, 0));

  hipStream_t stream;
  checkCudaErrors(hipStreamCreate(&stream));

  auto optimizedCudaGraphCreator = std::make_unique<OptimizedCudaGraphCreator>(stream, graph);

  //----------------------------------------------------------------------
  // STEP 1: Prepare the graph for topological traversal using Kahn's algorithm
  //----------------------------------------------------------------------
  
  // Calculate in-degrees for each node in the optimized graph
  std::map<int, int> inDegrees;
  for (auto &[u, outEdges] : optimizedGraph.edges) {
    for (auto &v : outEdges) {
      inDegrees[v] += 1;
    }
  }

  // Find root nodes (nodes with no dependencies)
  std::queue<int> nodesToExecute;
  std::vector<int> rootNodes;
  for (auto &u : optimizedGraph.nodes) {
    if (inDegrees[u] == 0) {
      nodesToExecute.push(u);
      rootNodes.push_back(u);
    }
  }

  //----------------------------------------------------------------------
  // STEP 2: Configure the device and memory settings
  //----------------------------------------------------------------------
  
  // Set up device configuration for data movement
  int mainDeviceId = ConfigurationManager::getConfig().execution.mainDeviceId;
  int storageDeviceId = hipCpuDeviceId;  // Default: use host memory as storage
  hipMemcpyKind prefetchMemcpyKind = hipMemcpyHostToDevice;
  hipMemcpyKind offloadMemcpyKind = hipMemcpyDeviceToHost;

  // If NVLink is available, use a second GPU as storage instead of host memory
  if (ConfigurationManager::getConfig().execution.useNvlink) {
    storageDeviceId = ConfigurationManager::getConfig().execution.storageDeviceId;
    prefetchMemcpyKind = hipMemcpyDeviceToDevice;
    offloadMemcpyKind = hipMemcpyDeviceToDevice;
    enablePeerAccessForNvlink(ConfigurationManager::getConfig().execution.mainDeviceId, ConfigurationManager::getConfig().execution.storageDeviceId);
  }

  //----------------------------------------------------------------------
  // STEP 3: Initialize managed data distribution
  //----------------------------------------------------------------------
  
  LOG_TRACE_WITH_INFO("Initialize managed data distribution");

  // Move all managed data to storage (host or secondary GPU)
  auto& memManager = MemoryManager::getInstance();
  auto& managedAddresses = memManager.getEditableManagedAddresses();
  for (auto ptr : managedAddresses) {
    void *newPtr;
    if (ConfigurationManager::getConfig().execution.useNvlink) {
      // Allocate on secondary GPU
      checkCudaErrors(hipSetDevice(storageDeviceId));
      checkCudaErrors(hipMalloc(&newPtr, memManager.getSize(ptr)));
    } else {
      // Allocate on host memory
      checkCudaErrors(hipHostMalloc(&newPtr, memManager.getSize(ptr)));
    }

    // Create mapping and copy data to storage
    managedDeviceArrayToHostArrayMap[ptr] = newPtr;
    checkCudaErrors(hipMemcpy(
      newPtr,
      ptr,
      memManager.getSize(ptr),
      hipMemcpyDefault
    ));
    checkCudaErrors(hipFree(ptr));  // Free original device memory
  }
  
  // Switch back to main GPU
  checkCudaErrors(hipSetDevice(mainDeviceId));
  checkCudaErrors(hipDeviceSynchronize());

  //----------------------------------------------------------------------
  // STEP 4: Initialize data that needs to be on the device at the start
  //----------------------------------------------------------------------
  
  // Track memory addresses that have been updated (device copies)
  // address mapping original to new device address
  memManager.clearCurrentMappings();
  // Create a subgraph for initial data prefetching
  checkCudaErrors(hipStreamBeginCapture(stream, hipStreamCaptureModeGlobal));
  for (auto arrayId : optimizedGraph.arraysInitiallyAllocatedOnDevice) {
    auto ptr = memManager.getPointerByArrayId(arrayId);
    auto size = memManager.getSizeByArrayId(arrayId);
    auto newPtr = managedDeviceArrayToHostArrayMap[ptr];

    // Allocate on device and copy data from storage
    void *devicePtr;
    checkCudaErrors(hipMallocAsync(&devicePtr, size, stream));
    checkCudaErrors(hipMemcpyAsync(devicePtr, newPtr, size, prefetchMemcpyKind, stream));
    memManager.updateCurrentMapping(ptr, devicePtr);  // Update the address mapping
  }
  
  // End capture and instantiate the initial data distribution graph
  hipGraph_t graphForInitialDataDistribution;
  checkCudaErrors(hipStreamEndCapture(stream, &graphForInitialDataDistribution));

  // Execute the initial data distribution
  hipGraphExec_t graphExecForInitialDataDistribution;
  checkCudaErrors(hipGraphInstantiate(&graphExecForInitialDataDistribution, graphForInitialDataDistribution, nullptr, nullptr, 0));
  checkCudaErrors(hipGraphLaunch(graphExecForInitialDataDistribution, stream));
  checkCudaErrors(hipDeviceSynchronize());

  //----------------------------------------------------------------------
  // STEP 5: Build the optimized execution graph by processing nodes in topological order
  //----------------------------------------------------------------------
  
  LOG_TRACE_WITH_INFO("Record nodes to a new CUDA Graph");

  // Maps nodes to their dependencies in the CUDA graph
  std::map<int, std::vector<hipGraphNode_t>> nodeToDependentNodesMap;

  // Kahn's Algorithm for topological sort and graph construction
  while (!nodesToExecute.empty()) {
    // Get the next node to process
    auto u = nodesToExecute.front();
    nodesToExecute.pop();

    std::vector<hipGraphNode_t> newLeafNodes;
    auto nodeType = optimizedGraph.nodeIdToNodeTypeMap[u];
    
    // Process different node types
    if (nodeType == OptimizationOutput::NodeType::dataMovement) {
      //----------------------------------------------------------------------
      // STEP 5a: Handle data movement nodes (prefetch or offload)
      //----------------------------------------------------------------------
      
      optimizedCudaGraphCreator->beginCaptureOperation(nodeToDependentNodesMap[u]);
      auto &dataMovement = optimizedGraph.nodeIdToDataMovementMap[u];
      auto dataMovementAddress = memManager.getPointerByArrayId(dataMovement.arrayId);
      auto dataMovementSize = memManager.getSizeByArrayId(dataMovement.arrayId);
      
      if (dataMovement.direction == OptimizationOutput::DataMovement::Direction::hostToDevice) {
        // PREFETCH: Move data from storage to device
        void *devicePtr;
        checkCudaErrors(hipMallocAsync(&devicePtr, dataMovementSize, stream));
        checkCudaErrors(hipMemcpyAsync(
          devicePtr,
          managedDeviceArrayToHostArrayMap[dataMovementAddress],
          dataMovementSize,
          prefetchMemcpyKind,
          stream
        ));
        memManager.updateCurrentMapping(dataMovementAddress, devicePtr);
      } else {
        // OFFLOAD: Move data from device back to storage and free device memory
        void *devicePtr = memManager.getCurrentAddressMap().at(dataMovementAddress);
        checkCudaErrors(hipMemcpyAsync(
          managedDeviceArrayToHostArrayMap[dataMovementAddress],
          devicePtr,
          dataMovementSize,
          offloadMemcpyKind,
          stream
        ));
        checkCudaErrors(hipFreeAsync(devicePtr, stream));
        memManager.removeCurrentMapping(dataMovementAddress);
      }
      
      checkCudaErrors(hipPeekAtLastError());
      newLeafNodes = optimizedCudaGraphCreator->endCaptureOperation();
      checkCudaErrors(hipPeekAtLastError());
      
    } else if (nodeType == OptimizationOutput::NodeType::task) {
      //----------------------------------------------------------------------
      // STEP 5b: Handle computation task nodes
      //----------------------------------------------------------------------
      
      optimizedCudaGraphCreator->beginCaptureOperation(nodeToDependentNodesMap[u]);
      // Execute the task with current memory address mapping
      executeRandomTask(
        optimizedGraph.nodeIdToTaskIdMap[u],
        memManager.getEditableCurrentAddressMap(),
        stream
      );
      newLeafNodes = optimizedCudaGraphCreator->endCaptureOperation();
      
    } else if (nodeType == OptimizationOutput::NodeType::empty) {
      //----------------------------------------------------------------------
      // STEP 5c: Handle empty nodes (for dependencies)
      //----------------------------------------------------------------------
      
      newLeafNodes.push_back(
        optimizedCudaGraphCreator->addEmptyNode(nodeToDependentNodesMap[u])
      );
    } else {
      LOG_TRACE_WITH_INFO("Unsupported node type: %d", nodeType);
      exit(-1);
    }

    // Update dependencies and process nodes that have all dependencies satisfied
    for (auto &v : optimizedGraph.edges[u]) {
      inDegrees[v]--;

      // Add dependencies for the next nodes
      nodeToDependentNodesMap[v].insert(
        nodeToDependentNodesMap[v].end(),
        newLeafNodes.begin(),
        newLeafNodes.end()
      );

      // If all dependencies are satisfied, add to the queue
      if (inDegrees[v] == 0) {
        nodesToExecute.push(v);
      }
    }
  }

  // Export graph for debugging/visualization
  LOG_TRACE_WITH_INFO("Printing the new CUDA Graph to newGraph.dot");
  checkCudaErrors(hipGraphDebugDotPrint(graph, "newGraph.dot", 0));

  //----------------------------------------------------------------------
  // STEP 6: Execute the optimized graph
  //----------------------------------------------------------------------
  
  LOG_TRACE_WITH_INFO("Execute the new CUDA Graph");
  
  // Set up profiling if requested
  PeakMemoryUsageProfiler peakMemoryUsageProfiler;
  CudaEventClock cudaEventClock;
  
  // Instantiate the graph for execution
  hipGraphExec_t graphExec;
  checkCudaErrors(hipGraphInstantiate(&graphExec, graph, nullptr, nullptr, 0));

  // Upload the graph to the device for faster execution
  checkCudaErrors(hipGraphUpload(graphExec, stream));
  checkCudaErrors(hipStreamSynchronize(stream));

  // Start memory usage profiling if requested
  if (ConfigurationManager::getConfig().execution.measurePeakMemoryUsage) {
    peakMemoryUsageProfiler.start();
  }

  // Execute and time the graph
  cudaEventClock.start();
  checkCudaErrors(hipGraphLaunch(graphExec, stream));
  cudaEventClock.end();
  checkCudaErrors(hipDeviceSynchronize());

  // Report peak memory usage if requested
  if (ConfigurationManager::getConfig().execution.measurePeakMemoryUsage) {
    const auto peakMemoryUsage = peakMemoryUsageProfiler.end();
    LOG_TRACE_WITH_INFO(
      "Peak memory usage (MiB): %.2f",
      static_cast<float>(peakMemoryUsage) / 1024.0 / 1024.0
    );
  }

  //----------------------------------------------------------------------
  // STEP 7: Clean up resources and copy any remaining data back to storage
  //----------------------------------------------------------------------
  
  LOG_TRACE_WITH_INFO("Clean up");
  
  // Copy any remaining device data back to storage
  auto &currentAddressMap = memManager.getEditableCurrentAddressMap();
  for (auto &[oldAddr, newAddr] : currentAddressMap) {
    checkCudaErrors(hipMemcpy(
      managedDeviceArrayToHostArrayMap[oldAddr],
      newAddr,
      memManager.getSize(oldAddr),
      offloadMemcpyKind
    ));
    checkCudaErrors(hipFree(newAddr));
  }
  checkCudaErrors(hipDeviceSynchronize());

  // Clean up CUDA resources
  checkCudaErrors(hipGraphExecDestroy(graphExecForInitialDataDistribution));
  checkCudaErrors(hipGraphExecDestroy(graphExec));
  checkCudaErrors(hipGraphDestroy(graphForInitialDataDistribution));
  checkCudaErrors(hipGraphDestroy(graph));
  checkCudaErrors(hipStreamDestroy(stream));

  // Disable peer access if using NVLink
  if (ConfigurationManager::getConfig().execution.useNvlink) {
    disablePeerAccessForNvlink(mainDeviceId, storageDeviceId);
  }

  // Store the execution time
  runningTime = cudaEventClock.getTimeInSeconds();
}

void Executor::executeOptimizedGraphRepeatedly(
  OptimizationOutput &optimizedGraph,
  ExecuteRandomTask executeRandomTask,
  ShouldContinue shouldContinue,
  int &numIterations,
  float &runningTime,
  std::map<void *, void *> &managedDeviceArrayToHostArrayMap
) {
  LOG_TRACE_WITH_INFO("Initialize");

  // Reset the memory mapping
  managedDeviceArrayToHostArrayMap.clear();

  // Create CUDA resources
  hipGraph_t graph;
  checkCudaErrors(hipGraphCreate(&graph, 0));

  hipStream_t stream;
  checkCudaErrors(hipStreamCreate(&stream));

  auto optimizedCudaGraphCreator = std::make_unique<OptimizedCudaGraphCreator>(stream, graph);

  //----------------------------------------------------------------------
  // STEP 1: Prepare the graph for topological traversal using Kahn's algorithm
  //----------------------------------------------------------------------
  
  // Calculate in-degrees for each node in the optimized graph
  std::map<int, int> inDegrees;
  for (auto &[u, outEdges] : optimizedGraph.edges) {
    for (auto &v : outEdges) {
      inDegrees[v] += 1;
    }
  }

  // Find root nodes (nodes with no dependencies)
  std::queue<int> nodesToExecute;
  std::vector<int> rootNodes;
  for (auto &u : optimizedGraph.nodes) {
    if (inDegrees[u] == 0) {
      nodesToExecute.push(u);
      rootNodes.push_back(u);
    }
  }

  //----------------------------------------------------------------------
  // STEP 2: Configure the device and memory settings
  //----------------------------------------------------------------------
  
  // Set up device configuration for data movement
  int mainDeviceId = ConfigurationManager::getConfig().execution.mainDeviceId;
  int storageDeviceId = hipCpuDeviceId;  // Default: use host memory as storage
  hipMemcpyKind prefetchMemcpyKind = hipMemcpyHostToDevice;
  hipMemcpyKind offloadMemcpyKind = hipMemcpyDeviceToHost;

  // If NVLink is available, use a second GPU as storage instead of host memory
  if (ConfigurationManager::getConfig().execution.useNvlink) {
    storageDeviceId = ConfigurationManager::getConfig().execution.storageDeviceId;
    prefetchMemcpyKind = hipMemcpyDeviceToDevice;
    offloadMemcpyKind = hipMemcpyDeviceToDevice;
    enablePeerAccessForNvlink(ConfigurationManager::getConfig().execution.mainDeviceId, ConfigurationManager::getConfig().execution.storageDeviceId);
  }

  //----------------------------------------------------------------------
  // STEP 3: Initialize managed data distribution
  //----------------------------------------------------------------------
  
  LOG_TRACE_WITH_INFO("Initialize managed data distribution");

  // Move all managed data to storage (host or secondary GPU)
  MemoryManager::getInstance().configureStorage(
    mainDeviceId, 
    storageDeviceId, 
    ConfigurationManager::getConfig().execution.useNvlink
  );
  MemoryManager::getInstance().offloadAllManagedMemoryToStorage(managedDeviceArrayToHostArrayMap);
  
  // Store in internal storage map for future use
  MemoryManager::getInstance().getEditableDeviceToHostArrayMap() = managedDeviceArrayToHostArrayMap;
  
  // Switch back to main GPU
  checkCudaErrors(hipSetDevice(mainDeviceId));
  checkCudaErrors(hipDeviceSynchronize());

  // Start timing the graph creation process
  SystemWallClock clock;
  clock.start();

  LOG_TRACE_WITH_INFO("Record nodes to a new CUDA Graph");

  //----------------------------------------------------------------------
  // STEP 4: Build the optimized execution graph
  //----------------------------------------------------------------------
  
  // Track memory addresses that have been updated (device copies)
  auto &memManager = MemoryManager::getInstance();
  memManager.clearCurrentMappings();
  // DIFFERENCE: Initial data allocation is integrated with main graph capture
  // rather than as a separate step like in executeOptimizedGraph
  std::vector<hipGraphNode_t> newLeafNodes;
  for (auto arrayId : optimizedGraph.arraysInitiallyAllocatedOnDevice) {
    auto ptr = memManager.getPointerByArrayId(arrayId);
    auto size = memManager.getSizeByArrayId(arrayId);
    auto newPtr = managedDeviceArrayToHostArrayMap[ptr];

    // Allocate on device and copy data from storage
    void *devicePtr;
    optimizedCudaGraphCreator->beginCaptureOperation(newLeafNodes);
    checkCudaErrors(hipMallocAsync(&devicePtr, size, stream));
    checkCudaErrors(hipMemcpyAsync(devicePtr, newPtr, size, prefetchMemcpyKind, stream));
    newLeafNodes = optimizedCudaGraphCreator->endCaptureOperation();
    memManager.updateCurrentMapping(ptr, devicePtr);
  }

  // Maps nodes to their dependencies in the CUDA graph
  std::map<int, std::vector<hipGraphNode_t>> nodeToDependentNodesMap;

  // Set up dependencies for root nodes
  for (auto u : rootNodes) {
    nodeToDependentNodesMap[u] = newLeafNodes;
  }

  // Process nodes in topological order (Kahn's Algorithm)
  while (!nodesToExecute.empty()) {
    // Get the next node to process
    auto u = nodesToExecute.front();
    nodesToExecute.pop();

    newLeafNodes.clear();

    // Process different node types
    auto nodeType = optimizedGraph.nodeIdToNodeTypeMap[u];
    if (nodeType == OptimizationOutput::NodeType::dataMovement) {
      // Handle data movement nodes (prefetch or offload)
      optimizedCudaGraphCreator->beginCaptureOperation(nodeToDependentNodesMap[u]);
      auto &dataMovement = optimizedGraph.nodeIdToDataMovementMap[u];
      auto dataMovementAddress = memManager.getPointerByArrayId(dataMovement.arrayId);
      auto dataMovementSize = memManager.getSizeByArrayId(dataMovement.arrayId);
      
      if (dataMovement.direction == OptimizationOutput::DataMovement::Direction::hostToDevice) {
        // PREFETCH: Move data from storage to device
        void *devicePtr;
        checkCudaErrors(hipMallocAsync(&devicePtr, dataMovementSize, stream));
        checkCudaErrors(hipMemcpyAsync(
          devicePtr,
          managedDeviceArrayToHostArrayMap[dataMovementAddress],
          dataMovementSize,
          prefetchMemcpyKind,
          stream
        ));
        memManager.updateCurrentMapping(dataMovementAddress, devicePtr);
      } else {
        // OFFLOAD: Move data from device back to storage and free device memory
        void *devicePtr = memManager.getCurrentAddressMap().at(dataMovementAddress);
        checkCudaErrors(hipMemcpyAsync(
          managedDeviceArrayToHostArrayMap[dataMovementAddress],
          devicePtr,
          dataMovementSize,
          offloadMemcpyKind,
          stream
        ));
        checkCudaErrors(hipFreeAsync(devicePtr, stream));
        memManager.removeCurrentMapping(dataMovementAddress);
      }
      checkCudaErrors(hipPeekAtLastError());
      newLeafNodes = optimizedCudaGraphCreator->endCaptureOperation();
      checkCudaErrors(hipPeekAtLastError());
    } else if (nodeType == OptimizationOutput::NodeType::task) {
      // Handle computation task nodes
      optimizedCudaGraphCreator->beginCaptureOperation(nodeToDependentNodesMap[u]);
      // Execute the task with current memory address mapping
      executeRandomTask(
        optimizedGraph.nodeIdToTaskIdMap[u],
        memManager.getEditableCurrentAddressMap(),
        stream
      );
      newLeafNodes = optimizedCudaGraphCreator->endCaptureOperation();
    } else if (nodeType == OptimizationOutput::NodeType::empty) {
      // Handle empty nodes (for dependencies)
      newLeafNodes.push_back(
        optimizedCudaGraphCreator->addEmptyNode(nodeToDependentNodesMap[u])
      );
    } else {
      LOG_TRACE_WITH_INFO("Unsupported node type: %d", nodeType);
      exit(-1);
    }

    // Update dependencies and process nodes that have all dependencies satisfied
    for (auto &v : optimizedGraph.edges[u]) {
      inDegrees[v]--;

      // Add dependencies for the next nodes
      nodeToDependentNodesMap[v].insert(
        nodeToDependentNodesMap[v].end(),
        newLeafNodes.begin(),
        newLeafNodes.end()
      );

      // If all dependencies are satisfied, add to the queue
      if (inDegrees[v] == 0) {
        nodesToExecute.push(v);
      }
    }
  }

  // Add cleanup operations for any remaining device memory
  newLeafNodes = getNodesWithZeroOutDegree(graph);
  auto &currentAddressMap = memManager.getEditableCurrentAddressMap(); 
  for (auto &[oldAddr, newAddr] : currentAddressMap) {
    // Copy any remaining device data back to storage and free device memory
    optimizedCudaGraphCreator->beginCaptureOperation(newLeafNodes);
    checkCudaErrors(hipMemcpyAsync(
      managedDeviceArrayToHostArrayMap[oldAddr],
      newAddr,
      memManager.getSize(oldAddr),
      offloadMemcpyKind,
      stream
    ));
    checkCudaErrors(hipFreeAsync(newAddr, stream));
    newLeafNodes = optimizedCudaGraphCreator->endCaptureOperation();
  }
  checkCudaErrors(hipDeviceSynchronize());

  // Report time taken to build the graph
  clock.end();
  LOG_TRACE_WITH_INFO("Time taken for recording graph: %.6f", clock.getTimeInSeconds());

  // Export graph for debugging/visualization
  LOG_TRACE_WITH_INFO("Printing the new CUDA Graph to newGraph.dot");
  checkCudaErrors(hipGraphDebugDotPrint(graph, "newGraph.dot", 0));

  //----------------------------------------------------------------------
  // STEP 5: Execute the optimized graph repeatedly
  //----------------------------------------------------------------------
  
  LOG_TRACE_WITH_INFO("Execute the new CUDA Graph");
  
  // Set up profiling if requested
  PeakMemoryUsageProfiler peakMemoryUsageProfiler;
  CudaEventClock cudaEventClock;
  
  // Instantiate the graph for execution
  hipGraphExec_t graphExec;
  checkCudaErrors(hipGraphInstantiate(&graphExec, graph, nullptr, nullptr, 0));

  // Upload the graph to the device for faster execution
  checkCudaErrors(hipGraphUpload(graphExec, stream));
  checkCudaErrors(hipStreamSynchronize(stream));

  // Start memory usage profiling if requested
  if (ConfigurationManager::getConfig().execution.measurePeakMemoryUsage) {
    peakMemoryUsageProfiler.start();
  }

  // Initialize iteration counter
  numIterations = 0;

  // Execute and time the graph
  cudaEventClock.start();
  // DIFFERENCE: Execute graph repeatedly until termination condition is met
  while (shouldContinue()) {
    checkCudaErrors(hipGraphLaunch(graphExec, stream));
    numIterations++;
    checkCudaErrors(hipDeviceSynchronize());
  }
  cudaEventClock.end();
  checkCudaErrors(hipDeviceSynchronize());

  // Report peak memory usage if requested
  if (ConfigurationManager::getConfig().execution.measurePeakMemoryUsage) {
    const auto peakMemoryUsage = peakMemoryUsageProfiler.end();
    LOG_TRACE_WITH_INFO(
      "Peak memory usage (MiB): %.2f",
      static_cast<float>(peakMemoryUsage) / 1024.0 / 1024.0
    );
  }

  //----------------------------------------------------------------------
  // STEP 6: Clean up resources
  //----------------------------------------------------------------------
  
  LOG_TRACE_WITH_INFO("Clean up");
  
  // Clean up CUDA resources
  checkCudaErrors(hipGraphExecDestroy(graphExec));
  checkCudaErrors(hipGraphDestroy(graph));
  checkCudaErrors(hipStreamDestroy(stream));

  // Disable peer access if using NVLink
  if (ConfigurationManager::getConfig().execution.useNvlink) {
    disablePeerAccessForNvlink(mainDeviceId, storageDeviceId);
  }

  // Store the execution time
  runningTime = cudaEventClock.getTimeInSeconds();
}

/*
 * executeOptimizedGraphBase - Executes a CUDA graph with memory optimization
 *
 * This function executes a computation graph that has been optimized to reduce memory usage
 * by dynamically managing data transfers between the main GPU and storage (host memory or secondary GPU).
 * It enables processing of workloads larger than would fit in GPU memory alone.
 *
 * Parameters:
 * - optimizedGraph: The optimized computation graph to execute
 * - executeRandomTask: Callback to execute specific computation tasks
 * - runningTime: Output parameter to store the execution time
 * - managedDeviceArrayToHostArrayMap: Mapping between device (key) and host (host) memory addresses
 */
void Executor::executeOptimizedGraph(
  OptimizationOutput &optimizedGraph,
  ExecuteRandomTaskBase executeRandomTaskBase,
  float &runningTime,
  std::map<void *, void *> &managedDeviceArrayToHostArrayMap
  // Register if 
) {
  LOG_TRACE_WITH_INFO("Initialize");

  // Reset the memory mapping
  managedDeviceArrayToHostArrayMap.clear();

  // Create CUDA resources 
  hipGraph_t graph;
  checkCudaErrors(hipGraphCreate(&graph, 0));

  hipStream_t stream;
  checkCudaErrors(hipStreamCreate(&stream));

  auto optimizedCudaGraphCreator = std::make_unique<OptimizedCudaGraphCreator>(stream, graph);

  //----------------------------------------------------------------------
  // STEP 1: Prepare the graph for topological traversal using Kahn's algorithm
  //----------------------------------------------------------------------
  
  // Calculate in-degrees for each node in the optimized graph
  std::map<int, int> inDegrees;
  for (auto &[u, outEdges] : optimizedGraph.edges) {
    for (auto &v : outEdges) {
      inDegrees[v] += 1;
    }
  }

  // Find root nodes (nodes with no dependencies)
  std::queue<int> nodesToExecute;
  std::vector<int> rootNodes;
  for (auto &u : optimizedGraph.nodes) {
    if (inDegrees[u] == 0) {
      nodesToExecute.push(u);
      rootNodes.push_back(u);
    }
  }

  //----------------------------------------------------------------------
  // STEP 2: Configure the device and memory settings
  //----------------------------------------------------------------------
  
  // Set up device configuration for data movement
  int mainDeviceId = ConfigurationManager::getConfig().execution.mainDeviceId;
  int storageDeviceId = hipCpuDeviceId;  // Default: use host memory as storage
  hipMemcpyKind prefetchMemcpyKind = hipMemcpyHostToDevice;
  hipMemcpyKind offloadMemcpyKind = hipMemcpyDeviceToHost;

  // If NVLink is available, use a second GPU as storage instead of host memory
  if (ConfigurationManager::getConfig().execution.useNvlink) {
    storageDeviceId = ConfigurationManager::getConfig().execution.storageDeviceId;
    prefetchMemcpyKind = hipMemcpyDeviceToDevice;
    offloadMemcpyKind = hipMemcpyDeviceToDevice;
    enablePeerAccessForNvlink(ConfigurationManager::getConfig().execution.mainDeviceId, ConfigurationManager::getConfig().execution.storageDeviceId);
  }

  //----------------------------------------------------------------------
  // STEP 3: Initialize managed data distribution
  //----------------------------------------------------------------------
  
  LOG_TRACE_WITH_INFO("Initialize managed data distribution");
  MemoryManager::getInstance().configureStorage(
    mainDeviceId, 
    storageDeviceId, 
    ConfigurationManager::getConfig().execution.useNvlink
  );
  MemoryManager::getInstance().offloadAllManagedMemoryToStorage(managedDeviceArrayToHostArrayMap);
  // Old implementation has been replaced with the MemoryManager API call above
  
  // Switch back to main GPU
  checkCudaErrors(hipSetDevice(mainDeviceId));
  checkCudaErrors(hipDeviceSynchronize());

  //----------------------------------------------------------------------
  // STEP 4: Initialize data that needs to be on the device at the start
  //----------------------------------------------------------------------
  
  // Track memory addresses that have been updated (device copies)
  auto &memManager = MemoryManager::getInstance();
  memManager.clearCurrentMappings();
  // Create a subgraph for initial data prefetching
  checkCudaErrors(hipStreamBeginCapture(stream, hipStreamCaptureModeGlobal));
  for (auto arrayId : optimizedGraph.arraysInitiallyAllocatedOnDevice) {
    auto ptr = memManager.getPointerByArrayId(arrayId);
    auto size = memManager.getSizeByArrayId(arrayId);
    auto newPtr = managedDeviceArrayToHostArrayMap[ptr];

    // Allocate on device and copy data from storage
    void *devicePtr;
    checkCudaErrors(hipMallocAsync(&devicePtr, size, stream));
    checkCudaErrors(hipMemcpyAsync(devicePtr, newPtr, size, prefetchMemcpyKind, stream));
    memManager.updateCurrentMapping(ptr, devicePtr);  // Update the address mapping
  }
  
  // End capture and instantiate the initial data distribution graph
  hipGraph_t graphForInitialDataDistribution;
  checkCudaErrors(hipStreamEndCapture(stream, &graphForInitialDataDistribution));

  // Execute the initial data distribution
  hipGraphExec_t graphExecForInitialDataDistribution;
  checkCudaErrors(hipGraphInstantiate(&graphExecForInitialDataDistribution, graphForInitialDataDistribution, nullptr, nullptr, 0));
  checkCudaErrors(hipGraphLaunch(graphExecForInitialDataDistribution, stream));
  checkCudaErrors(hipDeviceSynchronize());

  //----------------------------------------------------------------------
  // STEP 5: Build the optimized execution graph by processing nodes in topological order
  //----------------------------------------------------------------------
  
  LOG_TRACE_WITH_INFO("Record nodes to a new CUDA Graph");

  // Maps nodes to their dependencies in the CUDA graph
  std::map<int, std::vector<hipGraphNode_t>> nodeToDependentNodesMap;

  // Kahn's Algorithm for topological sort and graph construction
  while (!nodesToExecute.empty()) {
    // Get the next node to process
    auto u = nodesToExecute.front();
    nodesToExecute.pop();

    std::vector<hipGraphNode_t> newLeafNodes;
    auto nodeType = optimizedGraph.nodeIdToNodeTypeMap[u];
    
    // Process different node types
    if (nodeType == OptimizationOutput::NodeType::dataMovement) {
      //----------------------------------------------------------------------
      // STEP 5a: Handle data movement nodes (prefetch or offload)
      //----------------------------------------------------------------------
      
      optimizedCudaGraphCreator->beginCaptureOperation(nodeToDependentNodesMap[u]);
      auto &dataMovement = optimizedGraph.nodeIdToDataMovementMap[u];
      auto dataMovementAddress = memManager.getPointerByArrayId(dataMovement.arrayId);
      auto dataMovementSize = memManager.getSizeByArrayId(dataMovement.arrayId);
      
      if (dataMovement.direction == OptimizationOutput::DataMovement::Direction::hostToDevice) {
        // PREFETCH: Move data from storage to device
        void *devicePtr;
        checkCudaErrors(hipMallocAsync(&devicePtr, dataMovementSize, stream));
        checkCudaErrors(hipMemcpyAsync(
          devicePtr,
          managedDeviceArrayToHostArrayMap[dataMovementAddress],
          dataMovementSize,
          prefetchMemcpyKind,
          stream
        ));
        memManager.updateCurrentMapping(dataMovementAddress, devicePtr);
      } else {
        // OFFLOAD: Move data from device back to storage and free device memory
        void *devicePtr = memManager.getCurrentAddressMap().at(dataMovementAddress);
        checkCudaErrors(hipMemcpyAsync(
          managedDeviceArrayToHostArrayMap[dataMovementAddress],
          devicePtr,
          dataMovementSize,
          offloadMemcpyKind,
          stream
        ));
        checkCudaErrors(hipFreeAsync(devicePtr, stream));
        memManager.removeCurrentMapping(dataMovementAddress);
      }
      
      checkCudaErrors(hipPeekAtLastError());
      newLeafNodes = optimizedCudaGraphCreator->endCaptureOperation();
      checkCudaErrors(hipPeekAtLastError());
      
    } else if (nodeType == OptimizationOutput::NodeType::task) {
      //----------------------------------------------------------------------
      // STEP 5b: Handle computation task nodes
      //----------------------------------------------------------------------
      
      optimizedCudaGraphCreator->beginCaptureOperation(nodeToDependentNodesMap[u]);
      // Execute the task with current memory address mapping
      executeRandomTaskBase(
        optimizedGraph.nodeIdToTaskIdMap[u],
        stream
      );
      newLeafNodes = optimizedCudaGraphCreator->endCaptureOperation();
      
    } else if (nodeType == OptimizationOutput::NodeType::empty) {
      //----------------------------------------------------------------------
      // STEP 5c: Handle empty nodes (for dependencies)
      //----------------------------------------------------------------------
      
      newLeafNodes.push_back(
        optimizedCudaGraphCreator->addEmptyNode(nodeToDependentNodesMap[u])
      );
    } else {
      LOG_TRACE_WITH_INFO("Unsupported node type: %d", nodeType);
      exit(-1);
    }

    // Update dependencies and process nodes that have all dependencies satisfied
    for (auto &v : optimizedGraph.edges[u]) {
      inDegrees[v]--;

      // Add dependencies for the next nodes
      nodeToDependentNodesMap[v].insert(
        nodeToDependentNodesMap[v].end(),
        newLeafNodes.begin(),
        newLeafNodes.end()
      );

      // If all dependencies are satisfied, add to the queue
      if (inDegrees[v] == 0) {
        nodesToExecute.push(v);
      }
    }
  }

  // Export graph for debugging/visualization
  LOG_TRACE_WITH_INFO("Printing the new CUDA Graph to newGraph.dot");
  checkCudaErrors(hipGraphDebugDotPrint(graph, "newGraph.dot", 0));

  //----------------------------------------------------------------------
  // STEP 6: Execute the optimized graph
  //----------------------------------------------------------------------
  
  LOG_TRACE_WITH_INFO("Execute the new CUDA Graph");
  
  // Set up profiling if requested
  PeakMemoryUsageProfiler peakMemoryUsageProfiler;
  CudaEventClock cudaEventClock;
  
  // Instantiate the graph for execution
  hipGraphExec_t graphExec;
  checkCudaErrors(hipGraphInstantiate(&graphExec, graph, nullptr, nullptr, 0));

  // Upload the graph to the device for faster execution
  checkCudaErrors(hipGraphUpload(graphExec, stream));
  checkCudaErrors(hipStreamSynchronize(stream));

  // Start memory usage profiling if requested
  if (ConfigurationManager::getConfig().execution.measurePeakMemoryUsage) {
    peakMemoryUsageProfiler.start();
  }

  // Execute and time the graph
  cudaEventClock.start();
  checkCudaErrors(hipGraphLaunch(graphExec, stream));
  cudaEventClock.end();
  checkCudaErrors(hipDeviceSynchronize());

  // Report peak memory usage if requested
  if (ConfigurationManager::getConfig().execution.measurePeakMemoryUsage) {
    const auto peakMemoryUsage = peakMemoryUsageProfiler.end();
    LOG_TRACE_WITH_INFO(
      "Peak memory usage (MiB): %.2f",
      static_cast<float>(peakMemoryUsage) / 1024.0 / 1024.0
    );
  }

  //----------------------------------------------------------------------
  // STEP 7: Clean up resources and copy any remaining data back to storage
  //----------------------------------------------------------------------
  
  LOG_TRACE_WITH_INFO("Clean up");
  
  // Copy any remaining device data back to storage
  auto &currentAddressMap = memManager.getEditableCurrentAddressMap();
  for (auto &[oldAddr, newAddr] : currentAddressMap) {
    checkCudaErrors(hipMemcpy(
      managedDeviceArrayToHostArrayMap[oldAddr],
      newAddr,
      memManager.getSize(oldAddr),
      offloadMemcpyKind
    ));
    checkCudaErrors(hipFree(newAddr));
  }
  checkCudaErrors(hipDeviceSynchronize());

  // Clean up CUDA resources
  checkCudaErrors(hipGraphExecDestroy(graphExecForInitialDataDistribution));
  checkCudaErrors(hipGraphExecDestroy(graphExec));
  checkCudaErrors(hipGraphDestroy(graphForInitialDataDistribution));
  checkCudaErrors(hipGraphDestroy(graph));
  checkCudaErrors(hipStreamDestroy(stream));

  // Disable peer access if using NVLink
  if (ConfigurationManager::getConfig().execution.useNvlink) {
    disablePeerAccessForNvlink(mainDeviceId, storageDeviceId);
  }

  // Store the execution time
  runningTime = cudaEventClock.getTimeInSeconds();
}

std::vector<hipGraphNode_t> Executor::getNodesWithZeroOutDegree(hipGraph_t graph) {
  // Get all nodes in the graph
  size_t numNodes;
  checkCudaErrors(hipGraphGetNodes(graph, nullptr, &numNodes));
  auto nodes = std::make_unique<hipGraphNode_t[]>(numNodes);
  checkCudaErrors(hipGraphGetNodes(graph, nodes.get(), &numNodes));

  // Get all edges in the graph
  size_t numEdges;
  checkCudaErrors(hipGraphGetEdges(graph, nullptr, nullptr, &numEdges));
  auto from = std::make_unique<hipGraphNode_t[]>(numEdges);
  auto to = std::make_unique<hipGraphNode_t[]>(numEdges);
  checkCudaErrors(hipGraphGetEdges(graph, from.get(), to.get(), &numEdges));

  // Track which nodes have outgoing edges
  std::map<hipGraphNode_t, bool> hasOutGoingEdge;
  for (size_t i = 0; i < numEdges; i++) {
    hasOutGoingEdge[from[i]] = true;
  }

  // Find leaf nodes (nodes with no outgoing edges)
  std::vector<hipGraphNode_t> leafNodes;
  for (size_t i = 0; i < numNodes; i++) {
    if (!hasOutGoingEdge[nodes[i]]) {
      leafNodes.push_back(nodes[i]);
    }
  }

  return leafNodes;
}

hipGraphExec_t Executor::initializeDataDistribution(
  OptimizationOutput &optimizedGraph,
  int mainDeviceId, 
  int storageDeviceId,
  bool useNvlink,
  std::map<void *, void *> &managedDeviceArrayToHostArrayMap,
  hipStream_t stream,
  hipMemcpyKind prefetchMemcpyKind
) {
  LOG_TRACE_WITH_INFO("Initialize data distribution");
  
  // Move all managed data to storage (host or secondary GPU) using MemoryManager
  auto& memManager = MemoryManager::getInstance();
  // Configure storage parameters
  memManager.configureStorage(mainDeviceId, storageDeviceId, useNvlink);
  // Use the simplified API that uses the configured parameters
  memManager.offloadAllManagedMemoryToStorage(managedDeviceArrayToHostArrayMap);
  
  // Reset the current assignment map
  memManager.clearCurrentMappings();
  
  // Create a subgraph for initial data prefetching
  hipGraph_t graphForInitialDataDistribution;
  checkCudaErrors(hipStreamBeginCapture(stream, hipStreamCaptureModeGlobal));
  
  // Prefetch arrays that need to be on device initially
  auto& currentMap = memManager.getEditableCurrentAddressMap();
  memManager.prefetchAllDataToDeviceAsync(
    optimizedGraph.arraysInitiallyAllocatedOnDevice,
    managedDeviceArrayToHostArrayMap,
    currentMap,
    prefetchMemcpyKind,
    stream
  );
  
  // End capture and instantiate graph
  checkCudaErrors(hipStreamEndCapture(stream, &graphForInitialDataDistribution));
  
  // Create executable graph
  hipGraphExec_t graphExecForInitialDataDistribution;
  checkCudaErrors(hipGraphInstantiate(
    &graphExecForInitialDataDistribution, 
    graphForInitialDataDistribution, 
    nullptr, 
    nullptr, 
    0
  ));
  
  // Execute the initial data distribution
  checkCudaErrors(hipGraphLaunch(graphExecForInitialDataDistribution, stream));
  checkCudaErrors(hipDeviceSynchronize());
  
  // We'll clean up the graph itself, but return the executable for the caller to clean up
  checkCudaErrors(hipGraphDestroy(graphForInitialDataDistribution));
  
  return graphExecForInitialDataDistribution;
}

}  // namespace memopt