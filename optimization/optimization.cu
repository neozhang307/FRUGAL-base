#include <map>

#include "../utilities/logger.hpp"
#include "executor.hpp"
#include "optimization.hpp"
#include "optimizer.hpp"

namespace memopt {

OptimizationOutput profileAndOptimize(hipGraph_t originalGraph) {
  LOG_TRACE();
  return Optimizer::getInstance()->profileAndOptimize(originalGraph);
}

void executeOptimizedGraph(
  OptimizationOutput &optimizedGraph,
  ExecuteRandomTask executeRandomTask,
  float &runningTime,
  MemoryManager &memManager
) {
  LOG_TRACE();
  Executor::getInstance()->executeOptimizedGraph(
    optimizedGraph,
    executeRandomTask,
    runningTime,
    memManager
  );
}

void executeOptimizedGraph(
  OptimizationOutput &optimizedGraph,
  ExecuteRandomTask executeRandomTask,
  float &runningTime
) {
  LOG_TRACE();
  Executor::getInstance()->executeOptimizedGraph(
    optimizedGraph,
    executeRandomTask,
    runningTime
  );
}

void executeOptimizedGraph(
  OptimizationOutput &optimizedGraph,
  ExecuteRandomTask executeRandomTask,
  float &runningTime,
  std::map<void *, void *> &managedDeviceArrayToHostArrayMap
) {
  LOG_TRACE();
  Executor::getInstance()->executeOptimizedGraph(
    optimizedGraph,
    executeRandomTask,
    runningTime,
    managedDeviceArrayToHostArrayMap
  );
}

void executeOptimizedGraph(
  OptimizationOutput &optimizedGraph,
  ExecuteRandomTaskBase executeRandomTaskBase,
  float &runningTime,
  std::map<void *, void *> &managedDeviceArrayToHostArrayMap
) {
  LOG_TRACE();
  Executor::getInstance()->executeOptimizedGraph(
    optimizedGraph,
    executeRandomTaskBase,
    runningTime,
    managedDeviceArrayToHostArrayMap
  );
}

void executeOptimizedGraph(
  OptimizationOutput &optimizedGraph,
  ExecuteRandomTaskBase executeRandomTaskBase,
  float &runningTime,
  MemoryManager &memManager
) {
  LOG_TRACE();
  // Convert the simplified task execution to the full version with address mapping
  Executor::getInstance()->executeOptimizedGraph(
    optimizedGraph,
    [executeRandomTaskBase](int taskId, std::map<void *, void *> addressMapping, hipStream_t stream) {
      // Ignore the address mapping and call the base task execution function
      executeRandomTaskBase(taskId, stream);
    },
    runningTime,
    memManager
  );
}

void executeOptimizedGraphRepeatedly(
  OptimizationOutput &optimizedGraph,
  ExecuteRandomTask executeRandomTask,
  ShouldContinue shouldContinue,
  int &numIterations,
  float &runningTime,
  MemoryManager &memManager
) {
  LOG_TRACE();
  Executor::getInstance()->executeOptimizedGraphRepeatedly(
    optimizedGraph,
    executeRandomTask,
    shouldContinue,
    numIterations,
    runningTime,
    memManager
  );
}

void executeOptimizedGraphRepeatedly(
  OptimizationOutput &optimizedGraph,
  ExecuteRandomTask executeRandomTask,
  ShouldContinue shouldContinue,
  int &numIterations,
  float &runningTime
) {
  LOG_TRACE();
  Executor::getInstance()->executeOptimizedGraphRepeatedly(
    optimizedGraph,
    executeRandomTask,
    shouldContinue,
    numIterations,
    runningTime
  );
}

void executeOptimizedGraphRepeatedly(
  OptimizationOutput &optimizedGraph,
  ExecuteRandomTask executeRandomTask,
  ShouldContinue shouldContinue,
  int &numIterations,
  float &runningTime,
  std::map<void *, void *> &managedDeviceArrayToHostArrayMap
) {
  LOG_TRACE();
  Executor::getInstance()->executeOptimizedGraphRepeatedly(
    optimizedGraph,
    executeRandomTask,
    shouldContinue,
    numIterations,
    runningTime,
    managedDeviceArrayToHostArrayMap
  );
}

}  // namespace memopt
