#include "memoryManager.hpp"

namespace memopt {

// Memory management methods specific to the optimized executor (executor_v2.cu)

// Configuration
void MemoryManager::configureStorage(int mainDeviceId, int storageDeviceId, bool useNvlink) {
  storageConfig.mainDeviceId = mainDeviceId;
  storageConfig.storageDeviceId = storageDeviceId;
  storageConfig.useNvlink = useNvlink;
  
  // Set appropriate memcpy kinds based on the configuration
  if (useNvlink) {
    storageConfig.prefetchMemcpyKind = hipMemcpyDeviceToDevice;
    storageConfig.offloadMemcpyKind = hipMemcpyDeviceToDevice;
  } else {
    storageConfig.prefetchMemcpyKind = hipMemcpyHostToDevice;
    storageConfig.offloadMemcpyKind = hipMemcpyDeviceToHost;
  }
}

void MemoryManager::cleanStorage() {
  if (storageConfig.useNvlink) {
    disablePeerAccessForNvlink(storageConfig.mainDeviceId, storageConfig.storageDeviceId);
  }
}

// Mapping management
void MemoryManager::updateCurrentMapping(void* originalAddr, void* currentAddr) {
  managedMemoryAddressToAssignedMap[originalAddr] = currentAddr;
}

void MemoryManager::removeCurrentMapping(void* originalAddr) {
  managedMemoryAddressToAssignedMap.erase(originalAddr);
}

void MemoryManager::clearCurrentMappings() {
  managedMemoryAddressToAssignedMap.clear();
}

// Memory prefetching methods
void MemoryManager::prefetchAllDataToDevice(
    const std::vector<ArrayId>& arrayIds,
    const std::map<void*, void*>& storageMap,
    std::map<void*, void*>& currentMap,
    hipMemcpyKind memcpyKind,
    hipStream_t stream) {
  for (auto arrayId : arrayIds) {
    void* originalPtr = managedMemoryAddresses[arrayId];
    void* storagePtr = storageMap.at(originalPtr);
    size_t size = getSize(originalPtr);
    
    // Allocate on device and copy data from storage
    void* devicePtr;
    checkCudaErrors(hipMallocAsync(&devicePtr, size, stream));
    checkCudaErrors(hipMemcpyAsync(
      devicePtr, 
      storagePtr, 
      size, 
      memcpyKind, 
      stream
    ));
    
    // Update the current mapping
    currentMap[originalPtr] = devicePtr;
  }
}

void MemoryManager::prefetchAllDataToDevice(
    const std::vector<ArrayId>& arrayIds,
    hipStream_t stream) {
  for (auto arrayId : arrayIds) {
    void* originalPtr = managedMemoryAddresses[arrayId];
    void* storagePtr = managedDeviceArrayToHostArrayMap.at(originalPtr);
    size_t size = getSize(originalPtr);
    
    // Allocate on device and copy data from storage
    void* devicePtr;
    checkCudaErrors(hipMallocAsync(&devicePtr, size, stream));
    checkCudaErrors(hipMemcpyAsync(
      devicePtr, 
      storagePtr, 
      size, 
      storageConfig.prefetchMemcpyKind, 
      stream
    ));
    
    // Update the current mapping
    managedMemoryAddressToAssignedMap[originalPtr] = devicePtr;
  }
}

void MemoryManager::prefetchToDevice(const ArrayId arrayId, hipStream_t stream) {
  void *devicePtr;
  auto dataMovementSize = this->getSizeByArrayId(arrayId);
  auto dataMovementAddress = this->getPointerByArrayId(arrayId);

  checkCudaErrors(hipMallocAsync(&devicePtr, dataMovementSize, stream));
  checkCudaErrors(hipMemcpyAsync(
    devicePtr,
    this->getDeviceToHostArrayMap().at(dataMovementAddress),
    dataMovementSize,
    storageConfig.prefetchMemcpyKind,
    stream
  ));
  this->updateCurrentMapping(dataMovementAddress, devicePtr);
}

void MemoryManager::offloadFromDevice(const ArrayId arrayId, hipStream_t stream) {
  auto dataMovementSize = this->getSizeByArrayId(arrayId);
  auto dataMovementAddress = this->getPointerByArrayId(arrayId);
  void *devicePtr = this->getCurrentAddressMap().at(dataMovementAddress);

  checkCudaErrors(hipMemcpyAsync(
    this->getDeviceToHostArrayMap().at(dataMovementAddress),
    devicePtr,
    dataMovementSize,
    storageConfig.offloadMemcpyKind,
    stream
  ));
  checkCudaErrors(hipFreeAsync(devicePtr, stream));
  this->removeCurrentMapping(dataMovementAddress);
}

// Memory storage operations
void MemoryManager::moveAllManagedMemoryToStorage() {
  // Ensure the storage map starts empty
  managedDeviceArrayToHostArrayMap.clear();
  
  // Move each managed memory address to storage
  for (auto ptr : managedMemoryAddresses) {
    offloadToStorage(ptr, storageConfig.storageDeviceId, storageConfig.useNvlink, managedDeviceArrayToHostArrayMap);
  }
  
  // Switch back to main GPU
  checkCudaErrors(hipSetDevice(storageConfig.mainDeviceId));
  checkCudaErrors(hipDeviceSynchronize());
}

void MemoryManager::moveRemainedManagedMemoryToStorage() {
  auto currentAddressMap = this->getEditableCurrentAddressMap();
  for (auto &[oldAddr, newAddr] : currentAddressMap) {
    checkCudaErrors(hipMemcpy(
      this->getDeviceToHostArrayMap().at(oldAddr),
      newAddr,
      this->getSize(oldAddr),
      storageConfig.offloadMemcpyKind
    ));
    checkCudaErrors(hipFree(newAddr));
  }
}

void MemoryManager::moveRemainedManagedMemoryToStorage(hipStream_t stream) {
  auto currentAddressMap = this->getEditableCurrentAddressMap();
  for (auto &[oldAddr, newAddr] : currentAddressMap) {
    checkCudaErrors(hipMemcpyAsync(
      this->getDeviceToHostArrayMap().at(oldAddr),
      newAddr,
      this->getSize(oldAddr),
      storageConfig.offloadMemcpyKind,
      stream
    ));
    checkCudaErrors(hipFreeAsync(newAddr, stream));
    this->removeCurrentMapping(oldAddr);
  }
}

} // namespace memopt