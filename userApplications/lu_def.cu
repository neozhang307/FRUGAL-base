#include "hip/hip_runtime.h"
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hipsolver.h>
// #include <fmt/core.h>

#include <algorithm>
#include <cassert>
#include <cmath>
#include <cstdlib>
#include <initializer_list>
#include <iomanip>
#include <iostream>
#include <limits>
#include <map>
#include <memory>
#include <set>
#include <tuple>
#include <vector>

#include "../include/argh.h"
// #include "../utilities/cudaUtilities.hpp"

size_t N = 15 * 1;
size_t B = N / 5;
size_t T = N / B;


template <typename T>
void __check(T result, char const *const func, const char *const file, int const line)
{
    if (result)
    {
        fprintf(stderr, "CUDA error at %s:%d code=%d \"%s\" \n", file, line, static_cast<unsigned int>(result), func);
        exit(EXIT_FAILURE);
    }
}

#define checkCudaErrors(val) __check((val), #val, __FILE__, __LINE__)

__global__ void warmUp()
{
    unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
    float ia, ib;
    ia = ib = 0.0f;
    ib += ia + static_cast<float>(tid);
}

void warmUpCudaDevice()
{
    warmUp<<<32, 32>>>();
    hipDeviceSynchronize();
}

void initializeCudaDevice(bool displayDeviceInfo)
{
    checkCudaErrors(hipSetDevice(0));

    if (displayDeviceInfo)
    {
        hipDeviceProp_t deviceProp;
        checkCudaErrors(hipGetDeviceProperties(&deviceProp, 0));
        printf("GPU Device %d: %s\n", 0, deviceProp.name);
        printf("Compute Capability: %d.%d\n\n", deviceProp.major, deviceProp.minor);
    }

    warmUpCudaDevice();
}

class CudaEventClock
{
public:
    CudaEventClock();
    ~CudaEventClock();
    void start(hipStream_t stream = 0);
    void end(hipStream_t stream = 0);
    float getTimeInSeconds();

private:
    hipEvent_t startEvent, endEvent;
};

CudaEventClock::CudaEventClock()
{
    checkCudaErrors(hipEventCreate(&this->startEvent));
    checkCudaErrors(hipEventCreate(&this->endEvent));
}

CudaEventClock::~CudaEventClock()
{
    checkCudaErrors(hipEventDestroy(this->startEvent));
    checkCudaErrors(hipEventDestroy(this->endEvent));
}

void CudaEventClock::start(hipStream_t stream)
{
    checkCudaErrors(hipEventRecord(this->startEvent, stream));
}

void CudaEventClock::end(hipStream_t stream)
{
    checkCudaErrors(hipEventRecord(this->endEvent, stream));
}

float CudaEventClock::getTimeInSeconds()
{
    float time;
    checkCudaErrors(hipEventElapsedTime(&time, this->startEvent, this->endEvent));
    return time * 1e-3f;
}


// Credit to: https://math.stackexchange.com/questions/357980/how-to-generate-random-symmetric-positive-definite-matrices-using-matlab
void generateRandomSymmetricPositiveDefiniteMatrix(double *h_A, const size_t n)
{
    // srand(time(NULL));
    srand(420);

    double *h_A_temp = (double *)malloc(n * n * sizeof(double));

    for (int i = 0; i < n; i++)
        for (int j = 0; j < n; j++)
            h_A_temp[i * n + j] = (float)rand() / (float)RAND_MAX;

    for (int i = 0; i < n; i++)
        for (int j = 0; j < n; j++)
            h_A[i * n + j] = 0.5 * (h_A_temp[i * n + j] + h_A_temp[j * n + i]);

    for (int i = 0; i < n; i++)
        h_A[i * n + i] = h_A[i * n + i] + n;
}

void printSquareMatrix(double *h_A, const size_t n)
{
    for (int i = 0; i < n; i++)
    {
        for (int j = 0; j < n; j++)
        {
            if (j != 0)
                std::cout << " ";
            std::cout << std::setw(6) << std::setprecision(3) << h_A[i * n + j];
        }
        std::cout << std::endl;
    }
}

// Set upper triangle entries (excluding diagonal entries) in column-major order to zero.
// Then, transpose to row-major order.
void cleanCusolverLUDecompositionResult(double *L, double *U, const int n)
{
    for (int i = 0; i < n; i++)
    {
        for (int j = i; j < n; j++)
        {
            std::swap(L[i + j * n], L[i * n + j]);
            U[i * n + j] = L[i * n + j];
            L[i * n + j] = 0;
        }
        L[i * n + i] = 1;
    }
}

bool verifyLUDecomposition(double *A, double *L, double *U, const int n)
{
    auto newA = std::make_unique<double[]>(n * n);
    memset(newA.get(), 0, n * n * sizeof(double));
    for (int i = 0; i < n; i++)
    {
        for (int j = 0; j < n; j++)
        {
            for (int k = 0; k < n; k++)
            {
                newA[i * n + j] += L[i * n + k] * U[k * n + j];
            }
        }
    }

    double error = 0;
    for (int i = 0; i < n; i++)
    {
        for (int j = 0; j < n; j++)
        {
            error += fabs(A[i * n + j] - newA[i * n + j]);
        }
    }

    printf("A:\n");
    printSquareMatrix(A, n);

    printf("\nnewA:\n");
    printSquareMatrix(newA.get(), n);

    printf("\nL:\n");
    printSquareMatrix(L, n);
    printf("\n");

    printf("\nU:\n");
    printSquareMatrix(U, n);
    printf("\n");


    printf("error = %.6f}\n", error);

    return error <= 1e-6;
}

void trivialLU(bool verify)
{
    // Initialize libaries
    hipsolverHandle_t cusolverDnHandle;
    checkCudaErrors(hipsolverDnCreate(&cusolverDnHandle));

    hipsolverDnParams_t cusolverDnParams;
    checkCudaErrors(hipsolverDnCreateParams(&cusolverDnParams));

    // Initialize data
    double *h_A = (double *)malloc(N * N * sizeof(double));
    generateRandomSymmetricPositiveDefiniteMatrix(h_A, N);

    double *d_A;
    checkCudaErrors(hipMalloc(&d_A, N * N * sizeof(double)));
    checkCudaErrors(hipMemcpy(d_A, h_A, N * N * sizeof(double), hipMemcpyHostToDevice));

    size_t workspaceInBytesOnDevice, workspaceInBytesOnHost;

    checkCudaErrors(hipsolverDnXgetrf_bufferSize(
        cusolverDnHandle,
        cusolverDnParams,
        N,
        N,
        HIP_R_64F,
        d_A,
        N,
        HIP_R_64F,
        &workspaceInBytesOnDevice,
        &workspaceInBytesOnHost));

    void *h_workspace = malloc(workspaceInBytesOnHost);

    void *d_workspace;
    checkCudaErrors(hipMalloc(&d_workspace, workspaceInBytesOnDevice));

    int *d_info;
    checkCudaErrors(hipMalloc(&d_info, sizeof(int)));
    CudaEventClock clock;

    // Calculate
    clock.start();
    checkCudaErrors(hipsolverDnXgetrf(
        cusolverDnHandle,
        cusolverDnParams,
        N,
        N,
        HIP_R_64F,
        d_A,
        N,
        NULL, // no pivoting
        HIP_R_64F,
        d_workspace,
        workspaceInBytesOnDevice,
        h_workspace,
        workspaceInBytesOnHost,
        d_info));
    clock.end();

    // Check
    int h_info = 0;
    checkCudaErrors(hipMemcpy(&h_info, d_info, sizeof(int), hipMemcpyDeviceToHost));
    if (h_info != 0)
    {
        std::cout << "Unsuccessful potrf execution\n\n"
                  << "d_info = " << h_info << "\n\n";
    }

    // Verify
    if (verify) {
        double *h_L = (double *)malloc(N * N * sizeof(double));
        double *h_U = (double *)malloc(N * N * sizeof(double));
        checkCudaErrors(hipMemcpy(h_L, d_A, N * N * sizeof(double), hipMemcpyDeviceToHost));
        cleanCusolverLUDecompositionResult(h_L, h_U, N);
        printf("Result passes verification: %d\n", verifyLUDecomposition(h_A, h_L, h_U, N));

        // Clean
        free(h_L);
        free(h_U);
    }
    
    printf("Total time used (s): %4.4f\n", clock.getTimeInSeconds());

    free(h_A);
    free(h_workspace);
    checkCudaErrors(hipsolverDnDestroy(cusolverDnHandle));
    checkCudaErrors(hipFree(d_A));
    checkCudaErrors(hipFree(d_workspace));
    checkCudaErrors(hipFree(d_info));
}

typedef std::pair<int, int> MatrixTile;

class TiledLUGraphCreator
{
public:
    TiledLUGraphCreator(hipStream_t stream, hipGraph_t graph) : stream(stream), graph(graph)
    {
        this->lastModifiedTile = std::make_pair(-1, -1);
    }
    void beginCaptureOperation(MatrixTile tileToWrite, std::initializer_list<MatrixTile> tilesToRead)
    {
        auto tiles = std::vector<MatrixTile>(tilesToRead);
        tiles.push_back(tileToWrite);
        auto dependencies = this->getDependencies(tiles);

        this->lastModifiedTile = tileToWrite;
        this->lastDependencies = dependencies;

        checkCudaErrors(hipStreamBeginCaptureToGraph(this->stream, this->graph, dependencies.data(), nullptr, dependencies.size(), hipStreamCaptureModeGlobal));
    }

    void endCaptureOperation()
    {
        assert(this->lastModifiedTile.first != -1 && this->lastModifiedTile.second != -1);
        checkCudaErrors(hipStreamEndCapture(this->stream, &this->graph));
        this->tileLastModifiedByMap[this->lastModifiedTile] = this->getTailOfLastCapturedNodeChain();
        this->lastModifiedTile = std::make_pair(-1, -1);
    };

private:
    std::map<MatrixTile, hipGraphNode_t> tileLastModifiedByMap;
    std::map<hipGraphNode_t, bool> visited;
    hipStream_t stream;
    hipGraph_t graph;
    MatrixTile lastModifiedTile;
    std::vector<hipGraphNode_t> lastDependencies;

    std::vector<hipGraphNode_t> getDependencies(std::vector<MatrixTile> tiles)
    {
        std::vector<hipGraphNode_t> dependencies;
        for (auto tile : tiles)
        {
            auto it = this->tileLastModifiedByMap.find(tile);
            if (it != this->tileLastModifiedByMap.end())
            {
                dependencies.push_back(it->second);
            }
        }

        auto dedupedEnd = std::unique(dependencies.begin(), dependencies.end());
        dependencies.resize(std::distance(dependencies.begin(), dedupedEnd));
        return dependencies;
    }

    hipGraphNode_t getTailOfLastCapturedNodeChain()
    {
        if (lastDependencies.size() == 0)
        {
            size_t numEdges;
            checkCudaErrors(hipGraphGetEdges(this->graph, nullptr, nullptr, &numEdges));
            auto from = std::make_unique<hipGraphNode_t[]>(numEdges);
            auto to = std::make_unique<hipGraphNode_t[]>(numEdges);
            checkCudaErrors(hipGraphGetEdges(this->graph, from.get(), to.get(), &numEdges));

            std::map<hipGraphNode_t, bool> hasOutGoingEdge;
            std::set<hipGraphNode_t> noOutGoingEdgeNodes;
            for (int i = 0; i < numEdges; i++)
            {
                hasOutGoingEdge[from[i]] = true;
                noOutGoingEdgeNodes.erase(from[i]);
                if (!hasOutGoingEdge[to[i]])
                    noOutGoingEdgeNodes.insert(to[i]);
            }

            assert(noOutGoingEdgeNodes.size() == 1);

            return *noOutGoingEdgeNodes.begin();
        }
        else
        {
            auto nodeBeforeChain = lastDependencies[0];
            size_t numDependentNodes;
            checkCudaErrors(hipGraphNodeGetDependentNodes(nodeBeforeChain, nullptr, &numDependentNodes));

            assert(numDependentNodes > 0);

            auto dependentNodes = std::make_unique<hipGraphNode_t[]>(numDependentNodes);
            checkCudaErrors(hipGraphNodeGetDependentNodes(nodeBeforeChain, dependentNodes.get(), &numDependentNodes));

            hipGraphNode_t chainBeginningNode;
            for (int i = 0; i < numDependentNodes; i++)
            {
                if (!visited[dependentNodes[i]])
                {
                    chainBeginningNode = dependentNodes[i];
                    break;
                }
            }

            auto u = chainBeginningNode;
            while (true)
            {
                visited[u] = true;
                checkCudaErrors(hipGraphNodeGetDependentNodes(u, nullptr, &numDependentNodes));
                if (numDependentNodes == 0)
                    break;

                assert(numDependentNodes == 1);

                hipGraphNode_t v;
                checkCudaErrors(hipGraphNodeGetDependentNodes(u, &v, &numDependentNodes));
                u = v;
            }

            return u;
        }
    }
};

void tiledLU(bool verify)
{
    // Initialize data
    auto originalMatrix = std::make_unique<double[]>(N * N); // Column-major
    generateRandomSymmetricPositiveDefiniteMatrix(originalMatrix.get(), N);

    // Copy to device
    double *d_matrix;
    checkCudaErrors(hipMallocManaged(&d_matrix, N * N * sizeof(double)));
    checkCudaErrors(hipMemcpy(d_matrix, originalMatrix.get(), N * N * sizeof(double), hipMemcpyHostToDevice));

    auto getMatrixBlock = [&](int i, int j)
    {
        return d_matrix + i * B + j * B * N;
    };

    // Initialize libraries
    hipsolverHandle_t cusolverDnHandle;
    hipsolverDnParams_t cusolverDnParams;
    hipblasHandle_t cublasHandle;
    checkCudaErrors(hipsolverDnCreate(&cusolverDnHandle));
    checkCudaErrors(hipsolverDnCreateParams(&cusolverDnParams));
    checkCudaErrors(hipblasCreate(&cublasHandle));

    // Prepare constants
    // double *one, *minusOne;
    // checkCudaErrors(hipMallocManaged(&one, sizeof(double)));
    // checkCudaErrors(hipMallocManaged(&minusOne, sizeof(double)));
    double one = 1.0;
    double minusOne = -1.0;

    // Prepare buffer for potrf
    size_t workspaceInBytesOnDevice, workspaceInBytesOnHost;
        
    checkCudaErrors(hipsolverDnXgetrf_bufferSize(
        cusolverDnHandle,
        cusolverDnParams,
        B,
        B,
        HIP_R_64F,
        d_matrix,
        N,
        HIP_R_64F,
        &workspaceInBytesOnDevice,
        &workspaceInBytesOnHost));

    void *h_workspace, *d_workspace;
    int *d_info;
    checkCudaErrors(hipMallocManaged(&h_workspace, workspaceInBytesOnHost));
    checkCudaErrors(hipMallocManaged(&d_workspace, workspaceInBytesOnDevice));
    checkCudaErrors(hipMallocManaged(&d_info, sizeof(int)));

    hipGraph_t graph;
    checkCudaErrors(hipGraphCreate(&graph, 0));

    hipStream_t s;
    checkCudaErrors(hipStreamCreate(&s));

    checkCudaErrors(hipsolverSetStream(cusolverDnHandle, s));
    checkCudaErrors(hipblasSetStream(cublasHandle, s));
    // checkCudaErrors(hipblasSetWorkspace(cublasHandle, d_workspace, workspaceInBytesOnDevice));

    auto tiledLUGraphCreator = std::make_unique<TiledLUGraphCreator>(s, graph);

    for (int k = 0; k < T; k++)
    {
        // A[k][k] = GETRF(A[k][k])
        // L[k][k]*U[k][k] = A[k][k]
        tiledLUGraphCreator->beginCaptureOperation(
            std::make_pair(k, k),
            {std::make_pair(k, k)});
        checkCudaErrors(hipsolverDnXgetrf(
            cusolverDnHandle,
            cusolverDnParams,
            B,
            B,
            HIP_R_64F,
            getMatrixBlock(k, k),
            N,
            NULL, // no pivoting
            HIP_R_64F,
            d_workspace,
            workspaceInBytesOnDevice,
            h_workspace,
            workspaceInBytesOnHost,
            d_info));
        tiledLUGraphCreator->endCaptureOperation();

        for (int i = k + 1; i < T; i++)
        {
            // L[i][k] = TRSM(A[i][k], A[k][k]) // the U part of A[k][k]
            tiledLUGraphCreator->beginCaptureOperation(
                std::make_pair(k, i),
                {std::make_pair(k, k), std::make_pair(k, i)});
            checkCudaErrors(hipblasDtrsm(
                cublasHandle,
                HIPBLAS_SIDE_LEFT, // used to be right for cholesky
                HIPBLAS_FILL_MODE_LOWER,
                HIPBLAS_OP_N,// HIPBLAS_OP_T for cholesky
                HIPBLAS_DIAG_UNIT, // HIPBLAS_DIAG_NON_UNIT for cholesky
                B, B,
                &one,
                getMatrixBlock(k, k), N, // k + k * N;
                getMatrixBlock(k, i), N)); // k + (i + B) * N;
            tiledLUGraphCreator->endCaptureOperation();

        }

        for (int i = k + 1; i < T; i++)
        {
            // U[k][i] = TRSM(A[k][k], A[k][i]) // the L part of A[k][k]
            tiledLUGraphCreator->beginCaptureOperation(
                std::make_pair(i, k),
                {std::make_pair(k, k), std::make_pair(i, k)});
            checkCudaErrors(hipblasDtrsm(
                cublasHandle,
                HIPBLAS_SIDE_RIGHT, 
                HIPBLAS_FILL_MODE_UPPER,
                HIPBLAS_OP_N, 
                HIPBLAS_DIAG_NON_UNIT, 
                B, B,
                &one,
                getMatrixBlock(k, k), N, // k + k * N;
                getMatrixBlock(i, k), N)); // (i + B) + k * N;
            tiledLUGraphCreator->endCaptureOperation();

            for (int j = k + 1; j < T; j++)
            {
                // A[j][i] = GEMM(A[j][k], A[i][k])
                // A[j][i] = A[j][i] - L[j][k] * L[i][k]^T
                tiledLUGraphCreator->beginCaptureOperation(
                    std::make_pair(i, j),
                    {std::make_pair(i, k), std::make_pair(k, j), std::make_pair(i, j)});
                checkCudaErrors(hipblasGemmEx(
                    cublasHandle,
                    HIPBLAS_OP_N,
                    HIPBLAS_OP_N, // HIPBLAS_OP_T
                    B, B, B,
                    &minusOne,
                    getMatrixBlock(i, k), HIP_R_64F, N, // i + k * N
                    getMatrixBlock(k, j), HIP_R_64F, N, // j + i * N
                    &one,
                    getMatrixBlock(i, j), HIP_R_64F, N, // k + i * N
                    HIPBLAS_COMPUTE_64F,
                    HIPBLAS_GEMM_DEFAULT));
                tiledLUGraphCreator->endCaptureOperation();
            }
        }
    }

    CudaEventClock clock;
    
    checkCudaErrors(hipGraphDebugDotPrint(graph, "./graph.dot", 0));

    hipGraphExec_t graphExec;
    checkCudaErrors(hipGraphInstantiate(&graphExec, graph, nullptr, nullptr, 0));
    
    clock.start(s);
    checkCudaErrors(hipGraphLaunch(graphExec, s));
    checkCudaErrors(hipStreamSynchronize(s));
    clock.end(s);
    checkCudaErrors(hipDeviceSynchronize());

    if (verify) {
        double *h_U = (double *)malloc(N * N * sizeof(double));
        memset(h_U, 0, N * N * sizeof(double));
        cleanCusolverLUDecompositionResult(d_matrix, h_U, N);
        printf("Result passes verification: %d\n", verifyLUDecomposition(originalMatrix.get(), d_matrix, h_U, N));

        free(h_U);
    }
    printf("Total time used (s): %4.4f\n", clock.getTimeInSeconds());

    free(h_workspace);
    hipFree(d_matrix);
    hipFree(d_workspace);
}

void LU(bool tiled, bool verify)
{
    if (tiled)
    {
        tiledLU(verify);
    }
    else
    {
        trivialLU(verify);
    }
}

int main(int argc, char **argv)
{
    argh::parser cmdl({"n", "N", "t", "T"});
    cmdl.parse(argc, argv);

    if (!(cmdl({"N", "n"}, N) >> N)) {
        std::cerr << "Must provide a valid N value! Got '" << cmdl({"N", "n"}).str() << "'" << std::endl;
        return 0;
    }
    if (!(cmdl({"t", "T"}, T) >> T)) {
        std::cerr << "Must provide a valid T value! Got '" << cmdl({"T", "t"}).str() << "'" << std::endl;
        return 0;
    }
    if (N % T > 0) {
        std::cerr << "N must be divisible by T! Got 'N=" << N << " & T=" << T << "'" << std::endl;
        return 0;
    }
    B = N / T;
    
    if (cmdl["tiled"])
        std::cout << "TILED ";
    else
        std::cout << "Single-kernel ";
    std::cout << "with 'N=" << N << " & T=" << T << " & B=" << B << "'" << std::endl;

    LU(cmdl["tiled"], cmdl["verify"]);

    return 0;
}