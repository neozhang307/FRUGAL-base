#include "hip/hip_runtime.h"
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hiprand.h>
#include <hipsolver.h>
#include <fmt/core.h>

#include <algorithm>
#include <cassert>
#include <cmath>
#include <cstdlib>
#include <initializer_list>
#include <iostream>
#include <limits>
#include <map>
#include <memory>
#include <set>
#include <tuple>
#include <vector>

#include "../include/argh.h"
#include "memopt.hpp"
// #include "../optimization/taskManager_v2.hpp"

using namespace memopt;

const std::string INPUT_MATRIX_FILE_PATH = "tiledCholeskyInputMatrix.in";

// Implementation Note: This file has been updated to use TaskManager_v2 for all operations.
// The algorithm records a CUDA graph by executing tasks through TaskManager_v2, which can then
// be executed directly or with memory optimization.

size_t N; // total matrix size (N×N)
size_t B; // batch size in 1d (B×B per tile)
size_t T; // tile amount in 1d (T×T tiles)

__global__ void makeMatrixSymmetric(double *d_matrix, size_t n) {
  size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  size_t x = idx / n;
  size_t y = idx % n;

  if (x >= y || x >= n || y >= n) {
    return;
  }

  double average = 0.5 * (d_matrix[x * n + y] + d_matrix[y * n + x]);
  d_matrix[x * n + y] = average;
  d_matrix[y * n + x] = average;
}

__global__ void addIdenticalMatrix(double *d_matrix, size_t n) {
  size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= n) {
    return;
  }
  d_matrix[idx * n + idx] += n;
}

// Credit to: https://math.stackexchange.com/questions/357980/how-to-generate-random-symmetric-positive-definite-matrices-using-matlab
void generateRandomSymmetricPositiveDefiniteMatrix(double *h_A, const size_t n) {
  double *d_A;
  checkCudaErrors(hipMalloc(&d_A, n * n * sizeof(double)));

  // Generate random matrix d_A
  hiprandGenerator_t prng;
  hiprandCreateGenerator(&prng, HIPRAND_RNG_PSEUDO_XORWOW);
  hiprandSetPseudoRandomGeneratorSeed(prng, (unsigned long long)clock());
  hiprandGenerateUniformDouble(prng, d_A, n * n);

  // d_A = (d_A + d_A^T) / 2
  size_t numThreads = 1024;
  size_t numBlocks = (N * N + numThreads - 1) / numThreads;
  makeMatrixSymmetric<<<numBlocks, numThreads>>>(d_A, N);

  // d_A = d_A + n * I
  numThreads = 1024;
  numBlocks = (N + numThreads - 1) / numThreads;
  addIdenticalMatrix<<<numBlocks, numThreads>>>(d_A, N);

  checkCudaErrors(hipDeviceSynchronize());

  checkCudaErrors(hipMemcpy(h_A, d_A, n * n * sizeof(double), hipMemcpyDefault));

  checkCudaErrors(hipDeviceSynchronize());
  checkCudaErrors(hipFree(d_A));
}

// Only verify the last row of L * L^T = A
bool verifyCholeskyDecompositionPartially(double *A, std::vector<double *> &d_tiles, const size_t n, const size_t b) {
  const size_t t = n / b;

  // Use pinned memory for h_tiles for faster data transfer
  std::vector<double*> h_tiles(t * t, nullptr);
  for (int i = 0; i < t * t; i++) {
    checkCudaErrors(hipHostMalloc(&h_tiles[i], b * b * sizeof(double)));
    checkCudaErrors(hipMemcpy(h_tiles[i], MemoryManager::getInstance().getAddress(d_tiles[i]), B * B * sizeof(double), hipMemcpyDefault));
    checkCudaErrors(hipDeviceSynchronize());
  }

  auto getAEntry = [&](size_t row, size_t col) {
    return A[row + col * n];
  };

  auto getLEntry = [&](size_t row, size_t col) {
    if (row < col) {
      return static_cast<double>(0);
    }
    const size_t i = row / b;
    const size_t k = row - (i * b);
    const size_t j = col / b;
    const size_t l = col - (j * b);

    return h_tiles[i + j * t][k + l * b];
  };

  // Only check the last row;
  const size_t rowIndex = n - 1;

  const size_t rowLength = min((size_t)1024, n);

  // Use pinned memory for firstRow as well
  double* firstRow = nullptr;
  checkCudaErrors(hipHostMalloc(&firstRow, rowLength * sizeof(double)));
  memset(firstRow, 0, rowLength * sizeof(double));
  
  for (int j = 0; j < rowLength; j++) {
    for (int k = 0; k < n; k++) {
      firstRow[j] += getLEntry(rowIndex, k) * getLEntry(j, k);
    }
  }

  double error = 0;
  for (int j = 0; j < rowLength; j++) {
    error += fabs(getAEntry(rowIndex, j) - firstRow[j]);
  }

  fmt::print("error = {:.6f}\n", error);
  
  // Free pinned memory for h_tiles and firstRow
  for (int i = 0; i < t * t; i++) {
    checkCudaErrors(hipHostFree(h_tiles[i]));
  }
  checkCudaErrors(hipHostFree(firstRow));

  return error <= 1e-6;
}

// Define a matrix tile as a pair of coordinates (i,j)
typedef std::pair<int, int> MatrixTile;

void initializeHostData(double *h_originalMatrix) {
  generateRandomSymmetricPositiveDefiniteMatrix(h_originalMatrix, N);
}

void initializeDeviceData(double *h_originalMatrix, std::vector<double *> &d_tiles) {
  for (int i = 0; i < T; i++) {
    for (int j = 0; j < T; j++) {
      for (int k = 0; k < B; k++) {
        checkCudaErrors(hipMemcpy(
          MemoryManager::getAddress(d_tiles[i + j * T]) + B * k,
          h_originalMatrix + N * (j * B + k) + B * i,
          B * sizeof(double),
          hipMemcpyDefault
        ));
      }
    }
  }
  checkCudaErrors(hipDeviceSynchronize());
}

class TiledCholeskyGraphCreator {
 public:
  TiledCholeskyGraphCreator(hipStream_t stream, hipGraph_t graph) : stream(stream), graph(graph) {
    this->lastModifiedTile = {-1, -1};
  }

  void beginCaptureOperation(MatrixTile tileToWrite, std::initializer_list<MatrixTile> tilesToRead) {
    auto tiles = std::vector<MatrixTile>(tilesToRead);
    tiles.push_back(tileToWrite);
    auto dependencies = this->getDependencies(tiles);

    this->lastModifiedTile = tileToWrite;
    this->lastDependencies = dependencies;

    checkCudaErrors(hipStreamBeginCaptureToGraph(this->stream, this->graph, dependencies.data(), nullptr, dependencies.size(), hipStreamCaptureModeGlobal));
  }

  void endCaptureOperation() {
    assert(this->lastModifiedTile.first != -1 && this->lastModifiedTile.second != -1);
    checkCudaErrors(hipStreamEndCapture(this->stream, &this->graph));
    this->tileLastModifiedByMap[this->lastModifiedTile] = this->getTailOfLastCapturedNodeChain();
    this->lastModifiedTile = {-1, -1};
  };

 private:
  std::map<MatrixTile, hipGraphNode_t> tileLastModifiedByMap;
  std::map<hipGraphNode_t, bool> visited;
  hipStream_t stream;
  hipGraph_t graph;
  MatrixTile lastModifiedTile;
  std::vector<hipGraphNode_t> lastDependencies;

  std::vector<hipGraphNode_t> getDependencies(std::vector<MatrixTile> tiles) {
    std::vector<hipGraphNode_t> dependencies;
    for (auto tile : tiles) {
      auto it = this->tileLastModifiedByMap.find(tile);
      if (it != this->tileLastModifiedByMap.end()) {
        dependencies.push_back(it->second);
      }
    }

    auto dedupedEnd = std::unique(dependencies.begin(), dependencies.end());
    dependencies.resize(std::distance(dependencies.begin(), dedupedEnd));
    return dependencies;
  }

  hipGraphNode_t getTailOfLastCapturedNodeChain() {
    if (lastDependencies.size() == 0) {
      size_t numEdges;
      checkCudaErrors(hipGraphGetEdges(this->graph, nullptr, nullptr, &numEdges));
      auto from = std::make_unique<hipGraphNode_t[]>(numEdges);
      auto to = std::make_unique<hipGraphNode_t[]>(numEdges);
      checkCudaErrors(hipGraphGetEdges(this->graph, from.get(), to.get(), &numEdges));

      std::map<hipGraphNode_t, bool> hasOutGoingEdge;
      std::set<hipGraphNode_t> noOutGoingEdgeNodes;
      for (int i = 0; i < numEdges; i++) {
        hasOutGoingEdge[from[i]] = true;
        noOutGoingEdgeNodes.erase(from[i]);
        if (!hasOutGoingEdge[to[i]])
          noOutGoingEdgeNodes.insert(to[i]);
      }

      assert(noOutGoingEdgeNodes.size() == 1);

      return *noOutGoingEdgeNodes.begin();
    } else {
      auto nodeBeforeChain = lastDependencies[0];
      size_t numDependentNodes;
      checkCudaErrors(hipGraphNodeGetDependentNodes(nodeBeforeChain, nullptr, &numDependentNodes));

      assert(numDependentNodes > 0);

      auto dependentNodes = std::make_unique<hipGraphNode_t[]>(numDependentNodes);
      checkCudaErrors(hipGraphNodeGetDependentNodes(nodeBeforeChain, dependentNodes.get(), &numDependentNodes));

      hipGraphNode_t chainBeginningNode;
      for (int i = 0; i < numDependentNodes; i++) {
        if (!visited[dependentNodes[i]]) {
          chainBeginningNode = dependentNodes[i];
          break;
        }
      }

      auto u = chainBeginningNode;
      while (true) {
        visited[u] = true;
        checkCudaErrors(hipGraphNodeGetDependentNodes(u, nullptr, &numDependentNodes));
        if (numDependentNodes == 0) break;

        assert(numDependentNodes == 1);

        hipGraphNode_t v;
        checkCudaErrors(hipGraphNodeGetDependentNodes(u, &v, &numDependentNodes));
        u = v;
      }

      return u;
    }
  }
};

/*
 * TILED CHOLESKY DECOMPOSITION ALGORITHM
 * 
 * OVERVIEW:
 * This function implements a tiled Cholesky decomposition for large matrices on GPUs.
 * It decomposes a symmetric positive-definite matrix A into a product L×L^T
 * where L is a lower triangular matrix.
 * 
 * The implementation uses a tiled approach where an N×N matrix is divided into
 * T×T tiles, each of size B×B (where N = B×T). This improves memory locality
 * and enables processing of matrices larger than GPU memory.
 * 
 * ALGORITHM STEPS:
 * For each tile-column k = 0 to T-1:
 *   1. POTRF: Factor diagonal tile A[k,k] = L[k,k]×L[k,k]^T
 *   2. TRSM: For each tile below diagonal (i > k), solve L[i,k]×L[k,k]^T = A[i,k]
 *   3. SYRK: For each future diagonal tile (i > k), update A[i,i] = A[i,i] - L[i,k]×L[i,k]^T
 *   4. GEMM: For remaining tiles (j > i > k), update A[j,i] = A[j,i] - L[j,k]×L[i,k]^T
 * 
 * KEY COMPONENTS:
 * - CUDA Graph Construction: Records operations with dependencies
 * - Task Management: Tracks operations for replay during optimization
 * - Memory Management: Handles data placement and movement
 * - Multiple Execution Paths: Standard, optimized, and beyond-device-capacity modes
 * 
 * EXECUTION MODES:
 * 1. Standard Execution: Direct CUDA graph execution
 * 2. Optimized Execution: With memory optimization (prefetching/offloading)
 * 3. Beyond-Device-Capacity: For matrices exceeding GPU memory
 * 
 * PERFORMANCE FEATURES:
 * - High-performance GPU libraries (cuSOLVER, cuBLAS)
 * - Parallelism through independent tile operations
 * - Memory optimization through CUDA graphs
 * - Optional performance profiling
 */
void tiledCholesky(bool optimize, bool verify) {
  // Configuration check for beyond-device-capacity mode
  // When matrix is too large for GPU memory, we need both optimization and a pre-computed plan
  if (ConfigurationManager::getConfig().tiledCholesky.mode
      == Configuration::TiledCholesky::Mode::readInputMatrixFromFileAndRunBeyondDeviceCapacity) {
    if (!ConfigurationManager::getConfig().generic.optimize
        || !ConfigurationManager::getConfig().optimization.loadExistingPlan) {
      LOG_TRACE_WITH_INFO("Must enable optimization and load existing plan when problem size is beyond device memory capacity");
      exit(-1);
    }
  }

  // Initialize timing and CUDA device
  SystemWallClock clock;
  clock.start();

  initializeCudaDevice();

  const size_t tileSize = B * B * sizeof(double);

  // SECTION 1: HOST DATA INITIALIZATION
  // Allocate and initialize the input matrix using pinned memory for faster host-device transfers
  clock.logWithCurrentTime("Initialzing host data");
  
  // Use pinned memory (hipHostMalloc) instead of pageable memory for better transfer performance
  double* h_originalMatrix = nullptr;
  checkCudaErrors(hipHostMalloc(&h_originalMatrix, N * N * sizeof(double)));  // Column-major

  if (ConfigurationManager::getConfig().tiledCholesky.mode
      == Configuration::TiledCholesky::Mode::readInputMatrixFromFileAndRunBeyondDeviceCapacity) {
    clock.logWithCurrentTime("Loading input matrix from file");
    std::ifstream fin(INPUT_MATRIX_FILE_PATH);
    std::string s;
    int i = 0;
    while (std::getline(fin, s)) {
      h_originalMatrix[i++] = std::stod(s);
    }
    clock.logWithCurrentTime("Input matrix loaded");
  } else {
    initializeHostData(h_originalMatrix);
  }

  clock.logWithCurrentTime("Host data initialized");

  if (ConfigurationManager::getConfig().tiledCholesky.mode
      == Configuration::TiledCholesky::Mode::dumpInputMatrixToFile) {
    clock.logWithCurrentTime("Dumping input matrix");
    std::ofstream fout(INPUT_MATRIX_FILE_PATH);
    fout << std::setprecision(10);
    for (size_t i = 0; i < N * N; i++) {
      fout << h_originalMatrix[i] << '\n';
    }
    clock.logWithCurrentTime("Input matrix dumped");
    // Free allocated memory before return
    checkCudaErrors(hipHostFree(h_originalMatrix));
    return;
  }

  // SECTION 2: DEVICE MEMORY ALLOCATION
  // Allocate GPU memory for each tile - either standard device memory or unified memory
  clock.logWithCurrentTime("Initialzing device data");
  std::vector<double *> d_tiles(T * T);
  for (int i = 0; i < T * T; i++) {
    if (ConfigurationManager::getConfig().generic.useUM
        || ConfigurationManager::getConfig().tiledCholesky.mode == Configuration::TiledCholesky::Mode::readInputMatrixFromFileAndRunBeyondDeviceCapacity) {
      // Unified Memory allows easier management of data that exceeds GPU memory
      checkCudaErrors(hipMallocManaged(&d_tiles[i], tileSize));
    } else {
      // Standard device memory for better performance when data fits in GPU
      checkCudaErrors(hipMalloc(&d_tiles[i], tileSize));
    }
  }

  // Data will be initialized later before execution
  clock.logWithCurrentTime("Device data initialized");

  // Helper function to access tiles by logical indices
  auto getMatrixBlock = [&](int i, int j) {
    return d_tiles[i + j * T];
  };

  // SECTION 3: MEMORY REGISTRATION
  // Register memory addresses with the memory optimization framework
  // This allows the framework to track and manage memory during optimization
  
  // Register all tile addresses in the memory optimization system
  for (int i = 0; i < T; i++)
  {
    for (int j = 0; j < T; j++)
    {
      MemoryManager::getInstance().registerManagedMemoryAddress(getMatrixBlock(i, j), tileSize);
      MemoryManager::getInstance().registerApplicationInput(getMatrixBlock(i, j));
      // MemoryManager::getInstance().registerApplicaftionOutput(getMatrixBlock(i, j));
    }  
  }  

  clock.logWithCurrentTime("Addresses registered");

  // SECTION 4: CUDA LIBRARY INITIALIZATION
  // Initialize CUDA libraries for linear algebra operations
  hipsolverHandle_t cusolverDnHandle;  // For Cholesky factorization (POTRF)
  hipsolverDnParams_t cusolverDnParams;
  hipblasHandle_t cublasHandle;          // For matrix operations (TRSM, SYRK, GEMM)
  checkCudaErrors(hipsolverDnCreate(&cusolverDnHandle));
  checkCudaErrors(hipsolverDnCreateParams(&cusolverDnParams));
  checkCudaErrors(hipblasCreate(&cublasHandle));

  // Constants used in matrix operations
  double *one, *minusOne;
  checkCudaErrors(hipMallocManaged(&one, sizeof(double)));
  checkCudaErrors(hipMallocManaged(&minusOne, sizeof(double)));
  *one = 1.0;
  *minusOne = -1.0;

  // Allocate workspace memory for Cholesky factorization (POTRF)
  size_t workspaceInBytesOnDevice, workspaceInBytesOnHost;
  checkCudaErrors(hipsolverDnXpotrf_bufferSize(
    cusolverDnHandle,
    cusolverDnParams,
    HIPBLAS_FILL_MODE_LOWER,
    B,
    HIP_R_64F,
    d_tiles[0],
    B,
    HIP_R_64F,
    &workspaceInBytesOnDevice,
    &workspaceInBytesOnHost
  ));
  void *h_workspace, *d_workspace;
  int *d_info;
  checkCudaErrors(hipHostMalloc(&h_workspace, workspaceInBytesOnHost));
  checkCudaErrors(hipMalloc(&d_workspace, workspaceInBytesOnDevice));
  checkCudaErrors(hipMallocManaged(&d_info, sizeof(int)));

  // SECTION 5: CUDA GRAPH CREATION
  // Create a CUDA graph to record all operations with dependencies
  hipGraph_t graph;
  checkCudaErrors(hipGraphCreate(&graph, 0));

  // Create a stream for asynchronous execution
  hipStream_t s;
  checkCudaErrors(hipStreamCreate(&s));

  // Set streams for CUDA libraries
  checkCudaErrors(hipsolverSetStream(cusolverDnHandle, s));
  checkCudaErrors(hipblasSetStream(cublasHandle, s));

  clock.logWithCurrentTime("Preparation done, start to record graph");

  // Create TaskManager_v2 with debug mode enabled
  TaskManager_v2 tmanager_v2(true);

  // SECTION 6: TILED CHOLESKY ALGORITHM IMPLEMENTATION
  // Begin recording algorithm operations to CUDA graph
  // checkCudaErrors(hipStreamBeginCapture(s, hipStreamCaptureModeGlobal));
  auto tiledCholeskyGraphCreator = std::make_unique<TiledCholeskyGraphCreator>(s, graph);

  for (int k = 0; k < T; k++) {
    // =====================================================================
    // STEP 1: POTRF - Cholesky factorization of diagonal tile
    // A[k][k] = POTRF(A[k][k])
    // L[k][k] = POTRF(A[k][k])
    // =====================================================================
    
    // Register POTRF task using TaskManager_v2
    tiledCholeskyGraphCreator->beginCaptureOperation(
      {k, k},         // Tile to write
      {{k, k}}        // Tiles to read
    );

    TaskId taskId_v2 = tmanager_v2.registerTask<std::function<void(hipStream_t, double*)>, double*>(
      [cusolverDnHandle, cusolverDnParams, d_workspace, workspaceInBytesOnDevice, 
       h_workspace, workspaceInBytesOnHost, d_info](hipStream_t stream, double* matrixblock_k_k) {
        checkCudaErrors(hipsolverSetStream(cusolverDnHandle, stream));

        checkCudaErrors(hipsolverDnXpotrf(
            cusolverDnHandle,
            cusolverDnParams,
            HIPBLAS_FILL_MODE_LOWER,  // Lower triangular (for Cholesky)
            B,               // Matrix size
            HIP_R_64F,              // Data type (double)
            matrixblock_k_k,         // Input/output matrix
            B,               // Leading dimension
            HIP_R_64F,              // Output data type
            d_workspace,             // Device workspace
            workspaceInBytesOnDevice,
            h_workspace,             // Host workspace
            workspaceInBytesOnHost,
            d_info                   // Error information
          ));
      },
      {getMatrixBlock(k, k)},  // inputs
      {getMatrixBlock(k, k)},  // outputs
      TaskManager_v2::makeArgs(getMatrixBlock(k, k)),  // default args
      "POTRF_task"  // task name
    );
    
    // Execute the task using TaskManager_v2
    tmanager_v2.execute(taskId_v2, s);
    tiledCholeskyGraphCreator->endCaptureOperation();

    // =====================================================================
    // STEP 2: TRSM - Triangular solve for tiles below diagonal
    // For each tile below diagonal (i > k)
    // =====================================================================
    for (int i = k + 1; i < T; i++) {
      // A[i][k] = TRSM(A[k][k], A[i][k])
      // L[i][k] * L[k][k]^T = A[i][k]
      // Solving for L[i][k]
      tiledCholeskyGraphCreator->beginCaptureOperation(
        {i, k},               // Tile to write
        {{k, k}, {i, k}}      // Tiles to read
      );
      
      // Register TRSM task using TaskManager_v2
      TaskId trsmTaskId = tmanager_v2.registerTask<std::function<void(hipStream_t, double*, double*)>, double*, double*>(
        [cublasHandle, one](hipStream_t stream, double* matrixblock_k_k, double* matrixblock_i_k) {
          checkCudaErrors(hipblasSetStream(cublasHandle, stream));
          checkCudaErrors(hipblasDtrsm(
            cublasHandle,
            HIPBLAS_SIDE_RIGHT,      // Multiply from right
            HIPBLAS_FILL_MODE_LOWER, // Lower triangular
            HIPBLAS_OP_T,            // Transpose
            HIPBLAS_DIAG_NON_UNIT,   // Non-unit diagonal
            B, B,   // Matrix dimensions
            one,                    // Alpha = 1.0
            matrixblock_k_k, B, // Triangular matrix
            matrixblock_i_k, B  // Input/output matrix
          ));
        },
        {getMatrixBlock(k, k), getMatrixBlock(i, k)},  // inputs
        {getMatrixBlock(i, k)},  // outputs
        TaskManager_v2::makeArgs(getMatrixBlock(k, k), getMatrixBlock(i, k)),  // default args
        "TRSM_task_" + std::to_string(i) + "_" + std::to_string(k)  // task name
      );
      
      // Execute the task using TaskManager_v2
      tmanager_v2.execute(trsmTaskId, s);
      tiledCholeskyGraphCreator->endCaptureOperation();
    }

    // =====================================================================
    // STEP 3 & 4: SYRK and GEMM - Update remaining submatrix
    // =====================================================================
    for (int i = k + 1; i < T; i++) {
      tiledCholeskyGraphCreator->beginCaptureOperation(
        {i, i},              // Tile to write
        {{i, i}, {i, k}}     // Tiles to read
      );
      // STEP 3: SYRK - Update diagonal tiles
      // A[i][i] = SYRK(A[i][k], A[i][i])
      // A[i][i] = A[i][i] - L[i][k] * L[i][k]^T
      
      // Register SYRK task using TaskManager_v2
      TaskId syrkTaskId = tmanager_v2.registerTask<std::function<void(hipStream_t, double*, double*)>, double*, double*>(
        [cublasHandle, minusOne, one](hipStream_t stream, double* matrixblock_i_k, double* matrixblock_i_i) {
          checkCudaErrors(hipblasSetStream(cublasHandle, stream));
          checkCudaErrors(hipblasDsyrk(
            cublasHandle,
            HIPBLAS_FILL_MODE_LOWER, // Lower triangular
            HIPBLAS_OP_N,           // No transpose
            B, B,                  // Matrix dimensions
            minusOne,              // Alpha = -1.0
            matrixblock_i_k, B, // Input matrix
            one,                   // Beta = 1.0
            matrixblock_i_i, B  // Input/output matrix
          ));
        },
        {getMatrixBlock(i, i), getMatrixBlock(i, k)},  // inputs
        {getMatrixBlock(i, i)},  // outputs
        TaskManager_v2::makeArgs(getMatrixBlock(i, k), getMatrixBlock(i, i)),  // default args
        "SYRK_task_" + std::to_string(i) + "_" + std::to_string(k)  // task name
      );
      

      // Execute the task using TaskManager_v2
      tmanager_v2.execute(syrkTaskId, s);
      tiledCholeskyGraphCreator->endCaptureOperation();
      // STEP 4: GEMM - Update off-diagonal tiles
      // For each tile below diagonal in column i
      for (int j = i + 1; j < T; j++) {
        // A[j][i] = GEMM(A[j][k], A[i][k])
        // A[j][i] = A[j][i] - L[j][k] * L[i][k]^T
        tiledCholeskyGraphCreator->beginCaptureOperation(
          {j, i},                     // Tile to write
          {{j, i}, {j, k}, {i, k}}    // Tiles to read
        );
        
        // Register GEMM task using TaskManager_v2
        TaskId gemmTaskId = tmanager_v2.registerTask<std::function<void(hipStream_t, double*, double*, double*)>, double*, double*, double*>(
          [cublasHandle, minusOne, one](hipStream_t stream, double* matrixblock_j_k, double* matrixblock_i_k, double* matrixblock_j_i) {
            // General matrix multiplication using cuBLAS
            checkCudaErrors(hipblasSetStream(cublasHandle, stream));
            checkCudaErrors(hipblasGemmEx(
              cublasHandle,
              HIPBLAS_OP_N,           // No transpose for first matrix
              HIPBLAS_OP_T,           // Transpose second matrix
              B, B, B,               // Matrix dimensions
              minusOne,              // Alpha = -1.0
              matrixblock_j_k, HIP_R_64F, B, // First input matrix
              matrixblock_i_k, HIP_R_64F, B, // Second input matrix
              one,                   // Beta = 1.0
              matrixblock_j_i, HIP_R_64F, B, // Input/output matrix
              HIPBLAS_COMPUTE_64F,    // Computation precision
              HIPBLAS_GEMM_DEFAULT    // Algorithm selection
            ));
          },
          {getMatrixBlock(j, i), getMatrixBlock(j, k), getMatrixBlock(i, k)},  // inputs
          {getMatrixBlock(j, i)},  // outputs
          TaskManager_v2::makeArgs(getMatrixBlock(j, k), getMatrixBlock(i, k), getMatrixBlock(j, i)),  // default args
          "GEMM_task_" + std::to_string(j) + "_" + std::to_string(i) + "_" + std::to_string(k)  // task name
        );
        
   
        // Execute the task using TaskManager_v2
        tmanager_v2.execute(gemmTaskId, s);
        tiledCholeskyGraphCreator->endCaptureOperation();
      }
    }
  }
  // Finish recording operations to CUDA graph
  // checkCudaErrors(hipStreamEndCapture(s, &graph));
  // checkCudaErrors(hipStreamDestroy(s));

  // Graph creation completed - now we can export it for debugging
  clock.logWithCurrentTime("Graph recorded");
  LOG_TRACE_WITH_INFO("Printing original graph to graph.dot");
  checkCudaErrors(hipGraphDebugDotPrint(graph, "./graph.dot", 0));

  clock.logWithCurrentTime("Graph printed");

  // SECTION 7: EXECUTION PATHS
  // The function has three different execution paths depending on configuration:
  // 1. Beyond-device-capacity mode
  // 2. Optimized execution mode
  // 3. Standard execution mode

  // =====================================================================
  // EXECUTION PATH 1: Beyond-device-capacity mode
  // For matrices that exceed GPU memory capacity
  // =====================================================================
  tmanager_v2.setExecutionMode(TaskManager_v2::ExecutionMode::Production);
  if (ConfigurationManager::getConfig().tiledCholesky.mode
      == Configuration::TiledCholesky::Mode::readInputMatrixFromFileAndRunBeyondDeviceCapacity) {
    // Optimize the CUDA graph for memory usage
    auto optimizedGraph = profileAndOptimize(graph);

    // Initialize device data from the host matrix
    initializeDeviceData(h_originalMatrix, d_tiles);

    // Execute the optimized graph with memory management
    float runningTime;
    auto& memManager = MemoryManager::getInstance();
     
    executeOptimizedGraph(
      optimizedGraph,
      // Create a lambda that matches the ExecuteRandomTaskBase signature
      [&tmanager_v2](int taskId, hipStream_t stream) {
        // Execute the task using TaskManager_v2
        tmanager_v2.execute(taskId, stream);
        return true; // Indicates successful execution
      },
      runningTime,
      memManager
    );

    checkCudaErrors(hipDeviceSynchronize());
    fmt::print("Total time used (s): {}\n", runningTime);

    return;
  }

  // =====================================================================
  // EXECUTION PATH 2: Optimized execution mode
  // Uses memory optimization framework to reduce memory usage
  // =====================================================================
  if (optimize) {
    // Create an optimized version of the graph
    auto optimizedGraph = profileAndOptimize(graph);

    // Run the optimized graph for each repetition
    for (int i = 0; i < ConfigurationManager::getConfig().generic.repeat; i++) {
      // Initialize device data from host matrix
      initializeDeviceData(h_originalMatrix, d_tiles);

      // Execute optimized graph with memory management
      float runningTime;
      auto& memManager = MemoryManager::getInstance();
      
      executeOptimizedGraph(
        optimizedGraph,
        // Create a lambda that matches the ExecuteRandomTaskBase signature
        [&tmanager_v2](int taskId, hipStream_t stream) {
          // Execute the task using TaskManager_v2
          tmanager_v2.execute(taskId, stream);
          return true; // Indicates successful execution
        },
        runningTime,
        memManager
      );
      checkCudaErrors(hipDeviceSynchronize());
      fmt::print("Total time used (s): {}\n", runningTime);
      memManager.prefetchAllDataToDevice();
      checkCudaErrors(hipDeviceSynchronize());
      fmt::print("Finalized iteration\n");
      
    }
  } 
  // =====================================================================
  // EXECUTION PATH 3: Standard execution mode
  // Direct CUDA graph execution without optimization
  // =====================================================================
  else {
    // For memory usage tracking
    PeakMemoryUsageProfiler peakMemoryUsageProfiler;
    CudaEventClock cudaEventClock;
    
    // Instantiate the CUDA graph for execution
    hipGraphExec_t graphExec;
    checkCudaErrors(hipGraphInstantiate(&graphExec, graph, nullptr, nullptr, 0));

    clock.logWithCurrentTime("Graph instantiated, start execution");

    // Run for specified number of repetitions
    for (int i = 0; i < ConfigurationManager::getConfig().generic.repeat; i++) {
      // Initialize device data from host matrix
      initializeDeviceData(h_originalMatrix, d_tiles);

      // Create a new stream for execution
      hipStream_t execStream;
      checkCudaErrors(hipStreamCreate(&execStream));
      
      // Unified Memory prefetching (if enabled)
      if (ConfigurationManager::getConfig().generic.useUM) {
        // Limit available memory for unified memory
        size_t available = 1024ULL * 1024ULL * ConfigurationManager::getConfig().generic.availableMemoryForUMInMiB;
        reduceAvailableMemoryForUM(available);

        // Prefetch data to device (up to available memory limit)
        size_t sum = 0;
        for (int i = 0; i < T * T; i++) {
          if (sum + (tileSize) > available) {
            break;
          }
          checkCudaErrors(hipMemPrefetchAsync(
            d_tiles[i],
            tileSize,
            ConfigurationManager::getConfig().execution.mainDeviceId,
            execStream
          ));
        }
        checkCudaErrors(hipStreamSynchronize(execStream));
      }

      // Optional memory usage profiling
      if (ConfigurationManager::getConfig().execution.measurePeakMemoryUsage) {
        peakMemoryUsageProfiler.start();
      }
      
      // Execute and time the graph
      cudaEventClock.start();
      checkCudaErrors(hipGraphLaunch(graphExec, execStream));
      cudaEventClock.end();

      checkCudaErrors(hipDeviceSynchronize());

      // Report peak memory usage if measured
      if (ConfigurationManager::getConfig().execution.measurePeakMemoryUsage) {
        const auto peakMemoryUsage = peakMemoryUsageProfiler.end();
        fmt::print(
          "Peak memory usage (MiB): {:.2f}\n",
          static_cast<float>(peakMemoryUsage) / 1024.0 / 1024.0
        );
      }

      // Report execution time
      fmt::print("Total time used (s): {}\n", cudaEventClock.getTimeInSeconds());

      // Reset memory limit for unified memory
      if (ConfigurationManager::getConfig().generic.useUM) {
        resetAvailableMemoryForUM();
      }
      
      // Clean up the execution stream
      checkCudaErrors(hipStreamDestroy(execStream));
    }
  }

  clock.logWithCurrentTime("Synchronization done");

  // SECTION 8: RESULT VERIFICATION
  // Optionally verify the correctness of the Cholesky factorization
  if (verify) {
    clock.logWithCurrentTime("Start verification");
    fmt::print("Result passes verification: {}\n", verifyCholeskyDecompositionPartially(h_originalMatrix, d_tiles, N, B));
    clock.logWithCurrentTime("Verification done");
  }

  clock.logWithCurrentTime("All finished");

  // SECTION 9: CLEANUP
  // Free all allocated resources
  checkCudaErrors(hipHostFree(h_workspace));
  checkCudaErrors(hipFree(d_workspace));
  checkCudaErrors(hipHostFree(h_originalMatrix));  // Free the pinned host memory
  
  // Use freeManagedMemory to free both device and storage memory for each tile
  auto& memManager = MemoryManager::getInstance();
  for (auto d_tile : d_tiles) {
    memManager.freeManagedMemory(d_tile);
  }
}


int main(int argc, char **argv) {
  auto cmdl = argh::parser(argc, argv);
  std::string configFilePath;
  cmdl("configFile", "config.json") >> configFilePath;

  ConfigurationManager::exportDefaultConfiguration();
  ConfigurationManager::loadConfiguration(configFilePath);

  N = ConfigurationManager::getConfig().tiledCholesky.n;
  T = ConfigurationManager::getConfig().tiledCholesky.t;
  B = N / T;

  tiledCholesky(
    ConfigurationManager::getConfig().generic.optimize,
    ConfigurationManager::getConfig().generic.verify
  );
  return 0;
}
