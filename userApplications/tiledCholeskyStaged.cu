#include "hip/hip_runtime.h"
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hiprand.h>
#include <hipsolver.h>
#include <fmt/core.h>

#include <algorithm>
#include <cassert>
#include <cmath>
#include <cstdlib>
#include <fstream>
#include <initializer_list>
#include <iostream>
#include <iomanip>
#include <limits>
#include <map>
#include <memory>
#include <set>
#include <tuple>
#include <vector>

#include "../include/argh.h"
#include "memopt.hpp"

using namespace memopt;

// Global variables
size_t N; // Matrix dimension
size_t T; // Number of tiles 
size_t B; // Block size (N/T)
size_t* current_block_size = &B; // Pointer to current block size for kernels

const std::string INPUT_MATRIX_FILE_PATH = "tiledCholeskyInputMatrix.in";

// Kernels from original
__global__ void makeMatrixSymmetric(double *d_matrix, size_t n) {
  size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  size_t x = idx / n;
  size_t y = idx % n;

  if (x >= y || x >= n || y >= n) {
    return;
  }

  double average = 0.5 * (d_matrix[x * n + y] + d_matrix[y * n + x]);
  d_matrix[x * n + y] = average;
  d_matrix[y * n + x] = average;
}

__global__ void addIdenticalMatrix(double *d_matrix, size_t n) {
  size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= n) {
    return;
  }
  d_matrix[idx * n + idx] += n;
}

void generateRandomSymmetricPositiveDefiniteMatrix(double *h_A, const size_t n) {
  double *d_A;
  checkCudaErrors(hipMalloc(&d_A, n * n * sizeof(double)));

  hiprandGenerator_t prng;
  hiprandCreateGenerator(&prng, HIPRAND_RNG_PSEUDO_XORWOW);
  hiprandSetPseudoRandomGeneratorSeed(prng, (unsigned long long)clock());
  hiprandGenerateUniformDouble(prng, d_A, n * n);

  size_t numThreads = 1024;
  size_t numBlocks = (n * n + numThreads - 1) / numThreads;
  makeMatrixSymmetric<<<numBlocks, numThreads>>>(d_A, n);

  numThreads = 1024;
  numBlocks = (n + numThreads - 1) / numThreads;
  addIdenticalMatrix<<<numBlocks, numThreads>>>(d_A, n);

  checkCudaErrors(hipDeviceSynchronize());
  checkCudaErrors(hipMemcpy(h_A, d_A, n * n * sizeof(double), hipMemcpyDefault));
  checkCudaErrors(hipDeviceSynchronize());
  checkCudaErrors(hipFree(d_A));
  hiprandDestroyGenerator(prng);
}

void initializeDeviceData(double *h_originalMatrix, std::vector<double *> &d_tiles) {
  fmt::print("Initializing device data for {} tiles with {}x{} matrix\n", d_tiles.size(), N, N);
  
  for (int i = 0; i < T; i++) {
    for (int j = 0; j < T; j++) {
      for (int k = 0; k < B; k++) {
        auto& memManager = MemoryManager::getInstance();
        void* srcAddress = memManager.getAddress(d_tiles[i + j * T]);
        
        // Handle memory manager address resolution properly
        if (srcAddress == d_tiles[i + j * T]) {
          srcAddress = memManager.getStoragePtr(d_tiles[i + j * T]);
          if (srcAddress == nullptr) {
            srcAddress = d_tiles[i + j * T];
          }
        }

        checkCudaErrors(hipMemcpy(
          (char*)srcAddress + B * k * sizeof(double),
          h_originalMatrix + N * (j * B + k) + B * i,
          B * sizeof(double),
          hipMemcpyDefault
        ));
      }
    }
  }
  checkCudaErrors(hipDeviceSynchronize());
}

// Simplified structural verification
bool verifyCholeskyDecompositionPartially(double *A, std::vector<double *> &d_tiles) {
  auto& memManager = MemoryManager::getInstance();
  const size_t t = T;
  const size_t matrix_size_mb = (N * N * sizeof(double)) / (1024 * 1024);
  
  fmt::print("Performing structural verification (checking diagonal positivity, matrix size: {}MB)...\n", matrix_size_mb);
  
  bool validation_passed = true;
  
  // Check diagonal positivity
  for (int i = 0; i < t; i++) {
    double* h_tile;
    checkCudaErrors(hipHostMalloc(&h_tile, B * B * sizeof(double)));
    
    bool copySuccess = memManager.copyManagedArrayToHost(d_tiles[i + i * t], h_tile, B * B * sizeof(double));
    if (!copySuccess) {
      fmt::print("ERROR: Failed to copy diagonal tile [{},{}]\n", i, i);
      checkCudaErrors(hipHostFree(h_tile));
      return false;
    }
    
    // Check diagonal elements are positive
    for (int k = 0; k < B; k++) {
      if (h_tile[k * B + k] <= 0.0) {
        fmt::print("❌ DIAGONAL CHECK FAILED: Non-positive element at tile [{},{}], position [{},{}]: {:.6f}\n", 
                   i, i, k, k, h_tile[k * B + k]);
        validation_passed = false;
      }
    }
    
    checkCudaErrors(hipHostFree(h_tile));
  }
  
  if (validation_passed) {
    fmt::print("✅ STRUCTURAL VERIFICATION PASSED: All diagonal elements are positive\n");
    return true;
  } else {
    fmt::print("❌ STRUCTURAL VERIFICATION FAILED: Issues found in diagonal elements\n");
    return false;
  }
}

void tiledCholeskyMemoryOptimized() {
  fmt::print("=== Tiled Cholesky Memory Optimized Demo ===\n");
  fmt::print("Matrix: {}x{}, {} tiles, {}x{} blocks ({:.2f} MB)\n", 
             N, N, T*T, B, B, 
             (double)(N * N * sizeof(double)) / (1024.0 * 1024.0));
  
  initializeCudaDevice();

  // =========================================================================
  // PHASE 1: BUILD COMPUTATION GRAPH
  // =========================================================================
  fmt::print("\n--- PHASE 1: Build Computation Graph ---\n");
  
  const size_t tileSize = B * B * sizeof(double);
  
  // Generate matrix
  double* h_matrix = nullptr;
  checkCudaErrors(hipHostMalloc(&h_matrix, N * N * sizeof(double)));
  generateRandomSymmetricPositiveDefiniteMatrix(h_matrix, N);
  
  // Allocate GPU tiles
  std::vector<double*> d_tiles;
  auto& memManager = MemoryManager::getInstance();
  
  auto getMatrixBlock = [&d_tiles](int i, int j) -> double* {
    return d_tiles[i + j * T];
  };
  
  for (int i = 0; i < T * T; i++) {
    double *d_tile;
    checkCudaErrors(hipMalloc(&d_tile, tileSize));
    d_tiles.push_back(d_tile);
    memManager.registerManagedMemoryAddress(d_tile, tileSize);
  }
  
  double totalManagedMemoryMB = memManager.GetMemoryManagedSizeInMB();
  fmt::print("Total managed memory: {:.2f} MB\n", totalManagedMemoryMB);

  // CUDA library setup
  hipsolverHandle_t cusolverDnHandle;
  hipsolverDnParams_t cusolverDnParams;
  hipblasHandle_t cublasHandle;
  checkCudaErrors(hipsolverDnCreate(&cusolverDnHandle));
  checkCudaErrors(hipsolverDnCreateParams(&cusolverDnParams));
  checkCudaErrors(hipblasCreate(&cublasHandle));

  double *one, *minusOne;
  checkCudaErrors(hipMallocManaged(&one, sizeof(double)));
  checkCudaErrors(hipMallocManaged(&minusOne, sizeof(double)));
  *one = 1.0;
  *minusOne = -1.0;

  // Workspace for cuSOLVER
  size_t workspaceInBytesOnDevice, workspaceInBytesOnHost;
  checkCudaErrors(hipsolverDnXpotrf_bufferSize(
    cusolverDnHandle, cusolverDnParams, HIPBLAS_FILL_MODE_LOWER, B,
    HIP_R_64F, d_tiles[0], B, HIP_R_64F,
    &workspaceInBytesOnDevice, &workspaceInBytesOnHost
  ));
  
  void *h_workspace, *d_workspace;
  int *d_info;
  checkCudaErrors(hipHostMalloc(&h_workspace, workspaceInBytesOnHost));
  checkCudaErrors(hipMalloc(&d_workspace, workspaceInBytesOnDevice));
  checkCudaErrors(hipMallocManaged(&d_info, sizeof(int)));

  // Build CUDA graphs for each stage
  fmt::print("Building staged CUDA graphs...\n");
  
  hipStream_t s;
  checkCudaErrors(hipStreamCreate(&s));
  checkCudaErrors(hipsolverSetStream(cusolverDnHandle, s));
  checkCudaErrors(hipblasSetStream(cublasHandle, s));

  TaskManager_v2 tmanager_v2(true);
  
  // Store stage task IDs for staged execution
  std::vector<std::vector<TaskId>> stageTaskIds;
  
  // Build tasks for each stage separately  
  for (int k = 0; k < T; k++) {
    fmt::print("Building tasks for stage k={}/{}\n", k, T-1);
    std::vector<TaskId> currentStageTasks;
    // POTRF - Cholesky factorization of diagonal tile
    std::vector<void*> inputs = {static_cast<void*>(getMatrixBlock(k, k))};
    std::vector<void*> outputs = {static_cast<void*>(getMatrixBlock(k, k))};
    
    TaskId potrfTaskId = tmanager_v2.registerTask<std::function<void(hipStream_t, double*, void*, size_t, int*)>, double*, void*, size_t, int*>(
      [cusolverDnHandle, cusolverDnParams, workspaceInBytesOnHost](hipStream_t stream, double* matrixblock_k_k, void* d_workspace, size_t workspaceInBytesOnDevice, int* d_info) {
        checkCudaErrors(hipsolverSetStream(cusolverDnHandle, stream));
        checkCudaErrors(hipsolverDnXpotrf(
          cusolverDnHandle, cusolverDnParams, HIPBLAS_FILL_MODE_LOWER, *current_block_size,
          HIP_R_64F, matrixblock_k_k, *current_block_size, HIP_R_64F,
          d_workspace, workspaceInBytesOnDevice, nullptr, workspaceInBytesOnHost, d_info
        ));
      },
      inputs, outputs,
      TaskManager_v2::makeArgs(getMatrixBlock(k, k), d_workspace, workspaceInBytesOnDevice, d_info),
      "POTRF_task_" + std::to_string(k)
    );
    
    currentStageTasks.push_back(potrfTaskId);

    // TRSM - Triangular solve 
    for (int i = k + 1; i < T; i++) {
      inputs = {static_cast<void*>(getMatrixBlock(i, k)), static_cast<void*>(getMatrixBlock(k, k))};
      outputs = {static_cast<void*>(getMatrixBlock(i, k))};
      
      TaskId trsmTaskId = tmanager_v2.registerTask<std::function<void(hipStream_t, double*, double*)>, double*, double*>(
        [cublasHandle, one](hipStream_t stream, double* matrixblock_k_k, double* matrixblock_i_k) {
          checkCudaErrors(hipblasSetStream(cublasHandle, stream));
          checkCudaErrors(hipblasDtrsm(
            cublasHandle, HIPBLAS_SIDE_RIGHT, HIPBLAS_FILL_MODE_LOWER,
            HIPBLAS_OP_T, HIPBLAS_DIAG_NON_UNIT, *current_block_size, *current_block_size, one,
            matrixblock_k_k, *current_block_size, matrixblock_i_k, *current_block_size
          ));
        },
        inputs, outputs,
        TaskManager_v2::makeArgs(getMatrixBlock(k, k), getMatrixBlock(i, k)),
        "TRSM_task_" + std::to_string(i) + "_" + std::to_string(k)
      );
      
      currentStageTasks.push_back(trsmTaskId);
    }

    // SYRK - Update diagonal tiles
    for (int i = k + 1; i < T; i++) {
      inputs = {static_cast<void*>(getMatrixBlock(i, i)), static_cast<void*>(getMatrixBlock(i, k))};
      outputs = {static_cast<void*>(getMatrixBlock(i, i))};
      
      TaskId syrkTaskId = tmanager_v2.registerTask<std::function<void(hipStream_t, double*, double*)>, double*, double*>(
        [cublasHandle, minusOne, one](hipStream_t stream, double* matrixblock_i_i, double* matrixblock_i_k) {
          checkCudaErrors(hipblasSetStream(cublasHandle, stream));
          checkCudaErrors(hipblasDsyrk(
            cublasHandle, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N,
            *current_block_size, *current_block_size, minusOne, matrixblock_i_k, *current_block_size, one, matrixblock_i_i, *current_block_size
          ));
        },
        inputs, outputs,
        TaskManager_v2::makeArgs(getMatrixBlock(i, i), getMatrixBlock(i, k)),
        "SYRK_task_" + std::to_string(i) + "_" + std::to_string(k)
      );
      
      currentStageTasks.push_back(syrkTaskId);
    }

    // GEMM - Update off-diagonal tiles
    for (int i = k + 1; i < T; i++) {
      for (int j = k + 1; j < i; j++) {
        inputs = {static_cast<void*>(getMatrixBlock(i, j)), static_cast<void*>(getMatrixBlock(i, k)), static_cast<void*>(getMatrixBlock(j, k))};
        outputs = {static_cast<void*>(getMatrixBlock(i, j))};
        
        TaskId gemmTaskId = tmanager_v2.registerTask<std::function<void(hipStream_t, double*, double*, double*)>, double*, double*, double*>(
          [cublasHandle, minusOne, one](hipStream_t stream, double* matrixblock_i_j, double* matrixblock_i_k, double* matrixblock_j_k) {
            checkCudaErrors(hipblasSetStream(cublasHandle, stream));
            checkCudaErrors(hipblasDgemm(
              cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_T,
              *current_block_size, *current_block_size, *current_block_size, minusOne, matrixblock_i_k, *current_block_size, matrixblock_j_k, *current_block_size, one, matrixblock_i_j, *current_block_size
            ));
          },
          inputs, outputs,
          TaskManager_v2::makeArgs(getMatrixBlock(i, j), getMatrixBlock(i, k), getMatrixBlock(j, k)),
          "GEMM_task_" + std::to_string(i) + "_" + std::to_string(j) + "_" + std::to_string(k)
        );
        
        currentStageTasks.push_back(gemmTaskId);
      }
    }
    
    // Store tasks for this stage
    stageTaskIds.push_back(currentStageTasks);
    fmt::print("Stage {} has {} tasks\n", k, currentStageTasks.size());
  }

  // =========================================================================
  // PHASE 2: STAGED OPTIMIZATION
  // =========================================================================
  fmt::print("\n--- PHASE 2: Staged Memory Optimization ---\n");
  
  double initialPeakMemory = memManager.GetMemoryManagedSizeInMB();
  fmt::print("Initial peak memory: {:.2f} MB\n", initialPeakMemory);
  
  // Initialize data before optimization 
  initializeDeviceData(h_matrix, d_tiles);
  
  // Use staged profiling and optimization
  fmt::print("Using staged profiling and optimization with {} stages...\n", T);
  auto optimizedGraph = profileAndOptimizeStaged(tmanager_v2, stageTaskIds, s);
  
  fmt::print("Original peak memory usage (MiB): {:.2f}\n", optimizedGraph.originalMemoryUsage);
  fmt::print("Optimized peak memory usage (MiB): {:.2f}\n", optimizedGraph.anticipatedPeakMemoryUsage);
  fmt::print("Memory reduction: {:.2f} MiB ({:.1f}%)\n", 
             optimizedGraph.originalMemoryUsage - optimizedGraph.anticipatedPeakMemoryUsage,
             ((optimizedGraph.originalMemoryUsage - optimizedGraph.anticipatedPeakMemoryUsage) / optimizedGraph.originalMemoryUsage) * 100);

  // Move all data to storage before execution (required for optimized execution)
  fmt::print("Moving all data to storage for optimized execution...\n");
  memManager.offloadAllManagedMemoryToStorage();
  fmt::print("✅ All data moved to CPU/storage\n");
  
  // =========================================================================
  // PHASE 3: EXECUTE WITH OPTIMIZED MEMORY
  // =========================================================================
  fmt::print("\n--- PHASE 3: Execute with Optimized Memory ---\n");
  
  // Reinitialize data after optimization
  initializeDeviceData(h_matrix, d_tiles);
  
  // Get current GPU memory info
  size_t free_mem, total_mem;
  checkCudaErrors(hipMemGetInfo(&free_mem, &total_mem));
  fmt::print("GPU Memory - Total: {:.2f} MB, Free: {:.2f} MB\n", 
             (double)total_mem / (1024.0 * 1024.0), (double)free_mem / (1024.0 * 1024.0));
  
  // Start peak memory monitoring
  fmt::print("🔍 Starting continuous GPU memory monitoring during execution...\n");
  PeakMemoryUsageProfiler peakProfiler(10); // Sample every 10ms
  peakProfiler.start();
  
  // Run the optimized graph
  float runningTime;
  executeOptimizedGraph(
    optimizedGraph,
    [&tmanager_v2](int taskId, std::map<void*, void*> addressMapping, hipStream_t stream) {
      tmanager_v2.execute(taskId, stream);
    },
    runningTime,
    memManager
  );
  
  // Get peak memory usage
  size_t peakMemoryBytes = peakProfiler.end();
  double peakMemoryMB = (double)peakMemoryBytes / (1024.0 * 1024.0);
  
  fmt::print("✅ Optimized execution completed!\n");
  fmt::print("Execution time: {:.3f} ms\n", runningTime * 1000.0f);
  fmt::print("📊 Peak GPU memory usage during execution: {:.2f} MB\n", peakMemoryMB);
  
  // Verify results
  bool result = verifyCholeskyDecompositionPartially(h_matrix, d_tiles);
  
  // Cleanup
  checkCudaErrors(hipStreamDestroy(s));
  checkCudaErrors(hipsolverDnDestroyParams(cusolverDnParams));
  checkCudaErrors(hipsolverDnDestroy(cusolverDnHandle));
  checkCudaErrors(hipblasDestroy(cublasHandle));
  
  checkCudaErrors(hipFree(one));
  checkCudaErrors(hipFree(minusOne));
  checkCudaErrors(hipFree(d_workspace));
  checkCudaErrors(hipFree(d_info));
  checkCudaErrors(hipHostFree(h_workspace));
  
  // Free managed memory properly using MemoryManager
  for (auto d_tile : d_tiles) {
    memManager.freeManagedMemory(d_tile);
  }
  checkCudaErrors(hipHostFree(h_matrix));
  
  fmt::print("\nFinal result: {}\n", result ? "SUCCESS" : "FAILED");
}

int main(int argc, char *argv[]) {
  // Simple command line parsing or use defaults
  N = (argc > 1) ? std::atoi(argv[1]) : 1024;  // Matrix dimension
  T = (argc > 2) ? std::atoi(argv[2]) : 4;     // Number of tiles
  
  // Calculate block size
  B = N / T;
  
  // Validation
  if (N % T != 0) {
    fmt::print("ERROR: Matrix dimension must be divisible by tile count\n");
    return -1;
  }
  
  ConfigurationManager::exportDefaultConfiguration();
  ConfigurationManager::loadConfiguration("config.json");
  
  fmt::print("Configuration: N={}, T={}, B={}\n", N, T, B);
  
  // Run the memory optimized version
  tiledCholeskyMemoryOptimized();
  
  return 0;
}