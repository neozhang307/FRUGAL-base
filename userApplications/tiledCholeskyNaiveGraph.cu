#include "hip/hip_runtime.h"
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hiprand.h>
#include <hipsolver.h>
#include <fmt/core.h>

#include <algorithm>
#include <cassert>
#include <cmath>
#include <cstdlib>
#include <fstream>
#include <initializer_list>
#include <iostream>
#include <iomanip>
#include <limits>
#include <map>
#include <memory>
#include <set>
#include <tuple>
#include <vector>

#include "../include/argh.h"
#include "memopt.hpp"

using namespace memopt;

// Global variables
size_t N; // Matrix dimension
size_t T; // Number of tiles 
size_t B; // Block size (N/T)
size_t* current_block_size = &B; // Pointer to current block size for kernels

const std::string INPUT_MATRIX_FILE_PATH = "tiledCholeskyInputMatrix.in";

// Kernels from original
__global__ void makeMatrixSymmetric(double *d_matrix, size_t n) {
  size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  size_t x = idx / n;
  size_t y = idx % n;

  if (x >= y || x >= n || y >= n) {
    return;
  }

  double average = 0.5 * (d_matrix[x * n + y] + d_matrix[y * n + x]);
  d_matrix[x * n + y] = average;
  d_matrix[y * n + x] = average;
}

__global__ void addIdenticalMatrix(double *d_matrix, size_t n) {
  size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= n) {
    return;
  }
  d_matrix[idx * n + idx] += n;
}

void generateRandomSymmetricPositiveDefiniteMatrix(double *h_A, const size_t n) {
  double *d_A;
  checkCudaErrors(hipMalloc(&d_A, n * n * sizeof(double)));

  hiprandGenerator_t prng;
  hiprandCreateGenerator(&prng, HIPRAND_RNG_PSEUDO_XORWOW);
  hiprandSetPseudoRandomGeneratorSeed(prng, (unsigned long long)clock());
  hiprandGenerateUniformDouble(prng, d_A, n * n);

  size_t numThreads = 1024;
  size_t numBlocks = (n * n + numThreads - 1) / numThreads;
  makeMatrixSymmetric<<<numBlocks, numThreads>>>(d_A, n);

  numThreads = 1024;
  numBlocks = (n + numThreads - 1) / numThreads;
  addIdenticalMatrix<<<numBlocks, numThreads>>>(d_A, n);

  checkCudaErrors(hipDeviceSynchronize());
  checkCudaErrors(hipMemcpy(h_A, d_A, n * n * sizeof(double), hipMemcpyDefault));
  checkCudaErrors(hipDeviceSynchronize());
  checkCudaErrors(hipFree(d_A));
  hiprandDestroyGenerator(prng);
}

void initializeDeviceData(double *h_originalMatrix, std::vector<double *> &d_tiles) {
  fmt::print("Initializing device data for {} tiles with {}x{} matrix\n", d_tiles.size(), N, N);
  
  for (int i = 0; i < T; i++) {
    for (int j = 0; j < T; j++) {
      for (int k = 0; k < B; k++) {
        checkCudaErrors(hipMemcpy(
          (char*)d_tiles[i + j * T] + B * k * sizeof(double),
          h_originalMatrix + N * (j * B + k) + B * i,
          B * sizeof(double),
          hipMemcpyDefault
        ));
      }
    }
  }
  checkCudaErrors(hipDeviceSynchronize());
}

// Simplified structural verification
bool verifyCholeskyDecompositionPartially(double *A, std::vector<double *> &d_tiles) {
  const size_t t = T;
  const size_t matrix_size_mb = (N * N * sizeof(double)) / (1024 * 1024);
  
  fmt::print("Performing structural verification (checking diagonal positivity, matrix size: {}MB)...\n", matrix_size_mb);
  
  bool validation_passed = true;
  
  // Check diagonal positivity
  for (int i = 0; i < t; i++) {
    double* h_tile;
    checkCudaErrors(hipHostMalloc(&h_tile, B * B * sizeof(double)));
    
    checkCudaErrors(hipMemcpy(h_tile, d_tiles[i + i * t], B * B * sizeof(double), hipMemcpyDefault));
    
    // Check diagonal elements are positive
    for (int k = 0; k < B; k++) {
      if (h_tile[k * B + k] <= 0.0) {
        fmt::print("❌ DIAGONAL CHECK FAILED: Non-positive element at tile [{},{}], position [{},{}]: {:.6f}\n", 
                   i, i, k, k, h_tile[k * B + k]);
        validation_passed = false;
      }
    }
    
    checkCudaErrors(hipHostFree(h_tile));
  }
  
  if (validation_passed) {
    fmt::print("✅ STRUCTURAL VERIFICATION PASSED: All diagonal elements are positive\n");
    return true;
  } else {
    fmt::print("❌ STRUCTURAL VERIFICATION FAILED: Issues found in diagonal elements\n");
    return false;
  }
}

void tiledCholeskyNaiveGraph() {
  fmt::print("=== Tiled Cholesky Naive Graph Demo ===\n");
  fmt::print("Matrix: {}x{}, {} tiles, {}x{} blocks ({:.2f} MB)\n", 
             N, N, T*T, B, B, 
             (double)(N * N * sizeof(double)) / (1024.0 * 1024.0));
  
  initializeCudaDevice();

  // =========================================================================
  // PHASE 1: SETUP AND ALLOCATE MEMORY
  // =========================================================================
  fmt::print("\n--- PHASE 1: Setup and Allocate Memory ---\n");
  
  const size_t tileSize = B * B * sizeof(double);
  
  // Generate matrix
  double* h_matrix = nullptr;
  checkCudaErrors(hipHostMalloc(&h_matrix, N * N * sizeof(double)));
  generateRandomSymmetricPositiveDefiniteMatrix(h_matrix, N);
  
  // Allocate GPU tiles with simple hipMalloc (no memory management)
  std::vector<double*> d_tiles;
  
  auto getMatrixBlock = [&d_tiles](int i, int j) -> double* {
    return d_tiles[i + j * T];
  };
  
  for (int i = 0; i < T * T; i++) {
    double *d_tile;
    checkCudaErrors(hipMalloc(&d_tile, tileSize));
    d_tiles.push_back(d_tile);
  }
  
  fmt::print("Total allocated memory: {:.2f} MB\n", (T * T * tileSize) / (1024.0 * 1024.0));

  // Initialize data
  initializeDeviceData(h_matrix, d_tiles);

  // CUDA library setup
  hipsolverHandle_t cusolverDnHandle;
  hipsolverDnParams_t cusolverDnParams;
  hipblasHandle_t cublasHandle;
  checkCudaErrors(hipsolverDnCreate(&cusolverDnHandle));
  checkCudaErrors(hipsolverDnCreateParams(&cusolverDnParams));
  checkCudaErrors(hipblasCreate(&cublasHandle));

  double *one, *minusOne;
  checkCudaErrors(hipMallocManaged(&one, sizeof(double)));
  checkCudaErrors(hipMallocManaged(&minusOne, sizeof(double)));
  *one = 1.0;
  *minusOne = -1.0;

  // Workspace for cuSOLVER
  size_t workspaceInBytesOnDevice, workspaceInBytesOnHost;
  checkCudaErrors(hipsolverDnXpotrf_bufferSize(
    cusolverDnHandle, cusolverDnParams, HIPBLAS_FILL_MODE_LOWER, B,
    HIP_R_64F, d_tiles[0], B, HIP_R_64F,
    &workspaceInBytesOnDevice, &workspaceInBytesOnHost
  ));
  
  void *h_workspace, *d_workspace;
  int *d_info;
  checkCudaErrors(hipHostMalloc(&h_workspace, workspaceInBytesOnHost));
  checkCudaErrors(hipMalloc(&d_workspace, workspaceInBytesOnDevice));
  checkCudaErrors(hipMallocManaged(&d_info, sizeof(int)));

  // =========================================================================
  // PHASE 2: BUILD NAIVE CUDA GRAPH USING TASKMANAGER_V2
  // =========================================================================
  fmt::print("\n--- PHASE 2: Build Naive CUDA Graph ---\n");
  
  hipStream_t s;
  checkCudaErrors(hipStreamCreate(&s));
  checkCudaErrors(hipsolverSetStream(cusolverDnHandle, s));
  checkCudaErrors(hipblasSetStream(cublasHandle, s));

  // Initialize TaskManager_v2 for naive graph construction
  TaskManager_v2 tmanager_v2(true);

  // Register all tasks in the correct dependency order
  fmt::print("Registering tasks for Cholesky decomposition...\n");
  
  // Tiled Cholesky algorithm - register tasks sequentially
  for (int k = 0; k < T; k++) {
    // POTRF - Cholesky factorization of diagonal tile
    std::vector<void*> inputs = {static_cast<void*>(getMatrixBlock(k, k))};
    std::vector<void*> outputs = {static_cast<void*>(getMatrixBlock(k, k))};
    
    TaskId potrfTaskId = tmanager_v2.registerTask<std::function<void(hipStream_t, double*, void*, size_t, int*)>, double*, void*, size_t, int*>(
      [cusolverDnHandle, cusolverDnParams, workspaceInBytesOnHost](hipStream_t stream, double* matrixblock_k_k, void* d_workspace, size_t workspaceInBytesOnDevice, int* d_info) {
        checkCudaErrors(hipsolverSetStream(cusolverDnHandle, stream));
        checkCudaErrors(hipsolverDnXpotrf(
          cusolverDnHandle, cusolverDnParams, HIPBLAS_FILL_MODE_LOWER, *current_block_size,
          HIP_R_64F, matrixblock_k_k, *current_block_size, HIP_R_64F,
          d_workspace, workspaceInBytesOnDevice, nullptr, workspaceInBytesOnHost, d_info
        ));
      },
      inputs, outputs,
      TaskManager_v2::makeArgs(getMatrixBlock(k, k), d_workspace, workspaceInBytesOnDevice, d_info),
      "POTRF_task_" + std::to_string(k)
    );
    
    fmt::print("✓ Registered POTRF task {}\n", k);

    // TRSM - Triangular solve 
    for (int i = k + 1; i < T; i++) {
      inputs = {static_cast<void*>(getMatrixBlock(i, k)), static_cast<void*>(getMatrixBlock(k, k))};
      outputs = {static_cast<void*>(getMatrixBlock(i, k))};
      
      TaskId trsmTaskId = tmanager_v2.registerTask<std::function<void(hipStream_t, double*, double*)>, double*, double*>(
        [cublasHandle, one](hipStream_t stream, double* matrixblock_k_k, double* matrixblock_i_k) {
          checkCudaErrors(hipblasSetStream(cublasHandle, stream));
          checkCudaErrors(hipblasDtrsm(
            cublasHandle, HIPBLAS_SIDE_RIGHT, HIPBLAS_FILL_MODE_LOWER,
            HIPBLAS_OP_T, HIPBLAS_DIAG_NON_UNIT, *current_block_size, *current_block_size, one,
            matrixblock_k_k, *current_block_size, matrixblock_i_k, *current_block_size
          ));
        },
        inputs, outputs,
        TaskManager_v2::makeArgs(getMatrixBlock(k, k), getMatrixBlock(i, k)),
        "TRSM_task_" + std::to_string(i) + "_" + std::to_string(k)
      );
      
      fmt::print("✓ Registered TRSM task {}_{}\n", i, k);
    }

    // SYRK - Update diagonal tiles
    for (int i = k + 1; i < T; i++) {
      inputs = {static_cast<void*>(getMatrixBlock(i, i)), static_cast<void*>(getMatrixBlock(i, k))};
      outputs = {static_cast<void*>(getMatrixBlock(i, i))};
      
      TaskId syrkTaskId = tmanager_v2.registerTask<std::function<void(hipStream_t, double*, double*)>, double*, double*>(
        [cublasHandle, minusOne, one](hipStream_t stream, double* matrixblock_i_i, double* matrixblock_i_k) {
          checkCudaErrors(hipblasSetStream(cublasHandle, stream));
          checkCudaErrors(hipblasDsyrk(
            cublasHandle, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N,
            *current_block_size, *current_block_size, minusOne, matrixblock_i_k, *current_block_size, one, matrixblock_i_i, *current_block_size
          ));
        },
        inputs, outputs,
        TaskManager_v2::makeArgs(getMatrixBlock(i, i), getMatrixBlock(i, k)),
        "SYRK_task_" + std::to_string(i) + "_" + std::to_string(k)
      );
      
      fmt::print("✓ Registered SYRK task {}_{}\n", i, k);
    }

    // GEMM - Update off-diagonal tiles
    for (int i = k + 1; i < T; i++) {
      for (int j = k + 1; j < i; j++) {
        inputs = {static_cast<void*>(getMatrixBlock(i, j)), static_cast<void*>(getMatrixBlock(i, k)), static_cast<void*>(getMatrixBlock(j, k))};
        outputs = {static_cast<void*>(getMatrixBlock(i, j))};
        
        TaskId gemmTaskId = tmanager_v2.registerTask<std::function<void(hipStream_t, double*, double*, double*)>, double*, double*, double*>(
          [cublasHandle, minusOne, one](hipStream_t stream, double* matrixblock_i_j, double* matrixblock_i_k, double* matrixblock_j_k) {
            checkCudaErrors(hipblasSetStream(cublasHandle, stream));
            checkCudaErrors(hipblasDgemm(
              cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_T,
              *current_block_size, *current_block_size, *current_block_size, minusOne, matrixblock_i_k, *current_block_size, matrixblock_j_k, *current_block_size, one, matrixblock_i_j, *current_block_size
            ));
          },
          inputs, outputs,
          TaskManager_v2::makeArgs(getMatrixBlock(i, j), getMatrixBlock(i, k), getMatrixBlock(j, k)),
          "GEMM_task_" + std::to_string(i) + "_" + std::to_string(j) + "_" + std::to_string(k)
        );
        
        fmt::print("✓ Registered GEMM task {}_{}_{}\n", i, j, k);
      }
    }
  }

  fmt::print("✅ Task registration completed! Total tasks: {}\n", tmanager_v2.taskCount());
  
  // Generate naive CUDA graph from registered tasks
  fmt::print("Generating naive CUDA graph from task sequence...\n");
  hipGraph_t graph = tmanager_v2.generateNaiveGraph(s);
  
  // Get graph stats
  size_t numNodes;
  checkCudaErrors(hipGraphGetNodes(graph, nullptr, &numNodes));
  fmt::print("📊 Generated graph contains {} nodes\n", numNodes);

  // =========================================================================
  // PHASE 3: EXECUTE THE NAIVE CUDA GRAPH
  // =========================================================================
  fmt::print("\n--- PHASE 3: Execute Naive CUDA Graph ---\n");
  
  // Reinitialize data for fresh execution
  initializeDeviceData(h_matrix, d_tiles);
  
  // Get current GPU memory info
  size_t free_mem, total_mem;
  checkCudaErrors(hipMemGetInfo(&free_mem, &total_mem));
  fmt::print("GPU Memory - Total: {:.2f} MB, Free: {:.2f} MB\n", 
             (double)total_mem / (1024.0 * 1024.0), (double)free_mem / (1024.0 * 1024.0));
  
  // Start peak memory monitoring
  fmt::print("🔍 Starting continuous GPU memory monitoring during execution...\n");
  PeakMemoryUsageProfiler peakProfiler(10); // Sample every 10ms
  peakProfiler.start();
  
  // Instantiate and execute the graph
  hipGraphExec_t graphExec;
  checkCudaErrors(hipGraphInstantiate(&graphExec, graph, nullptr, nullptr, 0));
  
  // Measure execution time
  hipEvent_t start, stop;
  checkCudaErrors(hipEventCreate(&start));
  checkCudaErrors(hipEventCreate(&stop));
  
  checkCudaErrors(hipEventRecord(start, s));
  checkCudaErrors(hipGraphLaunch(graphExec, s));
  checkCudaErrors(hipEventRecord(stop, s));
  checkCudaErrors(hipStreamSynchronize(s));
  
  float runningTime;
  checkCudaErrors(hipEventElapsedTime(&runningTime, start, stop));
  
  // Get peak memory usage
  size_t peakMemoryBytes = peakProfiler.end();
  double peakMemoryMB = (double)peakMemoryBytes / (1024.0 * 1024.0);
  
  fmt::print("✅ Naive graph execution completed!\n");
  fmt::print("Execution time: {:.3f} ms\n", runningTime);
  fmt::print("📊 Peak GPU memory usage during execution: {:.2f} MB\n", peakMemoryMB);
  
  // =========================================================================
  // PHASE 4: VERIFY RESULTS
  // =========================================================================
  fmt::print("\n--- PHASE 4: Verify Results ---\n");
  
  // Verify results
  bool result = verifyCholeskyDecompositionPartially(h_matrix, d_tiles);
  
  // =========================================================================
  // PHASE 5: CLEANUP
  // =========================================================================
  fmt::print("\n--- PHASE 5: Cleanup ---\n");
  
  checkCudaErrors(hipEventDestroy(start));
  checkCudaErrors(hipEventDestroy(stop));
  checkCudaErrors(hipGraphExecDestroy(graphExec));
  checkCudaErrors(hipGraphDestroy(graph));
  checkCudaErrors(hipStreamDestroy(s));
  checkCudaErrors(hipsolverDnDestroyParams(cusolverDnParams));
  checkCudaErrors(hipsolverDnDestroy(cusolverDnHandle));
  checkCudaErrors(hipblasDestroy(cublasHandle));
  
  checkCudaErrors(hipFree(one));
  checkCudaErrors(hipFree(minusOne));
  checkCudaErrors(hipFree(d_workspace));
  checkCudaErrors(hipFree(d_info));
  checkCudaErrors(hipHostFree(h_workspace));
  
  // Free device tiles
  for (auto d_tile : d_tiles) {
    checkCudaErrors(hipFree(d_tile));
  }
  checkCudaErrors(hipHostFree(h_matrix));
  
  fmt::print("\nFinal result: {}\n", result ? "SUCCESS" : "FAILED");
}

int main(int argc, char *argv[]) {
  // Simple command line parsing or use defaults
  N = (argc > 1) ? std::atoi(argv[1]) : 1024;  // Matrix dimension
  T = (argc > 2) ? std::atoi(argv[2]) : 4;     // Number of tiles
  
  // Calculate block size
  B = N / T;
  
  // Validation
  if (N % T != 0) {
    fmt::print("ERROR: Matrix dimension must be divisible by tile count\n");
    return -1;
  }
  
  ConfigurationManager::exportDefaultConfiguration();
  ConfigurationManager::loadConfiguration("config.json");
  
  fmt::print("Configuration: N={}, T={}, B={}\n", N, T, B);
  
  // Run the naive graph version
  tiledCholeskyNaiveGraph();
  
  return 0;
}