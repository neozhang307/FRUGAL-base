#include "hip/hip_runtime.h"
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hiprand.h>
#include <hipsolver.h>
#include <fmt/core.h>

#include <algorithm>
#include <cassert>
#include <cmath>
#include <cstdlib>
#include <fstream>
#include <initializer_list>
#include <iostream>
#include <iomanip>
#include <limits>
#include <map>
#include <memory>
#include <set>
#include <tuple>
#include <vector>

#include "../include/argh.h"
#include "memopt.hpp"

using namespace memopt;

// Global variables
size_t N; // Matrix dimension
size_t T; // Number of tiles 
size_t B; // Block size (N/T)
size_t* current_block_size = &B; // Pointer to current block size for kernels

const std::string INPUT_MATRIX_FILE_PATH = "tiledCholeskyInputMatrix.in";

// Kernels from original
__global__ void makeMatrixSymmetric(double *d_matrix, size_t n) {
  size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  size_t x = idx / n;
  size_t y = idx % n;

  if (x >= y || x >= n || y >= n) {
    return;
  }

  double average = 0.5 * (d_matrix[x * n + y] + d_matrix[y * n + x]);
  d_matrix[x * n + y] = average;
  d_matrix[y * n + x] = average;
}

__global__ void addIdenticalMatrix(double *d_matrix, size_t n) {
  size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= n) {
    return;
  }
  d_matrix[idx * n + idx] += n;
}

void generateRandomSymmetricPositiveDefiniteMatrix(double *h_A, const size_t n) {
  double *d_A;
  checkCudaErrors(hipMalloc(&d_A, n * n * sizeof(double)));

  hiprandGenerator_t prng;
  hiprandCreateGenerator(&prng, HIPRAND_RNG_PSEUDO_XORWOW);
  hiprandSetPseudoRandomGeneratorSeed(prng, (unsigned long long)clock());
  hiprandGenerateUniformDouble(prng, d_A, n * n);

  size_t numThreads = 1024;
  size_t numBlocks = (n * n + numThreads - 1) / numThreads;
  makeMatrixSymmetric<<<numBlocks, numThreads>>>(d_A, n);

  numThreads = 1024;
  numBlocks = (n + numThreads - 1) / numThreads;
  addIdenticalMatrix<<<numBlocks, numThreads>>>(d_A, n);

  checkCudaErrors(hipDeviceSynchronize());
  checkCudaErrors(hipMemcpy(h_A, d_A, n * n * sizeof(double), hipMemcpyDefault));
  checkCudaErrors(hipDeviceSynchronize());
  checkCudaErrors(hipFree(d_A));
  hiprandDestroyGenerator(prng);
}

void initializeDeviceData(double *h_originalMatrix, std::vector<double *> &d_tiles) {
  fmt::print("Initializing device data for {} tiles with {}x{} matrix\n", d_tiles.size(), N, N);
  
  for (int i = 0; i < T; i++) {
    for (int j = 0; j < T; j++) {
      for (int k = 0; k < B; k++) {
        auto& memManager = MemoryManager::getInstance();
        void* srcAddress = memManager.getAddress(d_tiles[i + j * T]);
        
        // Handle memory manager address resolution properly
        if (srcAddress == d_tiles[i + j * T]) {
          srcAddress = memManager.getStoragePtr(d_tiles[i + j * T]);
          if (srcAddress == nullptr) {
            srcAddress = d_tiles[i + j * T];
          }
        }

        checkCudaErrors(hipMemcpy(
          (char*)srcAddress + B * k * sizeof(double),
          h_originalMatrix + N * (j * B + k) + B * i,
          B * sizeof(double),
          hipMemcpyDefault
        ));
      }
    }
  }
  checkCudaErrors(hipDeviceSynchronize());
}

// Simplified structural verification
bool verifyCholeskyDecompositionPartially(double *A, std::vector<double *> &d_tiles) {
  auto& memManager = MemoryManager::getInstance();
  const size_t t = T;
  const size_t matrix_size_mb = (N * N * sizeof(double)) / (1024 * 1024);
  
  fmt::print("Performing structural verification (checking diagonal positivity, matrix size: {}MB)...\n", matrix_size_mb);
  
  bool validation_passed = true;
  
  // Check diagonal positivity
  for (int i = 0; i < t; i++) {
    double* h_tile;
    checkCudaErrors(hipHostMalloc(&h_tile, B * B * sizeof(double)));
    
    bool copySuccess = memManager.copyManagedArrayToHost(d_tiles[i + i * t], h_tile, B * B * sizeof(double));
    if (!copySuccess) {
      fmt::print("ERROR: Failed to copy diagonal tile [{},{}]\n", i, i);
      checkCudaErrors(hipHostFree(h_tile));
      return false;
    }
    
    // Check diagonal elements are positive
    for (int k = 0; k < B; k++) {
      if (h_tile[k * B + k] <= 0.0) {
        fmt::print("❌ DIAGONAL CHECK FAILED: Non-positive element at tile [{},{}], position [{},{}]: {:.6f}\n", 
                   i, i, k, k, h_tile[k * B + k]);
        validation_passed = false;
      }
    }
    
    checkCudaErrors(hipHostFree(h_tile));
  }
  
  if (validation_passed) {
    fmt::print("✅ STRUCTURAL VERIFICATION PASSED: All diagonal elements are positive\n");
    return true;
  } else {
    fmt::print("❌ STRUCTURAL VERIFICATION FAILED: Issues found in diagonal elements\n");
    return false;
  }
}

void tiledCholeskyNaiveGraph() {
  fmt::print("=== Tiled Cholesky Naive Graph Demo ===\n");
  fmt::print("Matrix: {}x{}, {} tiles, {}x{} blocks ({:.2f} MB)\n", 
             N, N, T*T, B, B, 
             (double)(N * N * sizeof(double)) / (1024.0 * 1024.0));
  
  initializeCudaDevice();

  // =========================================================================
  // PHASE 1: SETUP AND ALLOCATE MEMORY
  // =========================================================================
  fmt::print("\n--- PHASE 1: Setup and Allocate Memory ---\n");
  
  const size_t tileSize = B * B * sizeof(double);
  
  // Generate matrix
  double* h_matrix = nullptr;
  checkCudaErrors(hipHostMalloc(&h_matrix, N * N * sizeof(double)));
  generateRandomSymmetricPositiveDefiniteMatrix(h_matrix, N);
  
  // Allocate GPU tiles with memory management
  std::vector<double*> d_tiles;
  auto& memManager = MemoryManager::getInstance();
  
  auto getMatrixBlock = [&d_tiles](int i, int j) -> double* {
    return d_tiles[i + j * T];
  };
  
  for (int i = 0; i < T * T; i++) {
    double *d_tile;
    checkCudaErrors(hipMalloc(&d_tile, tileSize));
    d_tiles.push_back(d_tile);
    memManager.registerManagedMemoryAddress(d_tile, tileSize);
  }
  
  double totalManagedMemoryMB = memManager.GetMemoryManagedSizeInMB();
  fmt::print("Total managed memory: {:.2f} MB\n", totalManagedMemoryMB);

  // Initialize data
  initializeDeviceData(h_matrix, d_tiles);

  // CUDA library setup
  hipsolverHandle_t cusolverDnHandle;
  hipsolverDnParams_t cusolverDnParams;
  hipblasHandle_t cublasHandle;
  checkCudaErrors(hipsolverDnCreate(&cusolverDnHandle));
  checkCudaErrors(hipsolverDnCreateParams(&cusolverDnParams));
  checkCudaErrors(hipblasCreate(&cublasHandle));

  double *one, *minusOne;
  checkCudaErrors(hipMallocManaged(&one, sizeof(double)));
  checkCudaErrors(hipMallocManaged(&minusOne, sizeof(double)));
  *one = 1.0;
  *minusOne = -1.0;

  // Workspace for cuSOLVER
  size_t workspaceInBytesOnDevice, workspaceInBytesOnHost;
  checkCudaErrors(hipsolverDnXpotrf_bufferSize(
    cusolverDnHandle, cusolverDnParams, HIPBLAS_FILL_MODE_LOWER, B,
    HIP_R_64F, d_tiles[0], B, HIP_R_64F,
    &workspaceInBytesOnDevice, &workspaceInBytesOnHost
  ));
  
  void *h_workspace, *d_workspace;
  int *d_info;
  checkCudaErrors(hipHostMalloc(&h_workspace, workspaceInBytesOnHost));
  checkCudaErrors(hipMalloc(&d_workspace, workspaceInBytesOnDevice));
  checkCudaErrors(hipMallocManaged(&d_info, sizeof(int)));

  // =========================================================================
  // PHASE 2: BUILD NAIVE CUDA GRAPH USING TASKMANAGER_V2
  // =========================================================================
  fmt::print("\n--- PHASE 2: Build Naive CUDA Graph ---\n");
  
  hipStream_t s;
  checkCudaErrors(hipStreamCreate(&s));
  checkCudaErrors(hipsolverSetStream(cusolverDnHandle, s));
  checkCudaErrors(hipblasSetStream(cublasHandle, s));

  // Initialize TaskManager_v2 for naive graph construction
  TaskManager_v2 tmanager_v2(true);

  // Register all tasks in the correct dependency order
  fmt::print("Registering tasks for Cholesky decomposition...\n");
  
  // Tiled Cholesky algorithm - register tasks sequentially
  for (int k = 0; k < T; k++) {
    // POTRF - Cholesky factorization of diagonal tile
    std::vector<void*> inputs = {static_cast<void*>(getMatrixBlock(k, k))};
    std::vector<void*> outputs = {static_cast<void*>(getMatrixBlock(k, k))};
    
    TaskId potrfTaskId = tmanager_v2.registerTask<std::function<void(hipStream_t, double*, void*, size_t, int*)>, double*, void*, size_t, int*>(
      [cusolverDnHandle, cusolverDnParams, workspaceInBytesOnHost](hipStream_t stream, double* matrixblock_k_k, void* d_workspace, size_t workspaceInBytesOnDevice, int* d_info) {
        checkCudaErrors(hipsolverSetStream(cusolverDnHandle, stream));
        checkCudaErrors(hipsolverDnXpotrf(
          cusolverDnHandle, cusolverDnParams, HIPBLAS_FILL_MODE_LOWER, *current_block_size,
          HIP_R_64F, matrixblock_k_k, *current_block_size, HIP_R_64F,
          d_workspace, workspaceInBytesOnDevice, nullptr, workspaceInBytesOnHost, d_info
        ));
      },
      inputs, outputs,
      TaskManager_v2::makeArgs(getMatrixBlock(k, k), d_workspace, workspaceInBytesOnDevice, d_info),
      "POTRF_task_" + std::to_string(k)
    );
    
    fmt::print("✓ Registered POTRF task {}\n", k);

    // TRSM - Triangular solve 
    for (int i = k + 1; i < T; i++) {
      inputs = {static_cast<void*>(getMatrixBlock(i, k)), static_cast<void*>(getMatrixBlock(k, k))};
      outputs = {static_cast<void*>(getMatrixBlock(i, k))};
      
      TaskId trsmTaskId = tmanager_v2.registerTask<std::function<void(hipStream_t, double*, double*)>, double*, double*>(
        [cublasHandle, one](hipStream_t stream, double* matrixblock_k_k, double* matrixblock_i_k) {
          checkCudaErrors(hipblasSetStream(cublasHandle, stream));
          checkCudaErrors(hipblasDtrsm(
            cublasHandle, HIPBLAS_SIDE_RIGHT, HIPBLAS_FILL_MODE_LOWER,
            HIPBLAS_OP_T, HIPBLAS_DIAG_NON_UNIT, *current_block_size, *current_block_size, one,
            matrixblock_k_k, *current_block_size, matrixblock_i_k, *current_block_size
          ));
        },
        inputs, outputs,
        TaskManager_v2::makeArgs(getMatrixBlock(k, k), getMatrixBlock(i, k)),
        "TRSM_task_" + std::to_string(i) + "_" + std::to_string(k)
      );
      
      fmt::print("✓ Registered TRSM task {}_{}\n", i, k);
    }

    // SYRK - Update diagonal tiles
    for (int i = k + 1; i < T; i++) {
      inputs = {static_cast<void*>(getMatrixBlock(i, i)), static_cast<void*>(getMatrixBlock(i, k))};
      outputs = {static_cast<void*>(getMatrixBlock(i, i))};
      
      TaskId syrkTaskId = tmanager_v2.registerTask<std::function<void(hipStream_t, double*, double*)>, double*, double*>(
        [cublasHandle, minusOne, one](hipStream_t stream, double* matrixblock_i_i, double* matrixblock_i_k) {
          checkCudaErrors(hipblasSetStream(cublasHandle, stream));
          checkCudaErrors(hipblasDsyrk(
            cublasHandle, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N,
            *current_block_size, *current_block_size, minusOne, matrixblock_i_k, *current_block_size, one, matrixblock_i_i, *current_block_size
          ));
        },
        inputs, outputs,
        TaskManager_v2::makeArgs(getMatrixBlock(i, i), getMatrixBlock(i, k)),
        "SYRK_task_" + std::to_string(i) + "_" + std::to_string(k)
      );
      
      fmt::print("✓ Registered SYRK task {}_{}\n", i, k);
    }

    // GEMM - Update off-diagonal tiles
    for (int i = k + 1; i < T; i++) {
      for (int j = k + 1; j < i; j++) {
        inputs = {static_cast<void*>(getMatrixBlock(i, j)), static_cast<void*>(getMatrixBlock(i, k)), static_cast<void*>(getMatrixBlock(j, k))};
        outputs = {static_cast<void*>(getMatrixBlock(i, j))};
        
        TaskId gemmTaskId = tmanager_v2.registerTask<std::function<void(hipStream_t, double*, double*, double*)>, double*, double*, double*>(
          [cublasHandle, minusOne, one](hipStream_t stream, double* matrixblock_i_j, double* matrixblock_i_k, double* matrixblock_j_k) {
            checkCudaErrors(hipblasSetStream(cublasHandle, stream));
            checkCudaErrors(hipblasDgemm(
              cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_T,
              *current_block_size, *current_block_size, *current_block_size, minusOne, matrixblock_i_k, *current_block_size, matrixblock_j_k, *current_block_size, one, matrixblock_i_j, *current_block_size
            ));
          },
          inputs, outputs,
          TaskManager_v2::makeArgs(getMatrixBlock(i, j), getMatrixBlock(i, k), getMatrixBlock(j, k)),
          "GEMM_task_" + std::to_string(i) + "_" + std::to_string(j) + "_" + std::to_string(k)
        );
        
        fmt::print("✓ Registered GEMM task {}_{}_{}\n", i, j, k);
      }
    }
  }

  fmt::print("✅ Task registration completed! Total tasks: {}\n", tmanager_v2.taskCount());
  
  // Generate naive CUDA graph from registered tasks
  fmt::print("Generating naive CUDA graph from task sequence...\n");
  hipGraph_t graph = tmanager_v2.generateNaiveGraph(s);
  
  // Get graph stats
  size_t numNodes;
  checkCudaErrors(hipGraphGetNodes(graph, nullptr, &numNodes));
  fmt::print("📊 Generated graph contains {} nodes\n", numNodes);

  // =========================================================================
  // PHASE 3: OPTIMIZE MEMORY ALLOCATION WITH NAIVE GRAPH
  // =========================================================================
  fmt::print("\n--- PHASE 3: Memory Optimization with Naive Graph ---\n");
  
  double initialPeakMemory = memManager.GetMemoryManagedSizeInMB();
  fmt::print("Initial peak memory: {:.2f} MB\n", initialPeakMemory);
  
  // Initialize data before optimization 
  initializeDeviceData(h_matrix, d_tiles);
  
  // Profile and optimize the naive graph
  fmt::print("Profiling and optimizing naive CUDA graph...\n");
  auto optimizedGraph = profileAndOptimize(graph);
  
  fmt::print("Original peak memory usage (MiB): {:.2f}\n", optimizedGraph.originalMemoryUsage);
  fmt::print("Optimized peak memory usage (MiB): {:.2f}\n", optimizedGraph.anticipatedPeakMemoryUsage);
  fmt::print("Memory reduction: {:.2f} MiB ({:.1f}%)\n", 
             optimizedGraph.originalMemoryUsage - optimizedGraph.anticipatedPeakMemoryUsage,
             ((optimizedGraph.originalMemoryUsage - optimizedGraph.anticipatedPeakMemoryUsage) / optimizedGraph.originalMemoryUsage) * 100);

  // Move all data to storage before execution (required for optimized execution)
  fmt::print("Moving all data to storage for optimized execution...\n");
  memManager.offloadAllManagedMemoryToStorage();
  fmt::print("✅ All data moved to CPU/storage\n");
  
  // =========================================================================
  // PHASE 4: EXECUTE WITH OPTIMIZED NAIVE GRAPH
  // =========================================================================
  fmt::print("\n--- PHASE 4: Execute with Optimized Naive Graph ---\n");
  
  // Reinitialize data after optimization
  initializeDeviceData(h_matrix, d_tiles);
  
  // Get current GPU memory info
  size_t free_mem, total_mem;
  checkCudaErrors(hipMemGetInfo(&free_mem, &total_mem));
  fmt::print("GPU Memory - Total: {:.2f} MB, Free: {:.2f} MB\n", 
             (double)total_mem / (1024.0 * 1024.0), (double)free_mem / (1024.0 * 1024.0));
  
  // Start peak memory monitoring
  fmt::print("🔍 Starting continuous GPU memory monitoring during execution...\n");
  PeakMemoryUsageProfiler peakProfiler(10); // Sample every 10ms
  peakProfiler.start();
  
  // Run the optimized naive graph
  float runningTime;
  executeOptimizedGraph(
    optimizedGraph,
    [&tmanager_v2](int taskId, std::map<void*, void*> addressMapping, hipStream_t stream) {
      tmanager_v2.execute(taskId, stream);
    },
    runningTime,
    memManager
  );
  
  // Get peak memory usage
  size_t peakMemoryBytes = peakProfiler.end();
  double peakMemoryMB = (double)peakMemoryBytes / (1024.0 * 1024.0);
  
  fmt::print("✅ Optimized naive graph execution completed!\n");
  fmt::print("Execution time: {:.3f} ms\n", runningTime * 1000.0f);
  fmt::print("📊 Peak GPU memory usage during execution: {:.2f} MB\n", peakMemoryMB);
  
  // =========================================================================
  // PHASE 5: VERIFY RESULTS
  // =========================================================================
  fmt::print("\n--- PHASE 5: Verify Results ---\n");
  
  // Always do partial verification first
  bool partialResult = verifyCholeskyDecompositionPartially(h_matrix, d_tiles);
  
  bool fullResult = true;
  const size_t matrix_size_mb = (N * N * sizeof(double)) / (1024 * 1024);
  
  // For smaller matrices, also do full cuSOLVER comparison
  if (matrix_size_mb < 1024) { // Less than 1GB
    fmt::print("\n--- PHASE 5.1: Comparison with cuSOLVER Direct Cholesky ---\n");
    
    // Allocate memory for direct Cholesky
    double* d_A_direct;
    checkCudaErrors(hipMalloc(&d_A_direct, N * N * sizeof(double)));
    checkCudaErrors(hipMemcpy(d_A_direct, h_matrix, N * N * sizeof(double), hipMemcpyHostToDevice));
    
    // Create new cuSOLVER handle for direct computation
    hipsolverHandle_t directSolverHandle;
    checkCudaErrors(hipsolverDnCreate(&directSolverHandle));
    
    // Query workspace size for direct Cholesky
    int directWorkspaceSize = 0;
    checkCudaErrors(hipsolverDnDpotrf_bufferSize(
      directSolverHandle, HIPBLAS_FILL_MODE_LOWER, N, d_A_direct, N, &directWorkspaceSize));
    
    double* d_directWorkspace;
    checkCudaErrors(hipMalloc(&d_directWorkspace, directWorkspaceSize * sizeof(double)));
    
    int* d_directInfo;
    checkCudaErrors(hipMalloc(&d_directInfo, sizeof(int)));
    
    // Perform direct Cholesky decomposition
    fmt::print("Running cuSOLVER direct Cholesky on {}x{} matrix ({:.1f}MB)...\n", N, N, (double)matrix_size_mb);
    checkCudaErrors(hipsolverDnDpotrf(
      directSolverHandle, HIPBLAS_FILL_MODE_LOWER, N, d_A_direct, N, 
      d_directWorkspace, directWorkspaceSize, d_directInfo));
    
    // Check if direct decomposition succeeded
    int h_directInfo;
    checkCudaErrors(hipMemcpy(&h_directInfo, d_directInfo, sizeof(int), hipMemcpyDeviceToHost));
    
    if (h_directInfo != 0) {
      fmt::print("ERROR: Direct Cholesky failed with info = {}\n", h_directInfo);
      fullResult = false;
    } else {
      fmt::print("✅ Direct Cholesky succeeded\n");
      
      // Copy direct result to host
      double* h_L_direct;
      checkCudaErrors(hipHostMalloc(&h_L_direct, N * N * sizeof(double)));
      checkCudaErrors(hipMemcpy(h_L_direct, d_A_direct, N * N * sizeof(double), hipMemcpyDeviceToHost));
      
      // Copy tiled result to contiguous memory for comparison
      double* h_L_tiled;
      checkCudaErrors(hipHostMalloc(&h_L_tiled, N * N * sizeof(double)));
      
      // Reconstruct tiled result into contiguous array
      for (size_t i = 0; i < T; i++) {
        for (size_t j = 0; j <= i; j++) {
          size_t tile_idx = i * T + j;
          double* h_tile_data;
          checkCudaErrors(hipHostMalloc(&h_tile_data, B * B * sizeof(double)));
          
          // Copy tile data using memory manager
          memManager.copyManagedArrayToHost(d_tiles[tile_idx], h_tile_data, B * B * sizeof(double));
          
          // Copy to contiguous array
          for (size_t bi = 0; bi < B; bi++) {
            for (size_t bj = 0; bj < B; bj++) {
              size_t global_i = i * B + bi;
              size_t global_j = j * B + bj;
              if (global_i >= global_j && global_i < N && global_j < N) {
                h_L_tiled[global_i * N + global_j] = h_tile_data[bi * B + bj];
              }
            }
          }
          checkCudaErrors(hipHostFree(h_tile_data));
        }
      }
      
      // Compare results
      const double TOLERANCE = 1e-10;
      size_t diff_count = 0;
      double max_diff = 0.0;
      
      for (size_t i = 0; i < N; i++) {
        for (size_t j = 0; j <= i; j++) {
          double diff = std::abs(h_L_tiled[i * N + j] - h_L_direct[i * N + j]);
          if (diff > TOLERANCE) {
            diff_count++;
            max_diff = std::max(max_diff, diff);
          }
        }
      }
      
      if (diff_count == 0) {
        fmt::print("✅ Tiled and direct Cholesky results MATCH perfectly!\n");
      } else {
        fmt::print("❌ Tiled and direct Cholesky DIFFER: {} differences, max diff = {:.2e}\n", 
                   diff_count, max_diff);
        fullResult = false;
      }
      
      checkCudaErrors(hipHostFree(h_L_direct));
      checkCudaErrors(hipHostFree(h_L_tiled));
    }
    
    // Cleanup direct Cholesky resources
    checkCudaErrors(hipFree(d_A_direct));
    checkCudaErrors(hipFree(d_directWorkspace));
    checkCudaErrors(hipFree(d_directInfo));
    checkCudaErrors(hipsolverDnDestroy(directSolverHandle));
  } else {
    fmt::print("Matrix too large ({:.1f}MB) for full cuSOLVER comparison, using partial validation only\n", (double)matrix_size_mb);
  }
  
  bool result = partialResult && fullResult;
  
  // =========================================================================
  // PHASE 6: CLEANUP
  // =========================================================================
  fmt::print("\n--- PHASE 6: Cleanup ---\n");
  
  checkCudaErrors(hipGraphDestroy(graph));
  checkCudaErrors(hipStreamDestroy(s));
  checkCudaErrors(hipsolverDnDestroyParams(cusolverDnParams));
  checkCudaErrors(hipsolverDnDestroy(cusolverDnHandle));
  checkCudaErrors(hipblasDestroy(cublasHandle));
  
  checkCudaErrors(hipFree(one));
  checkCudaErrors(hipFree(minusOne));
  checkCudaErrors(hipFree(d_workspace));
  checkCudaErrors(hipFree(d_info));
  checkCudaErrors(hipHostFree(h_workspace));
  
  // Free managed memory properly using MemoryManager
  for (auto d_tile : d_tiles) {
    memManager.freeManagedMemory(d_tile);
  }
  checkCudaErrors(hipHostFree(h_matrix));
  
  fmt::print("\nFinal result: {}\n", result ? "SUCCESS" : "FAILED");
}

int main(int argc, char *argv[]) {
  // Simple command line parsing or use defaults
  N = (argc > 1) ? std::atoi(argv[1]) : 1024;  // Matrix dimension
  T = (argc > 2) ? std::atoi(argv[2]) : 4;     // Number of tiles
  
  // Calculate block size
  B = N / T;
  
  // Validation
  if (N % T != 0) {
    fmt::print("ERROR: Matrix dimension must be divisible by tile count\n");
    return -1;
  }
  
  ConfigurationManager::exportDefaultConfiguration();
  ConfigurationManager::loadConfiguration("config.json");
  
  fmt::print("Configuration: N={}, T={}, B={}\n", N, T, B);
  
  // Run the naive graph version
  tiledCholeskyNaiveGraph();
  
  return 0;
}