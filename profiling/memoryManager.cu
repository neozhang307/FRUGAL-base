#include "memoryManager.hpp"
#include "../utilities/logger.hpp"

namespace memopt {

// Singleton instance access
MemoryManager& MemoryManager::getInstance() {
  static MemoryManager instance;
  return instance;
}

// Destructor
MemoryManager::~MemoryManager() {
  // Release any remaining dummy memory allocations
  if (!dummyAllocations.empty()) {
    fprintf(stderr, "[MEMORY-INFO] Cleaning up %zu dummy allocations in MemoryManager destructor\n", 
            dummyAllocations.size());
    releaseDummyMemory();
  }
}

// Const accessor methods
const std::vector<void*>& MemoryManager::getManagedAddresses() const { 
  return managedMemoryAddresses; 
}

const std::map<void*, ArrayId>& MemoryManager::getAddressToIndexMap() const { 
  return managedMemoryAddressToIndexMap; 
}

// const std::map<void*, size_t>& MemoryManager::getAddressToSizeMap() const { 
//   return managedMemoryAddressToSizeMap; 
// }

double MemoryManager::GetMemoryManagedSizeInMB() const {
  size_t totalSizeInBytes = 0;
  
  // Sum the sizes of all memory arrays in the memoryArrayInfos vector
  for (const auto& info : memoryArrayInfos) {
    totalSizeInBytes += info.size;
  }
  
  // Convert bytes to MB (1MB = 1,048,576 bytes)
  return static_cast<double>(totalSizeInBytes) / (1024.0 * 1024.0);
}

const std::set<void*>& MemoryManager::getApplicationInputs() const { 
  return applicationInputs; 
}

const std::set<void*>& MemoryManager::getApplicationOutputs() const { 
  return applicationOutputs; 
}


const std::map<void*, void*>& MemoryManager::getCurrentAddressMap() const { 
  // Use a static thread-local map to avoid modifying the shared map
  static thread_local std::map<void*, void*> localMap;
  localMap.clear();
  
  // Populate with active mappings from memoryArrayInfos
  for (size_t i = 0; i < memoryArrayInfos.size(); ++i) {
    const auto& info = memoryArrayInfos[i];
    // Only include entries that have a valid deviceAddress
    if (info.managedMemoryAddress != nullptr && info.deviceAddress != nullptr) {
      localMap[info.managedMemoryAddress] = info.deviceAddress;
    }
  }
  
  return localMap; 
}

// const std::map<void*, void*>& MemoryManager::getDeviceToHostArrayMap() const { 
//   return managedDeviceArrayToHostArrayMap; 
// }

// Editable accessor methods
std::vector<void*>& MemoryManager::getEditableManagedAddresses() { 
  return managedMemoryAddresses; 
}

std::map<void*, ArrayId>& MemoryManager::getEditableAddressToIndexMap() { 
  return managedMemoryAddressToIndexMap; 
}

// std::map<void*, size_t>& MemoryManager::getEditableAddressToSizeMap() { 
//   return managedMemoryAddressToSizeMap; 
// }

std::set<void*>& MemoryManager::getEditableApplicationInputs() { 
  return applicationInputs; 
}

std::set<void*>& MemoryManager::getEditableApplicationOutputs() { 
  return applicationOutputs; 
}


std::map<void*, void*>& MemoryManager::getEditableCurrentAddressMap() { 
  // Use a static thread-local map to avoid modifying the shared map directly
  static thread_local std::map<void*, void*> localMap;
  localMap.clear();
  
  // Populate with active mappings from memoryArrayInfos
  for (size_t i = 0; i < memoryArrayInfos.size(); ++i) {
    const auto& info = memoryArrayInfos[i];
    // Only include entries that have a valid deviceAddress
    if (info.managedMemoryAddress != nullptr && info.deviceAddress != nullptr) {
      localMap[info.managedMemoryAddress] = info.deviceAddress;
    }
  }
  
  // Return a reference to the map for modifications
  // Note: The callers will typically use this map to make changes, but those changes
  // won't be synchronized back to memoryArrayInfos until updateCurrentMapping or
  // removeCurrentMapping methods are called. This is consistent with existing behavior.
  return localMap; 
}

// std::map<void*, void*>& MemoryManager::getEditableDeviceToHostArrayMap() { 
//   return managedDeviceArrayToHostArrayMap; 
// }

// Memory information methods are now in memoryManager_v2.cu

void MemoryManager::prefetchAllDataToDeviceAsync(
    const std::vector<ArrayId>& arrayIds,
    hipMemcpyKind memcpyKind,
    hipStream_t stream) {
  for (auto arrayId : arrayIds) {
    void* originalPtr = managedMemoryAddresses[arrayId];
    void* storagePtr = memoryArrayInfos[arrayId].storageAddress;// managedDeviceArrayToHostArrayMap.at(originalPtr);
    size_t size = getSize(originalPtr);
    
    // Allocate on device and copy data from storage
    void* devicePtr;
    checkCudaErrors(hipMallocAsync(&devicePtr, size, stream));
    checkCudaErrors(hipMemcpyAsync(
      devicePtr, 
      storagePtr, 
      size, 
      memcpyKind, 
      stream
    ));
    
    // Update the current mapping
    // managedMemoryAddressToAssignedMap[originalPtr] = devicePtr;
    memoryArrayInfos[arrayId].deviceAddress=devicePtr;
  }
}

// Memory allocation and data transfer methods moved to memoryManager_v2.cu

void MemoryManager::offloadAllManagedMemoryToStorage(std::map<void*, void*>& storageMap) {
  // Only print verbose debug information if enabled
  bool verbose = ConfigurationManager::getConfig().execution.enableVerboseOutput;
  
  if (verbose) {
    fprintf(stderr, "[DEBUG-OFFLOAD-MAP] Starting offloadAllManagedMemoryToStorage with external map, address count: %zu\n", 
            managedMemoryAddresses.size());
  }
  
  // Ensure the storage map starts empty
  storageMap.clear();
  if (verbose) {
    fprintf(stderr, "[DEBUG-OFFLOAD-MAP] Cleared external storageMap\n");
  }
  
  releaseStoragePointers();
  if (verbose) {
    fprintf(stderr, "[DEBUG-OFFLOAD-MAP] Called releaseStoragePointers() to reset memoryArrayInfos\n");
  }
  
  // Move each managed memory address to storage
  for (size_t i = 0; i < managedMemoryAddresses.size(); i++) {
    void* ptr = managedMemoryAddresses[i];
    if (verbose) {
      fprintf(stderr, "[DEBUG-OFFLOAD-MAP] Processing address %zu/%zu: %p\n", 
              i+1, managedMemoryAddresses.size(), ptr);
    }
    
    // Would automatically update MemoryArrayInfo
    void* storagePtr = offloadToStorage(ptr, storageConfig.storageDeviceId, 
                                       storageConfig.useNvlink, storageMap);
                                       
    // Verify both data structures have the same storage pointer
    if (verbose) {
      fprintf(stderr, "[DEBUG-OFFLOAD-MAP] After offload: storageMap[%p] = %p\n", 
              ptr, storageMap[ptr]);
      
      if (i < memoryArrayInfos.size()) {
        fprintf(stderr, "[DEBUG-OFFLOAD-MAP] After offload: memoryArrayInfos[%zu].storageAddress = %p\n", 
                i, memoryArrayInfos[i].storageAddress);
                
        if (memoryArrayInfos[i].storageAddress != storageMap[ptr]) {
          fprintf(stderr, "[DEBUG-OFFLOAD-MAP] ERROR: Storage pointers don't match for address %p! (%p vs %p)\n", 
                  ptr, memoryArrayInfos[i].storageAddress, storageMap[ptr]);
        }
      } else {
        fprintf(stderr, "[DEBUG-OFFLOAD-MAP] ERROR: No memoryArrayInfos entry for index %zu\n", i);
      }
    }
  }
  
  // Also update internal map
  // managedDeviceArrayToHostArrayMap = storageMap;
  // fprintf(stderr, "[DEBUG-OFFLOAD-MAP] Updated internal managedDeviceArrayToHostArrayMap from external map\n");
  
  // Print a summary of both data structures
  if (verbose) {
    fprintf(stderr, "[DEBUG-OFFLOAD-MAP] Summary - storageMap entries: %zu\n", storageMap.size());
    // fprintf(stderr, "[DEBUG-OFFLOAD-MAP] Summary - managedDeviceArrayToHostArrayMap entries: %zu\n", 
            // managedDeviceArrayToHostArrayMap.size());
    fprintf(stderr, "[DEBUG-OFFLOAD-MAP] Summary - memoryArrayInfos entries: %zu\n", memoryArrayInfos.size());
  }
  
  // Switch back to main GPU
  checkCudaErrors(hipSetDevice(storageConfig.mainDeviceId));
  checkCudaErrors(hipDeviceSynchronize());
  
  if (verbose) {
    fprintf(stderr, "[DEBUG-OFFLOAD-MAP] Completed offloadAllManagedMemoryToStorage with external map\n");
  }
}
void* MemoryManager::offloadToStorage(
    void* managedMemoryAddress, 
    int storageDeviceId, 
    bool useNvlink, 
    std::map<void*, void*>& storageMap,
    hipStream_t stream) 
{
  // Only print debug information if verbose output is enabled
  bool verbose = ConfigurationManager::getConfig().execution.enableVerboseOutput;
  
  if (verbose) {
    fprintf(stderr, "[DEBUG-OFFLOAD] Starting offload for address %p\n", managedMemoryAddress);
  }
  
  // Allocate in storage
  void* storagePtr = allocateInStorage(managedMemoryAddress, storageDeviceId, useNvlink);
  if (verbose) {
    fprintf(stderr, "[DEBUG-OFFLOAD] Allocated storage %p for address %p\n", storagePtr, managedMemoryAddress);
  }
  
  // Copy data from device to storage
  copyMemoryDeviceToStorage(managedMemoryAddress, storagePtr, hipMemcpyDefault);
  if (verbose) {
    fprintf(stderr, "[DEBUG-OFFLOAD] Data transferred from %p to %p\n", managedMemoryAddress, storagePtr);
  }
  
  // Update mapping
  storageMap[managedMemoryAddress] = storagePtr;
  if (verbose) {
    fprintf(stderr, "[DEBUG-OFFLOAD] Updated storageMap: %p -> %p\n", managedMemoryAddress, storagePtr);
  }
  
  // Update MemoryArrayInfo structure
  ArrayId arrayId = getArrayId(managedMemoryAddress);
  if (verbose) {
    fprintf(stderr, "[DEBUG-OFFLOAD] getArrayId returned %d for address %p\n", arrayId, managedMemoryAddress);
  }
  
  if (arrayId >= 0 && arrayId < memoryArrayInfos.size()) {
    memoryArrayInfos[arrayId].storageAddress = storagePtr;
    if (verbose) {
      fprintf(stderr, "[DEBUG-OFFLOAD] Updated memoryArrayInfos[%d].storageAddress = %p\n", 
              arrayId, storagePtr);
    }
  } else {
    if (verbose) {
      fprintf(stderr, "[DEBUG-OFFLOAD] WARNING: Could not update memoryArrayInfos for address %p, arrayId=%d\n", 
              managedMemoryAddress, arrayId);
    }
  }
  
  // Verify storageAddress was set
  if (arrayId >= 0 && arrayId < memoryArrayInfos.size()) {
    if (verbose) {
      fprintf(stderr, "[DEBUG-OFFLOAD] Verification: memoryArrayInfos[%d].storageAddress = %p\n", 
              arrayId, memoryArrayInfos[arrayId].storageAddress);
    }
  }
  
  // Free original device memory
  checkCudaErrors(hipFreeAsync(memoryArrayInfos[arrayId].deviceAddress,stream));
  checkCudaErrors(hipStreamSynchronize(stream));
  if (verbose) {
    fprintf(stderr, "[DEBUG-OFFLOAD] Freed original device memory %p of %p\n", 
            memoryArrayInfos[arrayId].deviceAddress, managedMemoryAddress);
  }
  memoryArrayInfos[arrayId].deviceAddress=nullptr;
  return storagePtr;
}

void MemoryManager::offloadAllManagedMemoryToStorage(
    int mainDeviceId, 
    int storageDeviceId, 
    bool useNvlink,
    std::map<void*, void*>& storageMap) {
  // Configure storage with the provided parameters
  configureStorage(mainDeviceId, storageDeviceId, useNvlink);
  
  // Call the simpler version that uses the configured parameters and external map
  offloadAllManagedMemoryToStorage(storageMap);
}

void MemoryManager::offloadAllManagedMemoryToStorage(
    int mainDeviceId, 
    int storageDeviceId, 
    bool useNvlink) {
  // Configure storage with the provided parameters
  configureStorage(mainDeviceId, storageDeviceId, useNvlink);
  
  // Call the version with map parameter, using internal map
  std::map<void*, void*> storageMap;
  offloadAllManagedMemoryToStorage(storageMap);
}


// Asynchronous operations
void MemoryManager::offloadDataAsync(
    void* originalPtr, 
    std::map<void*, void*>& storageMap, 
    std::map<void*, void*>& currentMap,
    hipStream_t stream) {
  void* devicePtr = currentMap[originalPtr];
  void* storagePtr = storageMap[originalPtr];
  size_t size = getSize(originalPtr);
  
  // Copy data from device to storage and free device memory
  checkCudaErrors(hipMemcpyAsync(storagePtr, devicePtr, size, storageConfig.offloadMemcpyKind, stream));
  checkCudaErrors(hipFreeAsync(devicePtr, stream));
  
  // Remove from current map
  currentMap.erase(originalPtr);
}

void MemoryManager::offloadDataAsync(
    void* originalPtr, 
    std::map<void*, void*>& storageMap, 
    std::map<void*, void*>& currentMap, 
    hipMemcpyKind offloadMemcpyKind,
    hipStream_t stream) {
  void* devicePtr = currentMap[originalPtr];
  void* storagePtr = storageMap[originalPtr];
  size_t size = getSize(originalPtr);
  
  // Copy data from device to storage and free device memory
  checkCudaErrors(hipMemcpyAsync(storagePtr, devicePtr, size, offloadMemcpyKind, stream));
  checkCudaErrors(hipFreeAsync(devicePtr, stream));
  
  // Remove from current map
  currentMap.erase(originalPtr);
}

// Array ID utility methods moved to memoryManager_v2.cu

// Implementation of standalone function
void updateManagedMemoryAddress(const std::map<void *, void *> oldAddressToNewAddressMap) {
  auto& memManager = MemoryManager::getInstance();
  
  // Save the old size mapping for reference
  // auto oldManagedMemoryAddressToSizeMap = memManager.getAddressToSizeMap();

  // Clear existing mappings to rebuild them
  // memManager.getEditableAddressToSizeMap().clear();
  // memManager.getEditableAddressToIndexMap().clear();

  // Update all addresses and rebuild mappings
  for (int i = 0; i < memManager.getManagedAddresses().size(); i++) {
    // Ensure every old address has a corresponding new address
    assert(oldAddressToNewAddressMap.count(memManager.getManagedAddresses()[i]) == 1);

    const auto newAddr = oldAddressToNewAddressMap.at(memManager.getManagedAddresses()[i]);
    const auto oldAddr = memManager.getManagedAddresses()[i];

    // Update address in the main list
    // memManager.getEditableManagedAddresses()[i] = newAddr;
    
    // Transfer the size information to the new address (legacy map for backward compatibility)
    // memManager.getEditableAddressToSizeMap()[newAddr] = oldManagedMemoryAddressToSizeMap.at(oldAddr);
    
    // Maintain the same array ID
    // memManager.getEditableAddressToIndexMap()[newAddr] = i;
    
    // Update the MemoryArrayInfo structure
    if (i < memManager.memoryArrayInfos.size()) {
      // memManager.memoryArrayInfos[i].managedMemoryAddress = newAddr;
      memManager.memoryArrayInfos[i].deviceAddress = newAddr;
    }

    // // Update application input registry if this was an input
    // auto& appInputs = memManager.getEditableApplicationInputs();
    // if (memManager.getApplicationInputs().count(oldAddr) > 0) {
    //   appInputs.erase(oldAddr);
    //   appInputs.insert(newAddr);
    // }
    
    // // Update application output registry if this was an output
    // auto& appOutputs = memManager.getEditableApplicationOutputs();
    // if (memManager.getApplicationOutputs().count(oldAddr) > 0) {
    //   appOutputs.erase(oldAddr);
    //   appOutputs.insert(newAddr);
    // }
  }
}

// Functions moved to memoryManager_v2.cu with GB units
// consumeGPUMemory
// getConsumedGPUMemory
// releaseDummyMemory
// claimNecessaryMemory

// Stage management APIs for incremental profiling
bool MemoryManager::prepareStage(size_t stageId, const std::set<void*>& requiredArrays, hipStream_t stream) {
  LOG_TRACE_WITH_INFO("Preparing stage %zu with %zu required arrays", stageId, requiredArrays.size());
  
  // For each required array, fetch from storage to GPU if needed
  for (void* addr : requiredArrays) {
    ArrayId arrayId = getArrayId(addr);
    if (arrayId < 0 || arrayId >= memoryArrayInfos.size()) {
      LOG_TRACE_WITH_INFO("Warning: Array %p not found in registry", addr);
      continue;
    }
    
    auto& arrayInfo = memoryArrayInfos[arrayId];
    
    // If data is in storage, fetch it to GPU
    if (arrayInfo.storageAddress != nullptr && arrayInfo.deviceAddress == nullptr) {
      // Allocate GPU memory if not already allocated
      if (arrayInfo.deviceAddress == nullptr) {
        checkCudaErrors(hipMalloc(&arrayInfo.deviceAddress, arrayInfo.size));
      }
      
      // Copy from storage to device
      checkCudaErrors(hipMemcpyAsync(arrayInfo.deviceAddress, arrayInfo.storageAddress, 
                                       arrayInfo.size, storageConfig.prefetchMemcpyKind, stream));
      
      LOG_TRACE_WITH_INFO("Fetched array %p (%.2f MB) from storage to GPU", 
                          addr, arrayInfo.size / (1024.0 * 1024.0));
    }
  }
  
  if (stream) {
    checkCudaErrors(hipStreamSynchronize(stream));
  }
  
  return true;
}

bool MemoryManager::finalizeStage(size_t stageId, const std::set<void*>& modifiedArrays, 
                                  const std::set<void*>& arraysToKeep, hipStream_t stream) {
  LOG_TRACE_WITH_INFO("Finalizing stage %zu with %zu modified arrays", stageId, modifiedArrays.size());
  
  // For each modified array, offload from GPU to storage
  for (void* addr : modifiedArrays) {
    // Skip if we need to keep this array for next stage
    if (arraysToKeep.count(addr) > 0) {
      continue;
    }
    
    ArrayId arrayId = getArrayId(addr);
    if (arrayId < 0 || arrayId >= memoryArrayInfos.size()) {
      continue;
    }
    
    auto& arrayInfo = memoryArrayInfos[arrayId];
    
    // If data is on GPU, offload it to storage
    if (arrayInfo.deviceAddress != nullptr) {
      // Allocate storage if not already allocated
      if (arrayInfo.storageAddress == nullptr) {
        arrayInfo.storageAddress = allocateInStorage(addr, storageConfig.storageDeviceId, 
                                                     storageConfig.useNvlink);
      }
      
      // Copy from device to storage
      if (arrayInfo.storageAddress != nullptr) {
        checkCudaErrors(hipMemcpyAsync(arrayInfo.storageAddress, arrayInfo.deviceAddress,
                                         arrayInfo.size, storageConfig.offloadMemcpyKind, stream));
        
        LOG_TRACE_WITH_INFO("Offloaded array %p (%.2f MB) from GPU to storage", 
                            addr, arrayInfo.size / (1024.0 * 1024.0));
      }
      
      // Free GPU memory to save space
      if (arraysToKeep.count(addr) == 0) {
        checkCudaErrors(hipFree(arrayInfo.deviceAddress));
        arrayInfo.deviceAddress = nullptr;
      }
    }
  }
  
  if (stream) {
    checkCudaErrors(hipStreamSynchronize(stream));
  }
  
  return true;
}

bool MemoryManager::checkStageMemoryRequirement(const std::set<void*>& requiredArrays) const {
  size_t totalRequired = 0;
  
  for (void* addr : requiredArrays) {
    ArrayId arrayId = getArrayId(addr);
    if (arrayId >= 0 && arrayId < memoryArrayInfos.size()) {
      totalRequired += memoryArrayInfos[arrayId].size;
    }
  }
  
  size_t freeMem, totalMem;
  checkCudaErrors(hipMemGetInfo(&freeMem, &totalMem));
  
  const double SAFETY_FACTOR = 0.9;
  bool fits = totalRequired <= (freeMem * SAFETY_FACTOR);
  
  LOG_TRACE_WITH_INFO("Stage requires %.2f MB, GPU has %.2f MB free - %s",
                      totalRequired / (1024.0 * 1024.0),
                      freeMem / (1024.0 * 1024.0),
                      fits ? "FITS" : "DOES NOT FIT");
  
  return fits;
}

} // namespace memopt