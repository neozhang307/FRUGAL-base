#include "hip/hip_runtime.h"
#include <cassert>
#include <memory>
#include <queue>

#include "../profiling/memoryManager.hpp"
#include "../profiling/peakMemoryUsageProfiler.hpp"
#include "../utilities/configurationManager.hpp"
#include "../utilities/cudaGraphUtilities.hpp"
#include "../utilities/cudaUtilities.hpp"
#include "../utilities/utilities.hpp"
#include "../utilities/logger.hpp"
#include "executor.hpp"

namespace memopt {

/*
 * executeOptimizedGraph - Executes a CUDA graph with memory optimization
 *
 * This function executes a computation graph that has been optimized to reduce memory usage
 * by dynamically managing data transfers between the main GPU and storage (host memory or secondary GPU).
 * It enables processing of workloads larger than would fit in GPU memory alone.
 * 
 * This implementation uses MemoryManager's internal storage to track device-to-host mappings.
 *
 * Parameters:
 * - optimizedGraph: The optimized computation graph to execute
 * - executeRandomTask: Callback to execute specific computation tasks
 * - runningTime: Output parameter to store the execution time
 * - memManager: Reference to the MemoryManager instance to use
 */
void Executor::executeOptimizedGraph(
  OptimizationOutput &optimizedGraph,
  ExecuteRandomTask executeRandomTask,
  float &runningTime,
  MemoryManager &memManager
) {
  LOG_TRACE_WITH_INFO("Initialize");

  // Create CUDA resources 
  hipGraph_t graph;
  checkCudaErrors(hipGraphCreate(&graph, 0));

  hipStream_t stream;
  checkCudaErrors(hipStreamCreate(&stream));

  auto optimizedCudaGraphCreator = std::make_unique<OptimizedCudaGraphCreator>(stream, graph);

  //----------------------------------------------------------------------
  // STEP 1: Prepare the graph for topological traversal using Kahn's algorithm
  //----------------------------------------------------------------------
  
  // Calculate in-degrees for each node in the optimized graph
  std::map<int, int> inDegrees;
  for (auto &[u, outEdges] : optimizedGraph.edges) {
    for (auto &v : outEdges) {
      inDegrees[v] += 1;
    }
  }

  // Find root nodes (nodes with no dependencies)
  std::queue<int> nodesToExecute;
  std::vector<int> rootNodes;
  for (auto &u : optimizedGraph.nodes) {
    if (inDegrees[u] == 0) {
      nodesToExecute.push(u);
      rootNodes.push_back(u);
    }
  }

  //----------------------------------------------------------------------
  // STEP 2: Configure the device and memory settings
  //----------------------------------------------------------------------
  
  // Set up device configuration for data movement
  int mainDeviceId = ConfigurationManager::getConfig().execution.mainDeviceId;
  int storageDeviceId = hipCpuDeviceId;  // Default: use host memory as storage
  hipMemcpyKind prefetchMemcpyKind = hipMemcpyHostToDevice;
  hipMemcpyKind offloadMemcpyKind = hipMemcpyDeviceToHost;

  // If NVLink is available, use a second GPU as storage instead of host memory
  if (ConfigurationManager::getConfig().execution.useNvlink) {
    storageDeviceId = ConfigurationManager::getConfig().execution.storageDeviceId;
    prefetchMemcpyKind = hipMemcpyDeviceToDevice;
    offloadMemcpyKind = hipMemcpyDeviceToDevice;
    enablePeerAccessForNvlink(ConfigurationManager::getConfig().execution.mainDeviceId, ConfigurationManager::getConfig().execution.storageDeviceId);
  }
  memManager.configureStorage(ConfigurationManager::getConfig().execution.mainDeviceId,
                            ConfigurationManager::getConfig().execution.storageDeviceId,
                            ConfigurationManager::getConfig().execution.useNvlink);
  //----------------------------------------------------------------------
  // STEP 3: Initialize managed data distribution
  //----------------------------------------------------------------------
  
  LOG_TRACE_WITH_INFO("Initialize managed data distribution");

  // Configure MemoryManager storage parameters
  memManager.configureStorage(mainDeviceId, storageDeviceId, ConfigurationManager::getConfig().execution.useNvlink);
  
  // Move all managed data to storage (host or secondary GPU) using internal storage
  memManager.moveAllManagedMemoryToStorage();
  
  // Switch back to main GPU
  checkCudaErrors(hipSetDevice(mainDeviceId));
  checkCudaErrors(hipDeviceSynchronize());

  //----------------------------------------------------------------------
  // STEP 4: Initialize data that needs to be on the device at the start
  //----------------------------------------------------------------------
  
  // Track memory addresses that have been updated (device copies)
  memManager.clearCurrentMappings();
  
  // Create a subgraph for initial data prefetching
  checkCudaErrors(hipStreamBeginCapture(stream, hipStreamCaptureModeGlobal));
  
  // Use MemoryManager's prefetching method with internal storage
  memManager.prefetchAllDataToDevice(
    optimizedGraph.arraysInitiallyAllocatedOnDevice,
    prefetchMemcpyKind,
    stream
  );
  
  // End capture and instantiate the initial data distribution graph
  hipGraph_t graphForInitialDataDistribution;
  checkCudaErrors(hipStreamEndCapture(stream, &graphForInitialDataDistribution));

  // Execute the initial data distribution
  hipGraphExec_t graphExecForInitialDataDistribution;
  checkCudaErrors(hipGraphInstantiate(&graphExecForInitialDataDistribution, graphForInitialDataDistribution, nullptr, nullptr, 0));
  checkCudaErrors(hipGraphLaunch(graphExecForInitialDataDistribution, stream));
  checkCudaErrors(hipDeviceSynchronize());

  //----------------------------------------------------------------------
  // STEP 5: Build the optimized execution graph by processing nodes in topological order
  //----------------------------------------------------------------------
  
  LOG_TRACE_WITH_INFO("Record nodes to a new CUDA Graph");

  // Maps nodes to their dependencies in the CUDA graph
  std::map<int, std::vector<hipGraphNode_t>> nodeToDependentNodesMap;

  // Kahn's Algorithm for topological sort and graph construction
  while (!nodesToExecute.empty()) {
    // Get the next node to process
    auto u = nodesToExecute.front();
    nodesToExecute.pop();

    std::vector<hipGraphNode_t> newLeafNodes;
    auto nodeType = optimizedGraph.nodeIdToNodeTypeMap[u];
    
    // Process different node types
    if (nodeType == OptimizationOutput::NodeType::dataMovement) {
      //----------------------------------------------------------------------
      // STEP 5a: Handle data movement nodes (prefetch or offload)
      //----------------------------------------------------------------------
      
      optimizedCudaGraphCreator->beginCaptureOperation(nodeToDependentNodesMap[u]);
      auto &dataMovement = optimizedGraph.nodeIdToDataMovementMap[u];
      auto dataMovementAddress = memManager.getPointerByArrayId(dataMovement.arrayId);
      auto dataMovementSize = memManager.getSizeByArrayId(dataMovement.arrayId);
      
      if (dataMovement.direction == OptimizationOutput::DataMovement::Direction::hostToDevice) {
        // PREFETCH: Move data from storage to device
        void *devicePtr;
        checkCudaErrors(hipMallocAsync(&devicePtr, dataMovementSize, stream));
        checkCudaErrors(hipMemcpyAsync(
          devicePtr,
          memManager.getDeviceToHostArrayMap().at(dataMovementAddress),
          dataMovementSize,
          prefetchMemcpyKind,
          stream
        ));
       
        memManager.updateCurrentMapping(dataMovementAddress, devicePtr);
      } else {
        // OFFLOAD: Move data from device back to storage and free device memory
        void *devicePtr = memManager.getCurrentAddressMap().at(dataMovementAddress);
        checkCudaErrors(hipMemcpyAsync(
          memManager.getDeviceToHostArrayMap().at(dataMovementAddress),
          devicePtr,
          dataMovementSize,
          offloadMemcpyKind,
          stream
        ));
        checkCudaErrors(hipFreeAsync(devicePtr, stream));
        memManager.removeCurrentMapping(dataMovementAddress);
      }
      
      checkCudaErrors(hipPeekAtLastError());
      newLeafNodes = optimizedCudaGraphCreator->endCaptureOperation();
      checkCudaErrors(hipPeekAtLastError());
      
    } else if (nodeType == OptimizationOutput::NodeType::task) {
      //----------------------------------------------------------------------
      // STEP 5b: Handle computation task nodes
      //----------------------------------------------------------------------
      
      optimizedCudaGraphCreator->beginCaptureOperation(nodeToDependentNodesMap[u]);
      // Execute the task with current memory address mapping
      executeRandomTask(
        optimizedGraph.nodeIdToTaskIdMap[u],
        memManager.getEditableCurrentAddressMap(),
        stream
      );
      newLeafNodes = optimizedCudaGraphCreator->endCaptureOperation();
      
    } else if (nodeType == OptimizationOutput::NodeType::empty) {
      //----------------------------------------------------------------------
      // STEP 5c: Handle empty nodes (for dependencies)
      //----------------------------------------------------------------------
      
      newLeafNodes.push_back(
        optimizedCudaGraphCreator->addEmptyNode(nodeToDependentNodesMap[u])
      );
    } else {
      LOG_TRACE_WITH_INFO("Unsupported node type: %d", nodeType);
      exit(-1);
    }

    // Update dependencies and process nodes that have all dependencies satisfied
    for (auto &v : optimizedGraph.edges[u]) {
      inDegrees[v]--;

      // Add dependencies for the next nodes
      nodeToDependentNodesMap[v].insert(
        nodeToDependentNodesMap[v].end(),
        newLeafNodes.begin(),
        newLeafNodes.end()
      );

      // If all dependencies are satisfied, add to the queue
      if (inDegrees[v] == 0) {
        nodesToExecute.push(v);
      }
    }
  }

  // Export graph for debugging/visualization
  LOG_TRACE_WITH_INFO("Printing the new CUDA Graph to newGraph.dot");
  checkCudaErrors(hipGraphDebugDotPrint(graph, "newGraph.dot", 0));

  //----------------------------------------------------------------------
  // STEP 6: Execute the optimized graph
  //----------------------------------------------------------------------
  
  LOG_TRACE_WITH_INFO("Execute the new CUDA Graph");
  
  // Set up profiling if requested
  PeakMemoryUsageProfiler peakMemoryUsageProfiler;
  CudaEventClock cudaEventClock;
  
  // Instantiate the graph for execution
  hipGraphExec_t graphExec;
  checkCudaErrors(hipGraphInstantiate(&graphExec, graph, nullptr, nullptr, 0));

  // Upload the graph to the device for faster execution
  checkCudaErrors(hipGraphUpload(graphExec, stream));
  checkCudaErrors(hipStreamSynchronize(stream));

  // Start memory usage profiling if requested
  if (ConfigurationManager::getConfig().execution.measurePeakMemoryUsage) {
    peakMemoryUsageProfiler.start();
  }

  // Execute and time the graph
  cudaEventClock.start();
  checkCudaErrors(hipGraphLaunch(graphExec, stream));
  cudaEventClock.end();
  checkCudaErrors(hipDeviceSynchronize());

  // Report peak memory usage if requested
  if (ConfigurationManager::getConfig().execution.measurePeakMemoryUsage) {
    const auto peakMemoryUsage = peakMemoryUsageProfiler.end();
    LOG_TRACE_WITH_INFO(
      "Peak memory usage (MiB): %.2f",
      static_cast<float>(peakMemoryUsage) / 1024.0 / 1024.0
    );
  }

  //----------------------------------------------------------------------
  // STEP 7: Clean up resources and copy any remaining data back to storage
  //----------------------------------------------------------------------
  
  LOG_TRACE_WITH_INFO("Clean up");
  
  // Copy any remaining device data back to storage
  auto &currentAddressMap = memManager.getEditableCurrentAddressMap();
  for (auto &[oldAddr, newAddr] : currentAddressMap) {
    checkCudaErrors(hipMemcpy(
      memManager.getDeviceToHostArrayMap().at(oldAddr),
      newAddr,
      memManager.getSize(oldAddr),
      offloadMemcpyKind
    ));
    checkCudaErrors(hipFree(newAddr));
  }
  checkCudaErrors(hipDeviceSynchronize());

  // Clean up CUDA resources
  checkCudaErrors(hipGraphExecDestroy(graphExecForInitialDataDistribution));
  checkCudaErrors(hipGraphExecDestroy(graphExec));
  checkCudaErrors(hipGraphDestroy(graphForInitialDataDistribution));
  checkCudaErrors(hipGraphDestroy(graph));
  checkCudaErrors(hipStreamDestroy(stream));

  // Disable peer access if using NVLink
  if (ConfigurationManager::getConfig().execution.useNvlink) {
    disablePeerAccessForNvlink(mainDeviceId, storageDeviceId);
  }

  // Store the execution time
  runningTime = cudaEventClock.getTimeInSeconds();
}

/*
 * executeOptimizedGraph - Executes a CUDA graph with memory optimization
 * 
 * This version uses the singleton instance of MemoryManager.
 *
 * Parameters:
 * - optimizedGraph: The optimized computation graph to execute
 * - executeRandomTask: Callback to execute specific computation tasks
 * - runningTime: Output parameter to store the execution time
 */
void Executor::executeOptimizedGraph(
  OptimizationOutput &optimizedGraph,
  ExecuteRandomTask executeRandomTask,
  float &runningTime
) {
  // Call the implementation version that takes a MemoryManager instance
  executeOptimizedGraph(optimizedGraph, executeRandomTask, runningTime, MemoryManager::getInstance());
}

/*
 * executeOptimizedGraphRepeatedly - Executes a CUDA graph with memory optimization multiple times
 *
 * This function executes a computation graph that has been optimized to reduce memory usage
 * repeatedly until a termination condition is met. It enables benchmarking and iterative
 * workloads that process data larger than would fit in GPU memory alone.
 * 
 * Parameters:
 * - optimizedGraph: The optimized computation graph to execute
 * - executeRandomTask: Callback to execute specific computation tasks
 * - shouldContinue: Function that determines when to stop the execution loop
 * - numIterations: Output parameter that counts how many times the graph was executed
 * - runningTime: Output parameter to store the execution time
 * - memManager: Reference to the MemoryManager instance to use
 */
void Executor::executeOptimizedGraphRepeatedly(
  OptimizationOutput &optimizedGraph,
  ExecuteRandomTask executeRandomTask,
  ShouldContinue shouldContinue,
  int &numIterations,
  float &runningTime,
  MemoryManager &memManager
) {
  LOG_TRACE_WITH_INFO("Initialize");

  // Create CUDA resources
  hipGraph_t graph;
  checkCudaErrors(hipGraphCreate(&graph, 0));

  hipStream_t stream;
  checkCudaErrors(hipStreamCreate(&stream));

  auto optimizedCudaGraphCreator = std::make_unique<OptimizedCudaGraphCreator>(stream, graph);

  //----------------------------------------------------------------------
  // STEP 1: Prepare the graph for topological traversal using Kahn's algorithm
  //----------------------------------------------------------------------
  
  // Calculate in-degrees for each node in the optimized graph
  std::map<int, int> inDegrees;
  for (auto &[u, outEdges] : optimizedGraph.edges) {
    for (auto &v : outEdges) {
      inDegrees[v] += 1;
    }
  }

  // Find root nodes (nodes with no dependencies)
  std::queue<int> nodesToExecute;
  std::vector<int> rootNodes;
  for (auto &u : optimizedGraph.nodes) {
    if (inDegrees[u] == 0) {
      nodesToExecute.push(u);
      rootNodes.push_back(u);
    }
  }

  //----------------------------------------------------------------------
  // STEP 2: Configure the device and memory settings
  //----------------------------------------------------------------------
  
  // Set up device configuration for data movement
  int mainDeviceId = ConfigurationManager::getConfig().execution.mainDeviceId;
  int storageDeviceId = hipCpuDeviceId;  // Default: use host memory as storage
  hipMemcpyKind prefetchMemcpyKind = hipMemcpyHostToDevice;
  hipMemcpyKind offloadMemcpyKind = hipMemcpyDeviceToHost;

  // If NVLink is available, use a second GPU as storage instead of host memory
  if (ConfigurationManager::getConfig().execution.useNvlink) {
    storageDeviceId = ConfigurationManager::getConfig().execution.storageDeviceId;
    prefetchMemcpyKind = hipMemcpyDeviceToDevice;
    offloadMemcpyKind = hipMemcpyDeviceToDevice;
    enablePeerAccessForNvlink(ConfigurationManager::getConfig().execution.mainDeviceId, ConfigurationManager::getConfig().execution.storageDeviceId);
  }

  //----------------------------------------------------------------------
  // STEP 3: Initialize managed data distribution
  //----------------------------------------------------------------------
  
  LOG_TRACE_WITH_INFO("Initialize managed data distribution");

  // Configure MemoryManager storage parameters
  memManager.configureStorage(
    mainDeviceId, 
    storageDeviceId, 
    ConfigurationManager::getConfig().execution.useNvlink
  );
  
  // Move all managed data to storage (host or secondary GPU) using MemoryManager's internal storage
  memManager.moveAllManagedMemoryToStorage();
  
  // Switch back to main GPU
  checkCudaErrors(hipSetDevice(mainDeviceId));
  checkCudaErrors(hipDeviceSynchronize());

  // Start timing the graph creation process
  SystemWallClock clock;
  clock.start();

  LOG_TRACE_WITH_INFO("Record nodes to a new CUDA Graph");

  //----------------------------------------------------------------------
  // STEP 4: Build the optimized execution graph
  //----------------------------------------------------------------------
  
  // Track memory addresses that have been updated (device copies)
  memManager.clearCurrentMappings();
  
  // Initial data allocation is integrated with main graph capture
  // rather than as a separate step like in executeOptimizedGraph
  std::vector<hipGraphNode_t> newLeafNodes;
  
  // Use MemoryManager's prefetching method with internal storage
  for (auto arrayId : optimizedGraph.arraysInitiallyAllocatedOnDevice) {
    auto ptr = memManager.getPointerByArrayId(arrayId);
    auto size = memManager.getSizeByArrayId(arrayId);

    // Allocate on device and copy data from storage
    void *devicePtr;
    optimizedCudaGraphCreator->beginCaptureOperation(newLeafNodes);
    checkCudaErrors(hipMallocAsync(&devicePtr, size, stream));
    checkCudaErrors(hipMemcpyAsync(
      devicePtr, 
      memManager.getDeviceToHostArrayMap().at(ptr),
      size, 
      prefetchMemcpyKind, 
      stream
    ));
    newLeafNodes = optimizedCudaGraphCreator->endCaptureOperation();
    memManager.updateCurrentMapping(ptr, devicePtr);
  }

  // Maps nodes to their dependencies in the CUDA graph
  std::map<int, std::vector<hipGraphNode_t>> nodeToDependentNodesMap;

  // Set up dependencies for root nodes
  for (auto u : rootNodes) {
    nodeToDependentNodesMap[u] = newLeafNodes;
  }

  // Process nodes in topological order (Kahn's Algorithm)
  while (!nodesToExecute.empty()) {
    // Get the next node to process
    auto u = nodesToExecute.front();
    nodesToExecute.pop();

    newLeafNodes.clear();

    // Process different node types
    auto nodeType = optimizedGraph.nodeIdToNodeTypeMap[u];
    if (nodeType == OptimizationOutput::NodeType::dataMovement) {
      // Handle data movement nodes (prefetch or offload)
      optimizedCudaGraphCreator->beginCaptureOperation(nodeToDependentNodesMap[u]);
      auto &dataMovement = optimizedGraph.nodeIdToDataMovementMap[u];
      auto dataMovementAddress = memManager.getPointerByArrayId(dataMovement.arrayId);
      auto dataMovementSize = memManager.getSizeByArrayId(dataMovement.arrayId);
      
      if (dataMovement.direction == OptimizationOutput::DataMovement::Direction::hostToDevice) {
        memManager.prefetchToDevice(dataMovement.arrayId,stream);
      } else {
        memManager.offloadFromDevice(dataMovement.arrayId,stream);
      }
      checkCudaErrors(hipPeekAtLastError());
      newLeafNodes = optimizedCudaGraphCreator->endCaptureOperation();
      checkCudaErrors(hipPeekAtLastError());
    } else if (nodeType == OptimizationOutput::NodeType::task) {
      // Handle computation task nodes
      optimizedCudaGraphCreator->beginCaptureOperation(nodeToDependentNodesMap[u]);
      // Execute the task with current memory address mapping
      executeRandomTask(
        optimizedGraph.nodeIdToTaskIdMap[u],
        memManager.getEditableCurrentAddressMap(),
        stream
      );
      newLeafNodes = optimizedCudaGraphCreator->endCaptureOperation();
    } else if (nodeType == OptimizationOutput::NodeType::empty) {
      // Handle empty nodes (for dependencies)
      newLeafNodes.push_back(
        optimizedCudaGraphCreator->addEmptyNode(nodeToDependentNodesMap[u])
      );
    } else {
      LOG_TRACE_WITH_INFO("Unsupported node type: %d", nodeType);
      exit(-1);
    }

    // Update dependencies and process nodes that have all dependencies satisfied
    for (auto &v : optimizedGraph.edges[u]) {
      inDegrees[v]--;

      // Add dependencies for the next nodes
      nodeToDependentNodesMap[v].insert(
        nodeToDependentNodesMap[v].end(),
        newLeafNodes.begin(),
        newLeafNodes.end()
      );

      // If all dependencies are satisfied, add to the queue
      if (inDegrees[v] == 0) {
        nodesToExecute.push(v);
      }
    }
  }

  // Add cleanup operations for any remaining device memory
  newLeafNodes = getNodesWithZeroOutDegree(graph);
  auto &currentAddressMap = memManager.getEditableCurrentAddressMap(); 
  for (auto &[oldAddr, newAddr] : currentAddressMap) {
    // Copy any remaining device data back to storage and free device memory
    optimizedCudaGraphCreator->beginCaptureOperation(newLeafNodes);
    checkCudaErrors(hipMemcpyAsync(
      memManager.getDeviceToHostArrayMap().at(oldAddr),
      newAddr,
      memManager.getSize(oldAddr),
      offloadMemcpyKind,
      stream
    ));
    checkCudaErrors(hipFreeAsync(newAddr, stream));
    newLeafNodes = optimizedCudaGraphCreator->endCaptureOperation();
  }
  checkCudaErrors(hipDeviceSynchronize());

  // Report time taken to build the graph
  clock.end();
  LOG_TRACE_WITH_INFO("Time taken for recording graph: %.6f", clock.getTimeInSeconds());

  // Export graph for debugging/visualization
  LOG_TRACE_WITH_INFO("Printing the new CUDA Graph to newGraph.dot");
  checkCudaErrors(hipGraphDebugDotPrint(graph, "newGraph.dot", 0));

  //----------------------------------------------------------------------
  // STEP 5: Execute the optimized graph repeatedly
  //----------------------------------------------------------------------
  
  LOG_TRACE_WITH_INFO("Execute the new CUDA Graph");
  
  // Set up profiling if requested
  PeakMemoryUsageProfiler peakMemoryUsageProfiler;
  CudaEventClock cudaEventClock;
  
  // Instantiate the graph for execution
  hipGraphExec_t graphExec;
  checkCudaErrors(hipGraphInstantiate(&graphExec, graph, nullptr, nullptr, 0));

  // Upload the graph to the device for faster execution
  checkCudaErrors(hipGraphUpload(graphExec, stream));
  checkCudaErrors(hipStreamSynchronize(stream));

  // Start memory usage profiling if requested
  if (ConfigurationManager::getConfig().execution.measurePeakMemoryUsage) {
    peakMemoryUsageProfiler.start();
  }

  // Initialize iteration counter
  numIterations = 0;

  // Execute and time the graph
  cudaEventClock.start();
  // Execute graph repeatedly until termination condition is met
  while (shouldContinue()) {
    checkCudaErrors(hipGraphLaunch(graphExec, stream));
    numIterations++;
    checkCudaErrors(hipDeviceSynchronize());
  }
  cudaEventClock.end();
  checkCudaErrors(hipDeviceSynchronize());

  // Report peak memory usage if requested
  if (ConfigurationManager::getConfig().execution.measurePeakMemoryUsage) {
    const auto peakMemoryUsage = peakMemoryUsageProfiler.end();
    LOG_TRACE_WITH_INFO(
      "Peak memory usage (MiB): %.2f",
      static_cast<float>(peakMemoryUsage) / 1024.0 / 1024.0
    );
  }

  //----------------------------------------------------------------------
  // STEP 6: Clean up resources
  //----------------------------------------------------------------------
  
  LOG_TRACE_WITH_INFO("Clean up");
  
  // Clean up CUDA resources
  checkCudaErrors(hipGraphExecDestroy(graphExec));
  checkCudaErrors(hipGraphDestroy(graph));
  checkCudaErrors(hipStreamDestroy(stream));

  // Disable peer access if using NVLink
  if (ConfigurationManager::getConfig().execution.useNvlink) {
    disablePeerAccessForNvlink(mainDeviceId, storageDeviceId);
  }

  // Store the execution time
  runningTime = cudaEventClock.getTimeInSeconds();
}

/*
 * executeOptimizedGraphRepeatedly - Executes a CUDA graph with memory optimization repeatedly
 * 
 * This version uses the singleton instance of MemoryManager.
 */
void Executor::executeOptimizedGraphRepeatedly(
  OptimizationOutput &optimizedGraph,
  ExecuteRandomTask executeRandomTask,
  ShouldContinue shouldContinue,
  int &numIterations,
  float &runningTime
) {
  // Call the implementation version that takes a MemoryManager instance
  executeOptimizedGraphRepeatedly(
    optimizedGraph, 
    executeRandomTask, 
    shouldContinue, 
    numIterations, 
    runningTime, 
    MemoryManager::getInstance()
  );
}

/**
 * @brief Helper method for initializing data distribution using internal storage
 * 
 * Moves all managed memory to storage and initializes needed data on device.
 * Uses MemoryManager's internal storage map.
 * 
 * @param optimizedGraph The optimization plan to execute
 * @param mainDeviceId ID of the main computation device
 * @param storageDeviceId ID of the storage device
 * @param useNvlink Whether to use NVLink for data transfers
 * @param stream CUDA stream to use for operations
 * @return hipGraphExec_t Executable graph for initial data setup
 */
hipGraphExec_t Executor::initializeDataDistribution(
  OptimizationOutput &optimizedGraph,
  int mainDeviceId, 
  int storageDeviceId,
  bool useNvlink,
  hipStream_t stream,
  hipMemcpyKind prefetchMemcpyKind
) {
  LOG_TRACE_WITH_INFO("Initialize data distribution with internal storage");
  
  // Move all managed data to storage (host or secondary GPU) using MemoryManager
  auto& memManager = MemoryManager::getInstance();
  
  // Configure storage parameters
  memManager.configureStorage(mainDeviceId, storageDeviceId, useNvlink);
  
  // Use the simplified API that uses internal storage
  memManager.moveAllManagedMemoryToStorage();
  
  // Reset the current assignment map
  memManager.clearCurrentMappings();
  
  // Create a subgraph for initial data prefetching
  hipGraph_t graphForInitialDataDistribution;
  checkCudaErrors(hipStreamBeginCapture(stream, hipStreamCaptureModeGlobal));
  
  // Prefetch arrays that need to be on device initially
  memManager.prefetchAllDataToDevice(
    optimizedGraph.arraysInitiallyAllocatedOnDevice,
    prefetchMemcpyKind,
    stream
  );
  
  // End capture and instantiate graph
  checkCudaErrors(hipStreamEndCapture(stream, &graphForInitialDataDistribution));
  
  // Create executable graph
  hipGraphExec_t graphExecForInitialDataDistribution;
  checkCudaErrors(hipGraphInstantiate(
    &graphExecForInitialDataDistribution, 
    graphForInitialDataDistribution, 
    nullptr, 
    nullptr, 
    0
  ));
  
  // Execute the initial data distribution
  checkCudaErrors(hipGraphLaunch(graphExecForInitialDataDistribution, stream));
  checkCudaErrors(hipDeviceSynchronize());
  
  // We'll clean up the graph itself, but return the executable for the caller to clean up
  checkCudaErrors(hipGraphDestroy(graphForInitialDataDistribution));
  
  return graphExecForInitialDataDistribution;
}

}  // namespace memopt